#include "hip/hip_runtime.h"

#include "DFSPH_cuda_basic.h"

#include <stdio.h>
#include <chrono>
#include <iostream>
#include <thread>
#include <iomanip>

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"
#include "DFSPH_static_variables_structure_cuda.h"


#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"


#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "basic_kernels_cuda.cuh"


#include <sstream>


/*
AFTER THIS ARE ONLY THE TEST FUNCTION TO HAVE CUDA WORKING ...
*/


inline __host__ __device__ float3 make_float3(float s)
{
    return make_float3(s, s, s);
}

inline __host__ __device__ float4 make_float4(float s)
{
    return make_float4(s, s, s, s);
}

inline __host__ __device__ float4 operator*(float4& a, RealCuda b)
{
    return make_float4(a.x * b, a.y * b, a.z * b, 0);
}

inline __host__ __device__ float3 operator*(float3& a, RealCuda b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

inline __host__ __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}



template<typename T>
__global__ void test_vector_type_kernel(T* v1, T* v2, RealCuda factor, int count_elem) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count_elem) { return; }

    v1[i]+=v2[i]*factor;
}

void compare_vector3_struct_speed(){
    RealCuda factor=0.001;
    int count_elem= 1000000;
    Vector3d* v1_v3d;
    Vector3d* v2_v3d;
    float3* v1_f3;
    float3* v2_f3;
    float4* v1_f4;
    float4* v2_f4;
    Vector3d* v1_v3d_2=new Vector3d[count_elem];
    Vector3d* v2_v3d_2=new Vector3d[count_elem];
    float3* v1_f3_2=new float3[count_elem];
    float3* v2_f3_2=new float3[count_elem];
    float4* v1_f4_2=new float4[count_elem];
    float4* v2_f4_2=new float4[count_elem];
    hipMalloc(&(v1_v3d), count_elem * sizeof(Vector3d));
    hipMalloc(&(v2_v3d), count_elem * sizeof(Vector3d));

    hipMalloc(&(v1_f3), count_elem * sizeof(float3));
    hipMalloc(&(v2_f3), count_elem * sizeof(float3));

    hipMalloc(&(v1_f4), count_elem * sizeof(float4));
    hipMalloc(&(v2_f4), count_elem * sizeof(float4));

    for (int i=0;i<count_elem;++i){
        v1_v3d_2[i]=i;
        v2_v3d_2[i]=i;
        v1_f3_2[i]=make_float3(i);
        v2_f3_2[i]=make_float3(i);
        v1_f4_2[i]=make_float4(i);
        v2_f4_2[i]=make_float4(i);
    }

    gpuErrchk(hipMemcpy(v1_v3d, v1_v3d_2,count_elem * sizeof(Vector3d),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_v3d, v2_v3d_2,count_elem * sizeof(Vector3d),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v1_f3, v1_f3_2,count_elem * sizeof(float3),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_f3, v2_f3_2,count_elem * sizeof(float3),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v1_f4, v1_f4_2,count_elem * sizeof(float4),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_f4, v2_f4_2,count_elem * sizeof(float4),hipMemcpyHostToDevice));

    int numBlocks = (count_elem + BLOCKSIZE - 1) / BLOCKSIZE;
    gpuErrchk(hipDeviceSynchronize());

    float avg0=0;
    float avg1=0;
    float avg2=0;

    int iter=10;
    for (int i=0;i<iter;++i){
        std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();
        test_vector_type_kernel<float3> << <numBlocks, BLOCKSIZE >> > (v1_f3, v2_f3, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();
        test_vector_type_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (v1_v3d, v2_v3d, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();
        test_vector_type_kernel<float4> << <numBlocks, BLOCKSIZE >> > (v1_f4, v2_f4, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t3 = std::chrono::steady_clock::now();

        float time_0 = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
        float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;
        float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (t3 - t2).count() / 1000000.0f;

        printf("comparison between vector data struct  (float3, Vector3d, float4): %f   %f   %f\n", time_0, time_1, time_2);

        if (iter>0){
            avg0+=time_0;
            avg1+=time_1;
            avg2+=time_2;
        }
    }
    iter--;

    printf("comparison between vector data struct Global (float3, Vector3d, float4): %f   %f   %f\n",
           avg0/iter, avg1/iter, avg2/iter);

}







hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
//*
__global__ void addKernel(Vector3d* vect)
{
    int i = threadIdx.x;
    vect[i].z = vect[i].x + vect[i].y;
}

__global__ void setVectkernel(Vector3d& vect)
{
    vect.x = 5;
    vect.y = 6;
    vect.z = 7;
}
//*/
int test_cuda()
{
    //DFSPHCData* data;
    std::cout << "start cuda test basic" << std::endl;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    //*
    Vector3d* vect;
    hipMallocManaged(&vect, arraySize * sizeof(Vector3d));
    for (int i = 0; i < arraySize; ++i) {
        vect[i].x = a[i];
        vect[i].y = b[i];
    }
    //*/*

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    printf("macro val: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }


    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, arraySize >> > (vect);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("with vects {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           (int)(vect[0].z), (int)(vect[1].z), (int)(vect[2].z), (int)(vect[3].z), (int)(vect[4].z));

    hipFree(vect);



    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
    }
    //*/

    printf("Finished test cuda\n");


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}



void test_particleshift() {
    compare_vector3_struct_speed();

#define count_pos 100
    RealCuda pos[count_pos];
    pos[0] = 0;

    auto normalize_string = [](std::string s, int length) {
        int to_add = length - s.length();
        std::ostringstream oss;
        oss << s;
        for (int i = 0; i < to_add; ++i) {
            oss << " ";
        }
        return oss.str();
    };

#define W(p1,p2) (MAX_MACRO_CUDA(1-abs(p2-p1),0))
#define W_SIGNED(p1,p2)  W(p1,p2)*((p2>p1)?1:-1)
#define W_GRAD(p1,p2) ((W(p1,p2)>0)?((p2-p1)<0?-1:1):0)

    RealCuda mass[count_pos];
    RealCuda density[count_pos];
    RealCuda c[count_pos];
    RealCuda nabla_c[count_pos];
    for (int i = 0; i < count_pos; ++i) {
        mass[i] = 1;

        if (i > 0 && i < 33) {
            pos[i] = pos[i - 1] + 0.5;
        }else if(i>32){
            pos[i] = pos[i - 1] + 0.25;
        }

       // std::cout << "pos: " << i << "  " <<  pos[i] << std::endl;

    }

    std::vector<std::vector<RealCuda>> hist_pos;
    std::vector<std::vector<RealCuda>> hist_den;

    //concentration based
    if(false){
        for (int repeat = 0; repeat < 5; ++repeat) {

            for (int i = 0; i < count_pos; ++i) {
                density[i] = 0;
                for (int j = 0; j < count_pos; ++j) {
                    density[i] += mass[j]*W(pos[j],pos[i]);
                    if (i == 33) {
                        //std::cout << "W/grad: " <<j<<"  "<< W(pos[j], pos[i]) <<"  "<< W_GRAD(pos[j], pos[i]) << std::endl;
                    }
                }
            }


            for (int i = 0; i < count_pos; ++i) {
                c[i] = 0;
                for (int j = 0; j < count_pos; ++j) {
                    if (i != j) {
                        c[i] += (mass[j] / density[j]) * W(pos[j], pos[i]);
                    }
                }
            }

            for (int i = 0; i < count_pos; ++i) {
                nabla_c[i] = 0;
                for (int j = 0; j < count_pos; ++j) {
                    if (i != j) {
                        nabla_c[i] += (c[j]-c[i]) * (mass[j] / density[j]) * W_GRAD(pos[j], pos[i]);
                    }
                }
            }


            /*
            
            for (int i = 0; i < count_pos; ++i) {
                std::ostringstream oss;
                oss << std::setprecision(2) << pos[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            std::cout << std::endl;

            for (int i = 0; i < count_pos; ++i) {
                std::ostringstream oss;
                oss << std::setprecision(2) << density[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            std::cout << std::endl;

            for (int i = 0; i < count_pos; ++i) {
                std::ostringstream oss;
                oss << std::setprecision(2) << c[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            std::cout << std::endl;

            for (int i = 0; i < count_pos; ++i) {
                std::ostringstream oss;
                oss << std::setprecision(2) << nabla_c[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            std::cout << std::endl;
            //*/
            std::vector<RealCuda> vect_pos;
            std::vector<RealCuda> vect_den;
            for (int i = 0; i < count_pos; ++i) {
                vect_pos.push_back(pos[i]);
                vect_den.push_back(density[i]);
            }
            hist_pos.push_back(vect_pos);
            hist_den.push_back(vect_den);


            RealCuda pos2[count_pos];
            RealCuda density2[count_pos];

            RealCuda factor = 0.5;
            for (int i = 0; i < count_pos; ++i) {
                pos2[i] = pos[i] + nabla_c[i] * factor;
            }

            for (int i = 0; i < count_pos; ++i) {
                density2[i] = 0;
                for (int j = 0; j < count_pos; ++j) {
                    density2[i] += mass[j] * W(pos2[j], pos2[i]);
                }
            }


            for (int i = 0; i < count_pos; ++i) {
                std::cout << pos[i] << "   " << density[i] << "   " << c[i] << "   " << nabla_c[i] << "   " << pos2[i] << "   " << density2[i] << "   ";
                std::cout << std::endl;
            }

            for (int i = 0; i < count_pos; ++i) {
                pos[i] = pos2[i];
            }

        }
    }

    //simply densty based
    //not much better
    for (int repeat = 0; repeat < 5; ++repeat) {
        for (int i = 0; i < count_pos; ++i) {
            density[i] = 0;
            for (int j = 0; j < count_pos; ++j) {
                density[i] += mass[j] * W(pos[j], pos[i]);
            }
        }

        for (int i = 0; i < count_pos; ++i) {
            nabla_c[i] = 0;
            for (int j = 0; j < count_pos; ++j) {
                if (i != j) {
                    nabla_c[i] += (density[j] - density[i])* (mass[j]/density[j]) * W_SIGNED(pos[j], pos[i]);
                }
            }
        }

        for (int i = 0; i < count_pos; ++i) {
            //std::cout << pos[i] << "   " << density[i] << "   " << c[i] << "   " << nabla_c[i];
            {
                std::ostringstream oss;
                oss << std::setprecision(2) << pos[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            {
                std::ostringstream oss;
                oss << std::setprecision(2) << density[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            {
                std::ostringstream oss;
                oss << std::setprecision(2) << nabla_c[i];
                std::cout << normalize_string(oss.str(), 6) << "   ";
            }
            std::cout << std::endl;
        }

        std::vector<RealCuda> vect_pos;
        std::vector<RealCuda> vect_den;
        for (int i = 0; i < count_pos; ++i) {
            vect_pos.push_back(pos[i]);
            vect_den.push_back(density[i]);
        }
        hist_pos.push_back(vect_pos);
        hist_den.push_back(vect_den);


        for (int i = 0; i < count_pos; ++i) {
            if (abs(nabla_c[i]) > 0.15) {
                nabla_c[i] *= 0.15 / abs(nabla_c[i]);
            }
            pos[i] += nabla_c[i];
        }
    }
    

    //ok so let's trys with a regular space sampling


    /*
    std::cout << "end results" << std::endl;
    std::cout << "end results" << std::endl;
    std::cout << "end results" << std::endl;
    std::ostringstream oss;
    for (int k = 0; k < hist_pos.size(); ++k) {
        oss << "pos_gen_"<< k << "  " << "den_gen_" << k << "  ";
    }
    oss << std::endl;
    for (int i = 0; i < count_pos; ++i) {
        for (int k = 0; k < hist_pos.size(); ++k) {
            oss << hist_pos[k][i] << "  " << hist_den[k][i] << "  ";
        }
        oss << std::endl;
    }
    std::cout <<oss.str();
    //*/

#undef W
#undef W_SIGNED
#undef W_GRAD
}