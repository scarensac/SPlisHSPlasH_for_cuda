#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_cuda_basic.h"
#include <stdio.h>
#include "DFSPH_c_arrays_structure.h"

#define BLOCKSIZE 256
#define m_eps 1.0e-5

__global__ void DFSPH_density_kernel(SPH::DFSPHCData m_data)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) {return;}
	
	// Compute current density for particle i
	Real density = m_data.mass[i] * m_data.W_zero;
	const Vector3d &xi = m_data.posFluid[i];


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	for (unsigned int j = 0; j < m_data.getNumberOfNeighbourgs(i); j++)
	{
		const unsigned int neighborIndex = m_data.getNeighbour(i, j);
		const Vector3d &xj = m_data.posFluid[neighborIndex];
		density += m_data.mass[neighborIndex] * m_data.W(xi - xj);
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	for (unsigned int pid = 1; pid < 2; pid++)
	{
		for (unsigned int j = 0; j < m_data.getNumberOfNeighbourgs(i, pid); j++)
		{
			const unsigned int neighborIndex = m_data.getNeighbour(i, j, pid);
			const Vector3d &xj = m_data.posBoundary[neighborIndex];

			// Boundary: Akinci2012
			density += m_data.boundaryPsi[neighborIndex] * m_data.W(xi - xj);
		}
	}
	//*/

	m_data.density[i] = density;
}

__global__ void DFSPH_factor_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) {return;}

	//////////////////////////////////////////////////////////////////////////
	// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
	//////////////////////////////////////////////////////////////////////////
	const Vector3d &xi = m_data.posFluid[i];
	Real sum_grad_p_k = 0.0;
	Vector3d grad_p_i;
	grad_p_i.setZero();

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	for (unsigned int j = 0; j < m_data.getNumberOfNeighbourgs(i); j++)
	{
		const unsigned int neighborIndex = m_data.getNeighbour(i, j);
		const Vector3d &xj = m_data.posFluid[neighborIndex];
		const Vector3d grad_p_j = -m_data.mass[neighborIndex] * m_data.gradW(xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i -= grad_p_j;
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	for (unsigned int pid = 1; pid < 2; pid++)
	{
		for (unsigned int j = 0; j < m_data.getNumberOfNeighbourgs(i, pid); j++)
		{
			const unsigned int neighborIndex = m_data.getNeighbour(i, j, pid);
			const Vector3d &xj = m_data.posBoundary[neighborIndex];
			const Vector3d grad_p_j = -m_data.boundaryPsi[neighborIndex] * m_data.gradW(xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i -= grad_p_j;
		}
	}

	sum_grad_p_k += grad_p_i.squaredNorm();

	//////////////////////////////////////////////////////////////////////////
	// Compute pressure stiffness denominator
	//////////////////////////////////////////////////////////////////////////


	sum_grad_p_k = max(sum_grad_p_k, m_eps);
	m_data.factor[i] = -1.0 / (sum_grad_p_k);

}

__global__ void DFSPH_viscosity_XSPH_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	//set the gravitation
	m_data.accFluid[i] = m_data.gravitation;

	const Real invH = (1.0 / m_data.h);

	const Vector3d &xi = m_data.posFluid[i];
	const Vector3d &vi = m_data.velFluid[i];
	Vector3d &ai = m_data.accFluid[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	for (unsigned int j = 0; j < m_data.getNumberOfNeighbourgs(i); j++)
	{
		const unsigned int neighborIndex = m_data.getNeighbour(i, j);
		const Vector3d &xj = m_data.posFluid[neighborIndex];
		const Vector3d &vj = m_data.velFluid[neighborIndex];

		// Viscosity
		const Real density_j = m_data.density[neighborIndex];
		ai -= invH * m_data.viscosity * (m_data.mass[neighborIndex] / density_j) * (vi - vj) * m_data.W(xi - xj);
	}

}

__global__ void DFSPH_updateVelocity_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.velFluid[i] += m_data.h * m_data.accFluid[i];
}

__global__ void DFSPH_updatePosition_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.posFluid[i] += m_data.h * m_data.velFluid[i];
}

void cuda_compute_density(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_density_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

void cuda_computeDFSPHFactor(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_factor_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

void cuda_viscosity_XSPH(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_viscosity_XSPH_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

void cuda_updateVelocities(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_updateVelocity_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

void cuda_updatePositions(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_updatePosition_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}










/*
	THE NEXT FUNCTIONS ARE FOR THE MEMORY ALLOCATION
*/

void allocate_c_array_struct_cuda_managed(SPH::DFSPHCData& data) {
	//hipMallocManaged(&x, N * sizeof(float));
	//hipMallocManaged(&y, N * sizeof(float));

	hipMallocManaged(&(data.posBoundary), data.numBoundaryParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.velBoundary), data.numBoundaryParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.velBoundary), data.numBoundaryParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.boundaryPsi), data.numBoundaryParticles * sizeof(Real));


	//handle the fluid
	hipMallocManaged(&(data.mass), data.numFluidParticles * sizeof(Real));
	hipMallocManaged(&(data.posFluid), data.numFluidParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.velFluid), data.numFluidParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.accFluid), data.numFluidParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.numberOfNeighbourgs), data.numFluidParticles * 2 * sizeof(int));
	hipMallocManaged(&(data.neighbourgs), data.numFluidParticles * 2 * MAX_NEIGHBOURS * sizeof(int));

	hipMallocManaged(&(data.density), data.numFluidParticles * sizeof(Real));
	hipMallocManaged(&(data.factor), data.numFluidParticles * sizeof(Real));
	hipMallocManaged(&(data.kappa), data.numFluidParticles * sizeof(Real));
	hipMallocManaged(&(data.kappaV), data.numFluidParticles * sizeof(Real));
	hipMallocManaged(&(data.densityAdv), data.numFluidParticles * sizeof(Real));
}


void allocate_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel){
	hipMallocManaged(&(kernel.m_W), kernel.m_resolution * sizeof(Real));
	hipMallocManaged(&(kernel.m_gradW), (kernel.m_resolution+1) * sizeof(Real));
}

/*
	AFTER THIS ARE ONLY THE TEST FUNCTION TO HAVE CUDA WORKING ...
*/


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//*
__global__ void addKernel(Vector3d* vect)
{
	int i = threadIdx.x;
	vect[i].z = vect[i].x + vect[i].y;
}

__global__ void setVectkernel(Vector3d& vect)
{
	vect.x = 5;
	vect.y = 6;
	vect.z = 7;
}
//*/
int test_cuda()
{
	//DFSPHCData* data;

	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	//*
	Vector3d* vect;
	hipMallocManaged(&vect, arraySize * sizeof(Vector3d));
	for (int i = 0; i < arraySize; ++i) {
		vect[i].x = a[i];
		vect[i].y = b[i];
	}
	//*/*

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	printf("macro val: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	for (int i = 0; i < arraySize; ++i) {
		c[i] = 0;
	}


	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, arraySize >> > (vect);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	printf("with vects {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		(int)(vect[0].z), (int)(vect[1].z), (int)(vect[2].z), (int)(vect[3].z), (int)(vect[4].z));

	hipFree(vect);



	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	/*
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	//*/

	printf("Finished test cuda\n");


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
