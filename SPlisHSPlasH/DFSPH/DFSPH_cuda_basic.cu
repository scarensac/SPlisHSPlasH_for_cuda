#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_cuda_basic.h"
#include <stdio.h>
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"
#include <chrono>
#include "SPlisHSPlasH/Utilities/Timing.h"
#include <iostream>

#define BLOCKSIZE 256
#define m_eps 1.0e-5
#define CELL_ROW_LENGTH 256
#define CELL_COUNT CELL_ROW_LENGTH*CELL_ROW_LENGTH*CELL_ROW_LENGTH

#define USE_WARMSTART
#define USE_WARMSTART_V

#define BITSHIFT_INDEX_DYNAMIC_BODIES

#ifdef BITSHIFT_INDEX_DYNAMIC_BODIES
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(global_index,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(global_index,body_index,particle_index)
#else
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(global_index,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(global_index,body_index,particle_index)
#endif

//those defines are to create and read the dynamic bodies indexes
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)  particle_index + (body_index << 0x10)
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)  particle_index + (body_index * 1000000)

//WARNING his one declare the body/particle index by itself
//you just have to give it the variable name you want
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(global_index, body_index,particle_index)  \
const unsigned int particle_index = global_index & 0xFFFF;\
const unsigned int body_index = (global_index & ~0xFFFF) >> 0x10;

#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(global_index, body_index,particle_index)   \
const unsigned int particle_index = global_index % (1000000);\
const unsigned int body_index=global_index / 1000000;


#define BITSHIFT_INDEX_NEIGHBORS_CELL

#ifdef BITSHIFT_INDEX_NEIGHBORS_CELL
#define COMPUTE_CELL_INDEX(x,y,z) 
#else
#define COMPUTE_CELL_INDEX(x,y,z) x+y*CELL_ROW_LENGTH+z*CELL_ROW_LENGTH*CELL_ROW_LENGTH
#endif



//those two variables are the identifiers that  link the ongle buffers to cuda
hipGraphicsResource_t vboRes_pos;
hipGraphicsResource_t vboRes_vel;

//easy function to check errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
//this is the bases for all kernels based function
__global__ void DFSPH__kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

}
void cuda_(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH__kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
//*/

FUNCTION inline int* getNeighboursPtr(int * neighbourgs, int particle_id) {
	//	return neighbourgs + body_id*numFluidParticles*MAX_NEIGHBOURS + particle_id*MAX_NEIGHBOURS;
	return neighbourgs + particle_id*MAX_NEIGHBOURS;
}

FUNCTION inline unsigned int getNumberOfNeighbourgs(int* numberOfNeighbourgs, int particle_id, int body_id = 0) {
	//return numberOfNeighbourgs[body_id*numFluidParticles + particle_id]; 
	return numberOfNeighbourgs[particle_id * 3 + body_id];
}

__device__ void computeDensityChange(SPH::DFSPHCData& m_data, const unsigned int index) {
	unsigned int numNeighbors = m_data.getNumberOfNeighbourgs(index);
	// in case of particle deficiency do not perform a divergence solve
	if (numNeighbors < 20) {
		for (unsigned int pid = 1; pid < 2; pid++)
		{
			numNeighbors += m_data.getNumberOfNeighbourgs(index, pid);
		}
	}
	if (numNeighbors < 20) {
		m_data.densityAdv[index] = 0;
	}
	else {
		RealCuda densityAdv = 0;
		const Vector3d &xi = m_data.posFluid[index];
		const Vector3d &vi = m_data.velFluid[index];
		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		int* neighbors_ptr = m_data.getNeighboursPtr(index); 
		int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(index);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			densityAdv += m_data.mass[neighborIndex] * (vi - m_data.velFluid[neighborIndex]).dot(m_data.gradW(xi - m_data.posFluid[neighborIndex]));
		}
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
		end_ptr += m_data.getNumberOfNeighbourgs(index,1);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			densityAdv += m_data.boundaryPsi[neighborIndex] * (vi - m_data.velBoundary[neighborIndex]).dot(m_data.gradW(xi - m_data.posBoundary[neighborIndex]));
		}

		//////////////////////////////////////////////////////////////////////////
		// Dynamic Bodies
		//////////////////////////////////////////////////////////////////////////
		end_ptr += m_data.getNumberOfNeighbourgs(index, 2);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int identifier = *neighbors_ptr++;
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
			SPH::RigidBodyContainer& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			densityAdv += body.psi[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
		}

		
	

		// only correct positive divergence
		m_data.densityAdv[index] = MAX_MACRO_CUDA(densityAdv, 0.0);
	}
}
template <bool warm_start> __device__ void divergenceSolveParticle(SPH::DFSPHCData& m_data, const unsigned int i) {
	Vector3d v_i = Vector3d(0, 0, 0);
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? m_data.kappaV[i] : (m_data.densityAdv[i])*m_data.factor[i];

#ifdef USE_WARMSTART_V
	if (!warm_start) { m_data.kappaV[i] += ki; }
#endif

	const Vector3d &xi = m_data.posFluid[i];
	

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(i);
	while (neighbors_ptr!=end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		const RealCuda kSum = (ki + ((warm_start) ? m_data.kappaV[neighborIndex] : (m_data.densityAdv[neighborIndex])*m_data.factor[neighborIndex]));
		if (fabs(kSum) > m_eps)
		{
			// ki, kj already contain inverse density
			v_i += kSum *  m_data.mass[neighborIndex] * m_data.gradW(xi - m_data.posFluid[neighborIndex]);
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.getNumberOfNeighbourgs(i, 1);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			///TODO fuse those lines
			const Vector3d delta = ki * m_data.boundaryPsi[neighborIndex] * m_data.gradW(xi - m_data.posBoundary[neighborIndex]);
			v_i += delta;// ki already contains inverse density

		}

	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.getNumberOfNeighbourgs(i, 2);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int identifier = *neighbors_ptr++;
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
			SPH::RigidBodyContainer& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			///TODO fuse those lines
			const Vector3d delta = ki * body.psi[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
			v_i += delta;// ki already contains inverse density

			///TODO reactivate this for objects see theoriginal sign to see the the actual sign
			//we apply the force to the body particle (no invH since it has been fatorized at the end)
			body.F[neighborIndex] -= m_data.mass[i] * delta;
		}

	}


	

	m_data.velFluid[i] += v_i*m_data.h;
}
__device__ void computeDensityAdv(SPH::DFSPHCData& m_data, const unsigned int index) {
	const Vector3d xi = m_data.posFluid[index];
	const Vector3d vi = m_data.velFluid[index];
	RealCuda delta = 0;

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.getNeighboursPtr(index); 
	int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(index);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += m_data.mass[neighborIndex] * (vi - m_data.velFluid[neighborIndex]).dot(m_data.gradW(xi - m_data.posFluid[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	end_ptr += m_data.getNumberOfNeighbourgs(index, 1);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += m_data.boundaryPsi[neighborIndex] * (vi - m_data.velBoundary[neighborIndex]).dot(m_data.gradW(xi - m_data.posBoundary[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	end_ptr += m_data.getNumberOfNeighbourgs(index, 2);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int identifier = *neighbors_ptr++;
		READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
		SPH::RigidBodyContainer& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
		delta += body.psi[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
	}


	
	
	m_data.densityAdv[index] = MAX_MACRO_CUDA(m_data.density[index] + m_data.h_future*delta - m_data.density0, 0.0);
}

__device__ void computeDensityAdv(const unsigned int index, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
	RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
	SPH::RigidBodyContainer* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
	const Vector3d xi = posFluid[index];
	const Vector3d vi = velFluid[index];
	RealCuda delta = 0;

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = getNeighboursPtr(neighbourgs, index);
	int* end_ptr = neighbors_ptr + getNumberOfNeighbourgs(numberOfNeighbourgs, index);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += mass[neighborIndex] * (vi - velFluid[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posFluid[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 1);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += boundaryPsi[neighborIndex] * (vi - velBoundary[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posBoundary[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 2);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int identifier = *neighbors_ptr++;
		READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
		SPH::RigidBodyContainer& body = vector_dynamic_bodies_data_cuda[bodyIndex];
		delta += body.psi[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_kernel_precomp.gradW(xi - body.pos[neighborIndex]));
	}




	densityAdv[index] = MAX_MACRO_CUDA(density[index] + h_future*delta - density0, 0.0);
}

template <bool warm_start> __device__ void pressureSolveParticle(SPH::DFSPHCData& m_data, const unsigned int i) {
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? m_data.kappa[i] : (m_data.densityAdv[i])*m_data.factor[i];

#ifdef USE_WARMSTART
	if (!warm_start) { m_data.kappa[i] += ki; }
#endif


	Vector3d v_i = Vector3d(0, 0, 0);
	const Vector3d &xi = m_data.posFluid[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(i);
	while (neighbors_ptr != end_ptr) 
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		const RealCuda kSum = (ki + ((warm_start) ? m_data.kappa[neighborIndex] : (m_data.densityAdv[neighborIndex])*m_data.factor[neighborIndex]));
		if (fabs(kSum) > m_eps)
		{
			// ki, kj already contain inverse density
			v_i += kSum * m_data.mass[neighborIndex] * m_data.gradW(xi - m_data.posFluid[neighborIndex]);
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.getNumberOfNeighbourgs(i,1);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			const Vector3d delta = ki * m_data.boundaryPsi[neighborIndex] * m_data.gradW(xi - m_data.posBoundary[neighborIndex]);

			v_i += delta;// ki already contains inverse density
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.getNumberOfNeighbourgs(i, 2);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int identifier = *neighbors_ptr++;
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
			SPH::RigidBodyContainer& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			const Vector3d delta = ki * body.psi[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);

			v_i += delta;// ki already contains inverse density

			///TODO reactivate the external forces check the original formula to be sure of the sign
			//we apply the force to the body particle (no invH since it has been fatorized at the end)
			body.F[neighborIndex] -= m_data.mass[i] * delta;
		}
	}

	

	// Directly update velocities instead of storing pressure accelerations
	m_data.velFluid[i] += v_i*m_data.h_future;
}

__global__ void DFSPH_divergence_warmstart_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.kappaV[i] = MAX_MACRO_CUDA(m_data.kappaV[i] * m_data.h_ratio_to_past / 2, -0.25);
	computeDensityChange(m_data, i);
}
void cuda_divergence_warmstart_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_warmstart_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_warmstart_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

template<bool warmstart> __global__ void DFSPH_divergence_compute_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	if (warmstart) {
		if (m_data.densityAdv[i] > 0.0) {
			divergenceSolveParticle<warmstart>(m_data, i);
		}
	}
	else {
		divergenceSolveParticle<warmstart>(m_data, i);
	}

}
template<bool warmstart> void cuda_divergence_compute(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_divergence_compute<true>(SPH::DFSPHCData& data);
template void cuda_divergence_compute<false>(SPH::DFSPHCData& data);

__global__ void DFSPH_divergence_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	{
		///TODO when doing this kernel I can actually fuse the code for all those computation to limit the number
		///of time I read the particles positions
		computeDensityChange(m_data, i);

		//I can actually make the factor and desity computation here
		{
			//////////////////////////////////////////////////////////////////////////
			// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
			//////////////////////////////////////////////////////////////////////////
			const Vector3d &xi = m_data.posFluid[i];
			RealCuda sum_grad_p_k = 0;
			Vector3d grad_p_i;
			grad_p_i.setZero();

			RealCuda density = m_data.mass[i] * m_data.W_zero;

			//////////////////////////////////////////////////////////////////////////
			// Fluid
			//////////////////////////////////////////////////////////////////////////
			int* neighbors_ptr = m_data.getNeighboursPtr(i);
			int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(i);
			while (neighbors_ptr != end_ptr)
			{
				const unsigned int neighborIndex = *neighbors_ptr++;
				const Vector3d &xj = m_data.posFluid[neighborIndex];
				density += m_data.mass[neighborIndex] * m_data.W(xi - xj);
				const Vector3d grad_p_j = m_data.mass[neighborIndex] * m_data.gradW(xi - xj);
				sum_grad_p_k += grad_p_j.squaredNorm();
				grad_p_i += grad_p_j;
			}

			//////////////////////////////////////////////////////////////////////////
			// Boundary
			//////////////////////////////////////////////////////////////////////////
			end_ptr += m_data.getNumberOfNeighbourgs(i,1);
			while (neighbors_ptr != end_ptr)
			{
				const unsigned int neighborIndex = *neighbors_ptr++;
				const Vector3d &xj = m_data.posBoundary[neighborIndex];
				density += m_data.boundaryPsi[neighborIndex] * m_data.W(xi - xj);
				const Vector3d grad_p_j = m_data.boundaryPsi[neighborIndex] * m_data.gradW(xi - xj);
				sum_grad_p_k += grad_p_j.squaredNorm();
				grad_p_i += grad_p_j;
			}

			//////////////////////////////////////////////////////////////////////////
			// Dynamic bodies
			//////////////////////////////////////////////////////////////////////////
			//*
			end_ptr += m_data.getNumberOfNeighbourgs(i, 2);
			while (neighbors_ptr != end_ptr)
			{
				const unsigned int identifier = *neighbors_ptr++;
				READ_DYNAMIC_BODIES_PARTICLES_INDEX(identifier, bodyIndex, neighborIndex);
				SPH::RigidBodyContainer& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
				const Vector3d &xj = body.pos[neighborIndex];
				density += body.psi[neighborIndex] * m_data.W(xi - xj);
				const Vector3d grad_p_j = body.psi[neighborIndex] * m_data.gradW(xi - xj);
				sum_grad_p_k += grad_p_j.squaredNorm();
				grad_p_i += grad_p_j;
			}
			//*/
			

			sum_grad_p_k += grad_p_i.squaredNorm();

			//////////////////////////////////////////////////////////////////////////
			// Compute pressure stiffness denominator
			//////////////////////////////////////////////////////////////////////////
			m_data.factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
			m_data.density[i] = density;

		}

#ifdef USE_WARMSTART_V
		m_data.kappaV[i] = 0;
#endif
	}

}
void cuda_divergence_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_divergence_loop_end_kernel(SPH::DFSPHCData m_data, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityChange(m_data, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}
RealCuda cuda_divergence_loop_end(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	RealCuda* avg_density_err=NULL;
	if (avg_density_err == NULL) {
		hipMalloc(&(avg_density_err), sizeof(RealCuda));
	}

	DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, avg_density_err);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
	static void     *d_temp_storage = NULL;
	static size_t   temp_storage_bytes = 0;

	if (d_temp_storage == NULL) {
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.densityAdv, avg_density_err, data.numFluidParticles);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
	}
	// Run sum-reduction
	hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.densityAdv, avg_density_err, data.numFluidParticles);

	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));

	return result;
}

__global__ void DFSPH_viscosityXSPH_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	const Vector3d &xi = m_data.posFluid[i];
	const Vector3d &vi = m_data.velFluid[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.getNumberOfNeighbourgs(i);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;

		// Viscosity
		ai -= m_data.invH * m_data.viscosity * (m_data.mass[neighborIndex] / m_data.density[neighborIndex]) *
			(vi - m_data.velFluid[neighborIndex]) * m_data.W(xi - m_data.posFluid[neighborIndex]);

	}

	m_data.accFluid[i] = m_data.gravitation + ai;
}
void cuda_viscosityXSPH(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_viscosityXSPH_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_viscosityXSPH failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_CFL_kernel(SPH::DFSPHCData m_data, RealCuda* maxVel) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	for (unsigned int i = 0; i < m_data.numFluidParticles; i++)
	{
		const RealCuda velMag = (m_data.velFluid[i] + m_data.accFluid[i] * m_data.h).squaredNorm();
		if (velMag > *maxVel)
			*maxVel = velMag;
	}
}

__global__ void DFSPH_CFLVelSquaredNorm_kernel(SPH::DFSPHCData m_data, RealCuda* sqaredNorm) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	sqaredNorm[i] = (m_data.velFluid[i] + m_data.accFluid[i] * m_data.h).squaredNorm();
}

__global__ void DFSPH_CFLAdvanced_kernel(SPH::DFSPHCData m_data, RealCuda *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ RealCuda cache[256];


	RealCuda temp = 0;
	while (index + offset < n) {
		int i = index + offset;
		const RealCuda velMag = (m_data.velFluid[i] + m_data.accFluid[i] * m_data.h).squaredNorm();
		temp = fmaxf(temp, velMag);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = MAX_MACRO_CUDA(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) != 0);  //lock
		*max = MAX_MACRO_CUDA(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}
void cuda_CFL(SPH::DFSPHCData& m_data, const RealCuda minTimeStepSize, RealCuda m_cflFactor, RealCuda m_cflMaxTimeStepSize) {

	//we compute the square norm

	std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();

	RealCuda* out_buff;
	hipMalloc(&(out_buff), sizeof(RealCuda));

	if (true) {

		//cub version
		static RealCuda* temp_buff = NULL;
		if (temp_buff == NULL) {
			hipMallocManaged(&(temp_buff), m_data.numFluidParticles * sizeof(RealCuda));
		}
		int numBlocks = (m_data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_CFLVelSquaredNorm_kernel << <numBlocks, BLOCKSIZE >> > (m_data, temp_buff);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl squared norm failed: %d\n", (int)cudaStatus);
			exit(1598);
		}

		// Determine temporary device storage requirements
		static void     *d_temp_storage = NULL;
		static size_t   temp_storage_bytes = 0;
		if (d_temp_storage == NULL) {
			hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.numFluidParticles);
			// Allocate temporary storage
			hipMalloc(&d_temp_storage, temp_storage_bytes);
		}
		// Run max-reduction
		hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.numFluidParticles);

	}
	else {
		//manual
		int *d_mutex;
		hipMalloc((void**)&d_mutex, sizeof(int));
		hipMemset(d_mutex, 0, sizeof(float));

		int numBlocks = (m_data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_CFLAdvanced_kernel << < numBlocks, BLOCKSIZE >> > (m_data, out_buff, d_mutex, m_data.numFluidParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl failed: %d\n", (int)cudaStatus);
			exit(1598);
		}
		hipFree(d_mutex);
	}

	RealCuda maxVel;
	hipMemcpy(&maxVel, out_buff, sizeof(RealCuda), hipMemcpyDeviceToHost);
	hipFree(out_buff);

	std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

	RealCuda h = m_data.h;

	// Approximate max. time step size 		
	h = m_cflFactor * .4 * (2.0*m_data.particleRadius / (sqrt(maxVel)));

	h = min(h, m_cflMaxTimeStepSize);
	h = max(h, minTimeStepSize);

	m_data.updateTimeStep(h);//*/


	std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();



	float time_search = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
	float time_comp = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;

	printf("Time to do cfl (search,comp): %f    %f\n", time_search, time_comp);
}

__global__ void DFSPH_update_vel_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.velFluid[i] += m_data.h * m_data.accFluid[i];

#ifdef USE_WARMSTART	
	//done here to have one less kernel
	m_data.kappa[i] = MAX_MACRO_CUDA(m_data.kappa[i] * m_data.h_ratio_to_past2, -0.5);
#endif
}
void cuda_update_vel(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_update_vel_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_vel failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

template<bool warmstart> __global__ void DFSPH_pressure_compute_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	pressureSolveParticle<warmstart>(m_data, i);

}
template<bool warmstart> void cuda_pressure_compute(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_pressure_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_pressure_compute<true>(SPH::DFSPHCData& data);
template void cuda_pressure_compute<false>(SPH::DFSPHCData& data);

__global__ void DFSPH_pressure_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityAdv(m_data, i);

	m_data.factor[i] *= m_data.invH_future;
#ifdef USE_WARMSTART
	m_data.kappa[i] = 0;
#endif

}
void cuda_pressure_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_pressure_loop_end_kernel(SPH::DFSPHCData m_data, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityAdv(m_data, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}
//*
__global__ void DFSPH_pressure_loop_end_kernel(int numFluidParticles, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
	RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
	SPH::RigidBodyContainer* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }

	computeDensityAdv(i, posFluid, velFluid, neighbourgs, numberOfNeighbourgs,
		mass, m_kernel_precomp, boundaryPsi, posBoundary, velBoundary,
		vector_dynamic_bodies_data_cuda,  densityAdv, density, h_future, density0);
}//*/

RealCuda cuda_pressure_loop_end(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;

	std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
	static RealCuda* avg_density_err = NULL;
	if (avg_density_err == NULL) {
		hipMalloc(&(avg_density_err), sizeof(RealCuda));
	}

	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, avg_density_err);
	
	/*
	///LOL the detailed implementation is slower so no need to even think about developping data
	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles, data.posFluid, data.velFluid, 
		data.neighbourgs, data.numberOfNeighbourgs,
		data.mass, data.m_kernel_precomp, data.boundaryPsi, data.posBoundary, data.velBoundary,
		data.vector_dynamic_bodies_data_cuda, data.densityAdv, data.density, data.h_future, data.density0);
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
	static void     *d_temp_storage = NULL;
	static size_t   temp_storage_bytes = 0;

	if (d_temp_storage == NULL) {
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.densityAdv, avg_density_err, data.numFluidParticles);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
	}
	// Run sum-reduction
	hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.densityAdv, avg_density_err, data.numFluidParticles);


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));
	

	std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
	float time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
	float time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;

	//std::cout << "pressure loop end details: " << time1 << "  " << time2 << std::endl;

	return result;
}

__global__ void DFSPH_update_pos_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.posFluid[i] += m_data.h * m_data.velFluid[i];
}
void cuda_update_pos(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_update_pos_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_pos failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


int cuda_divergenceSolve(SPH::DFSPHCData& m_data, const unsigned int maxIter, const RealCuda maxError) {
	//////////////////////////////////////////////////////////////////////////
	// Init parameters
	//////////////////////////////////////////////////////////////////////////
	
	const RealCuda h = m_data.h;
	const int numParticles = m_data.numFluidParticles;
	const RealCuda density0 = m_data.density0;

#ifdef USE_WARMSTART_V
	cuda_divergence_warmstart_init(m_data);
	cuda_divergence_compute<true>(m_data);
#endif


	//////////////////////////////////////////////////////////////////////////
	// Compute velocity of density change
	//////////////////////////////////////////////////////////////////////////
	cuda_divergence_init(m_data);


	unsigned int m_iterationsV = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	// use maximal density error divided by time step size
	const RealCuda eta = maxError * 0.01 * density0 / h;  // maxError is given in percent

	RealCuda avg_density_err = 0.0;
	while (((avg_density_err > eta) || (m_iterationsV < 1)) && (m_iterationsV < maxIter))
	{

		//////////////////////////////////////////////////////////////////////////
		// Perform Jacobi iteration over all blocks
		//////////////////////////////////////////////////////////////////////////	
		cuda_divergence_compute<false>(m_data);

		avg_density_err = cuda_divergence_loop_end(m_data);

		avg_density_err /= numParticles;
		m_iterationsV++;
	}

	return m_iterationsV;
}
int cuda_pressureSolve(SPH::DFSPHCData& m_data, const unsigned int m_maxIterations, const RealCuda m_maxError) {
	const RealCuda density0 = m_data.density0;
	const int numParticles = (int)m_data.numFluidParticles;
	RealCuda avg_density_err = 0.0;


	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


#ifdef USE_WARMSTART		
	cuda_pressure_compute<true>(m_data);
#endif


	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();

	//////////////////////////////////////////////////////////////////////////
	// Compute rho_adv
	//////////////////////////////////////////////////////////////////////////
	cuda_pressure_init(m_data);


	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();


	unsigned int m_iterations = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	const RealCuda eta = m_maxError * 0.01 * density0;  // maxError is given in percent

	float time_3_1 = 0;
	float time_3_2 = 0;
	while (((avg_density_err > eta) || (m_iterations < 2)) && (m_iterations < m_maxIterations))
	{
		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
		cuda_pressure_compute<false>(m_data);
		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
		avg_density_err = cuda_pressure_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
		avg_density_err /= numParticles;

		m_iterations++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
	}
	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - start).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;
	
	std::cout << "detail pressure solve (iter total (warm init actual_comp (t1 t2))): " <<m_iterations <<"  "<< time_1 + time_2 +time_3 << 
		"  (" << time_1 << "  " << time_2<< "  "<< time_3 <<"("<< time_3_1<<" "<< time_3_2<<") )" << std::endl;

	//*/

	return m_iterations;

}


template<unsigned int grid_size, bool z_curve>
__global__ void DFSPH_computeGridIdx_kernel(Vector3d* in, unsigned int* out, RealCuda kernel_radius, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	if (z_curve) {

	}
	else {
		//the +50 is an offset so that I don't use the border of the grid
		//it allosw me to be sure that I won't have particles outside of the grid
		//the main thing is that their domain has negative position values
		//that +10 prevent having any negative index by positioning the bounding area of the particles 
		//incide the area  described by our cells
		Vector3d pos = (in[i] / kernel_radius) + 50;
		out[i] = (int)pos.x + ((int)pos.y)*grid_size + ((int)pos.z)*grid_size*grid_size;
	}
}

__global__ void DFSPH_Histogram_kernel(unsigned int* in, unsigned int* out, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	atomicAdd(&(out[in[i]]), 1);

}

__global__ void DFSPH_setBufferValueToItself_kernel(unsigned int* buff, unsigned int buff_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= buff_size) { return; }

	buff[i] = i;
}

__global__ void DFSPH_setVector3dBufferToZero_kernel(Vector3d* buff, unsigned int buff_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= buff_size) { return; }

	buff[i] = Vector3d(0,0,0);
}

__global__ void DFSPH_neighborsSearch_kernel(unsigned int numFluidParticles, RealCuda radius,
	Vector3d* posFluid, Vector3d* posBoundary, int* neighbors_buff, int* nb_neighbors_buff,
	unsigned int* p_id_sorted, unsigned int* cell_start_end, unsigned int* p_id_sorted_b, unsigned int* cell_start_end_b,
	SPH::RigidBodyContainer::NeighborKernelData* vect_dynamic_bodies, int nb_dynamic_bodies) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }

	RealCuda radius_sq = radius;
	Vector3d pos = posFluid[i];
	Vector3d pos_cell = (pos / radius_sq) + 50; //on that line the radius is not yet squared
	int x = (int)pos_cell.x;
	int y = (int)pos_cell.y;
	int z = (int)pos_cell.z;
	radius_sq *= radius_sq;

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr = neighbors_buff + i*MAX_NEIGHBOURS;
	//int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
	//int neighbors_boundary[MAX_NEIGHBOURS];
	//now we iterate on the 9 cell block surronding the cell in which we have our particle
	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			//for (int l = -1; l < 2; ++l) {// I don't need to iter on x since the 3cells are successives: large gains
				//we iterate on the particles inside that cell
			unsigned int cur_cell_id = (x + -1) + (y + m)*CELL_ROW_LENGTH + (z + k)*CELL_ROW_LENGTH*CELL_ROW_LENGTH;
			unsigned int end;
			//*
			//for the fluid particles
			end = cell_start_end[cur_cell_id + 3];
			for (unsigned int cur_particle = cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = p_id_sorted[cur_particle];
				if (i != j) {
					if ((pos - posFluid[j]).squaredNorm() < radius_sq) {
						*cur_neighbor_ptr++ = j;
						//neighbors_buff[i*MAX_NEIGHBOURS * 2 + nb_neighbors_fluid] = j;
						//neighbors_fluid[nb_neighbors_fluid] = j;
						nb_neighbors_fluid++;
					}
				}
			}
		}
	}

	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			//for (int l = -1; l < 2; ++l) {// I don't need to iter on x since the 3cells are successives: large gains
			//we iterate on the particles inside that cell
			unsigned int cur_cell_id = (x + -1) + (y + m)*CELL_ROW_LENGTH + (z + k)*CELL_ROW_LENGTH*CELL_ROW_LENGTH;
			unsigned int end;
			//for the boundaries particles
			end = cell_start_end_b[cur_cell_id + 3];
			for (unsigned int cur_particle = cell_start_end_b[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = p_id_sorted_b[cur_particle];
				if ((pos - posBoundary[j]).squaredNorm() < radius_sq) {
					*cur_neighbor_ptr++ = j;
					//neighbors_buff[i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS + nb_neighbors_boundary] = j;
					//neighbors_boundary[nb_neighbors_boundary] = j;
					nb_neighbors_boundary++;
				}
			}
			//*/
		//}
		}
	}
	if (vect_dynamic_bodies != NULL) {
		for (int id_body = 0; id_body < nb_dynamic_bodies; ++id_body) {
			const SPH::RigidBodyContainer::NeighborKernelData& body = vect_dynamic_bodies[id_body];
			for (int k = -1; k < 2; ++k) {
				for (int m = -1; m < 2; ++m) {
					//for (int l = -1; l < 2; ++l) {// I don't need to iter on x since the 3cells are successives: large gains
					//we iterate on the particles inside that cell
					unsigned int cur_cell_id = (x + -1) + (y + m)*CELL_ROW_LENGTH + (z + k)*CELL_ROW_LENGTH*CELL_ROW_LENGTH;
					unsigned int end;
					//for the boundaries particles
					end = body.cell_start_end[cur_cell_id + 3];
					for (unsigned int cur_particle = body.cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
						unsigned int j = body.p_id_sorted[cur_particle];
						if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
							*cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j);
							nb_neighbors_dynamic_objects++;
						}
					}
					//*/
					//}
				}
			}
		}
	}


	nb_neighbors_buff[3*i] = nb_neighbors_fluid;
	nb_neighbors_buff[3 * i + 1] = nb_neighbors_boundary; 
	nb_neighbors_buff[3 * i + 2] = nb_neighbors_dynamic_objects;

	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS*2), neighbors_fluid, sizeof(int)*nb_neighbors_fluid);
	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS), neighbors_boundary, sizeof(int)*nb_neighbors_boundary);


}

void cuda_neighborsSearchInternal_sortParticlesId(Vector3d* pos, RealCuda kernel_radius, int numParticles, void **d_temp_storage_pair_sort,
	size_t   &temp_storage_bytes_pair_sort, unsigned int* cell_id, unsigned int* cell_id_sorted,
	unsigned int* p_id, unsigned int* p_id_sorted) {
	hipError_t cudaStatus;

	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel<CELL_ROW_LENGTH, false> << <numBlocks, BLOCKSIZE >> > (pos, cell_id,
		kernel_radius, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//do the actual sort
	//first Determine temporary device storage requirements
	if ((*d_temp_storage_pair_sort) == NULL) {
		temp_storage_bytes_pair_sort = 0;
		hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
			cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
		// Allocate temporary storage
		hipMalloc(d_temp_storage_pair_sort, temp_storage_bytes_pair_sort);

	}

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
		cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
	//*/


	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sort failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

}

void cuda_neighborsSearchInternal_computeCellStartEnd(int numParticles, unsigned int* cell_id_sorted,
	unsigned int* hist, void **d_temp_storage_cumul_hist, size_t   &temp_storage_bytes_cumul_hist, unsigned int* cell_start_end) {
	hipError_t cudaStatus;
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "histogram value reset failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//compute the actual histogram (done here with atomic adds)
	DFSPH_Histogram_kernel << <numBlocks, BLOCKSIZE >> > (cell_id_sorted, hist, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "histogram failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value

	if ((*d_temp_storage_cumul_hist) == NULL) {
		temp_storage_bytes_cumul_hist = 0;
		//get the necessary size
		hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));
		// Allocate temporary storage
		hipMalloc(d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist);
	}
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cumulative histogram failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}



//this is the bases for all kernels based function
template<typename T>
__global__ void DFSPH_sortFromIndex_kernel(T* in, T* out, unsigned int* index, unsigned int nbElements) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nbElements) { return; }

	out[i] = in[index[i]];
}

#include <sstream>
void cuda_sortData(SPH::DFSPHCData& data, SPH::NeighborsSearchDataSet& neighborsDataSet, bool is_boundaries) {
	//*
	unsigned int numParticles = neighborsDataSet.numParticles;
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	unsigned int *p_id_sorted = neighborsDataSet.p_id_sorted;


	//hipError_t cudaStatus;

	Vector3d* pos = NULL;
	Vector3d* vel = NULL;

	if (is_boundaries) {
		pos = data.posBoundary;
		vel = data.velBoundary;

		//we need to sort the psi for the boundaries
		RealCuda* intermediate_buffer = NULL;
		hipMalloc(&(intermediate_buffer), numParticles * sizeof(RealCuda));
		
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (data.boundaryPsi, intermediate_buffer, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(data.boundaryPsi, intermediate_buffer, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		hipFree(intermediate_buffer); intermediate_buffer = NULL;
	}
	else {
		pos = data.posFluid;
		vel = data.velFluid;

		//when handling the fluid I also need to sort the intermediate buffers
		RealCuda* intermediate_buffer = NULL;
		hipMalloc(&(intermediate_buffer), numParticles * sizeof(RealCuda));

		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (data.kappa, intermediate_buffer, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(data.kappa, intermediate_buffer, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (data.kappaV, intermediate_buffer, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(data.kappaV, intermediate_buffer, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		hipFree(intermediate_buffer); intermediate_buffer = NULL;
	}
	//*
	Vector3d* intermediate_buffer = NULL;
	hipMallocManaged(&(intermediate_buffer), numParticles * sizeof(Vector3d));



	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (pos, intermediate_buffer, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(pos, intermediate_buffer, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (vel, intermediate_buffer, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(vel, intermediate_buffer, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	hipFree(intermediate_buffer); intermediate_buffer = NULL;



	//now that everything is sorted we can set each particle index to itself
	gpuErrchk(hipMemcpy(p_id_sorted, neighborsDataSet.p_id, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice));
	//*/

	/*{
		std::ostringstream oss;
		for (int i = 50000; i < numParticles; ++i) {
			oss << p_id_sorted[i] << " ";
		}
		fprintf(stderr, "%s\n", oss.str().c_str());
	}//*/
	fprintf(stderr, "actually worked Oo\n");
}


void cuda_neighborsSearch(SPH::DFSPHCData& data) {

	std::chrono::steady_clock::time_point begin_global = std::chrono::steady_clock::now();
	static unsigned int time_count =  0 ;
	float time_global;
	static float time_avg_global = 0;
	time_count++;

	hipError_t cudaStatus;
	{

		float time;
		static float time_avg =  0 ;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		

		//first let's generate the cell start end for the dynamic bodies
		{
			for (int i = 0; i < data.numDynamicBodies; ++i) {
				SPH::RigidBodyContainer& body = data.vector_dynamic_bodies_data[i];

				SPH::NeighborsSearchDataSet* dataSet = body.neighborsDataSet;
				cuda_neighborsSearchInternal_sortParticlesId(body.pos, data.m_kernel_precomp.getRadius(), dataSet->numParticles,
					&(dataSet->d_temp_storage_pair_sort), dataSet->temp_storage_bytes_pair_sort, dataSet->cell_id, dataSet->cell_id_sorted,
					dataSet->p_id, dataSet->p_id_sorted);


				cuda_neighborsSearchInternal_computeCellStartEnd(dataSet->numParticles, dataSet->cell_id_sorted, dataSet->hist,
					&(dataSet->d_temp_storage_cumul_hist), dataSet->temp_storage_bytes_cumul_hist, dataSet->cell_start_end);
			}
		}

		//now update the cell start end of the fluid particles
		{

			


			//update the positions of the particles in the grid
			SPH::NeighborsSearchDataSet* dataSet = data.neighborsdataSetFluid;
			cuda_neighborsSearchInternal_sortParticlesId(data.posFluid, data.m_kernel_precomp.getRadius(), dataSet->numParticles,
				&(dataSet->d_temp_storage_pair_sort), dataSet->temp_storage_bytes_pair_sort, dataSet->cell_id, dataSet->cell_id_sorted,
				dataSet->p_id, dataSet->p_id_sorted);

			//since it the init iter I'll sort both even if it's the boundaries
			static int step_count = 0;
			step_count++;
			if (step_count > 25) {
				cuda_sortData(data, *dataSet, false);
				step_count = 0;
			}


			cuda_neighborsSearchInternal_computeCellStartEnd(dataSet->numParticles, dataSet->cell_id_sorted, dataSet->hist,
				&(dataSet->d_temp_storage_cumul_hist), dataSet->temp_storage_bytes_cumul_hist, dataSet->cell_start_end);


			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "before neighbors search: %d\n", (int)cudaStatus);
				exit(1598);
			}

			
		}

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		//printf("Time to generate cell start end: %f ms   avg: %f ms \n", time, time_avg / time_count);
	}
	//and we can now do the actual search of the neaighbor for eahc fluid particle
	if (true) 
	{
		float time;
		static float time_avg = 0;

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		SPH::RigidBodyContainer::NeighborKernelData* vect_dynamic_bodies=NULL;
		if (data.numDynamicBodies > 0) {
			hipMallocManaged(&(vect_dynamic_bodies), data.numDynamicBodies * sizeof(SPH::RigidBodyContainer::NeighborKernelData));

			for (int i = 0; i < data.numDynamicBodies; ++i) {
				vect_dynamic_bodies[i] = data.vector_dynamic_bodies_data[i].getNeighborKerneldata();
			}
		}

		//cuda way
		int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_neighborsSearch_kernel<< <numBlocks, BLOCKSIZE >> > (data.numFluidParticles,
			data.m_kernel_precomp.getRadius(), data.posFluid, data.posBoundary,
			data.neighbourgs, data.numberOfNeighbourgs,
			data.neighborsdataSetFluid->p_id_sorted,
			data.neighborsdataSetFluid->cell_start_end,
			data.neighborsdataSetBoundaries->p_id_sorted,
			data.neighborsdataSetBoundaries->cell_start_end,
			vect_dynamic_bodies, data.numDynamicBodies);


		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda neighbors search failed: %d\n", (int)cudaStatus);
			exit(1598);
		}

		hipFree(vect_dynamic_bodies);

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		//printf("Time to generate neighbors buffers: %f ms   avg: %f ms \n", time, time_avg / time_count);

		/*
		//a simple check to know the max nbr of neighbors		
		static int absolute_max = 0;
		int max = 0;

		static int absolute_max_d[3] = { 0 };
		int max_d[3] = { 0 };



		for (int j = 0; j < data.numFluidParticles; j++)
		{
			//check the global value
			int count_neighbors = 0;
			for (int k = 0; k < 2; ++k) {
				count_neighbors += data.getNumberOfNeighbourgs(j, k);
			}
			if (count_neighbors > max)max = count_neighbors;
			
			//chekc the max for each category
			for (unsigned int k = 0; k < 3; ++k) {
				if ((int)data.getNumberOfNeighbourgs(j,k) > max_d[k])max_d[k] = data.getNumberOfNeighbourgs(j,k);
			}
			
		}
		if (max>absolute_max)absolute_max = max;
		for (unsigned int k = 0; k < 3; ++k) {
			if (max_d[k]>absolute_max_d[k])absolute_max_d[k] = max_d[k];
		}
		printf("max nbr of neighbors %d  (%d) \n", absolute_max, max); 
		printf("max nbr of neighbors %d  (%d)      absolute max  fluid // boundaries // bodies   %d // %d // %d\n",
			absolute_max, max, absolute_max_d[0], absolute_max_d[1], absolute_max_d[2]);
		//*/
	}

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	time_global = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin_global).count() / 1000000.0f;

	time_avg_global += time_global;
	//printf("time taken by the neighbor function: %f ms   avg: %f ms \n", time_global, time_avg_global / time_count);
}


void cuda_initNeighborsSearchDataSet(SPH::DFSPHCData& data, SPH::NeighborsSearchDataSet& dataSet, bool is_boundaries) {
	Vector3d* pos = NULL;

	if (is_boundaries) {
		pos = data.posBoundary;
	}
	else {
		pos = data.posFluid;

	}

	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(pos, data.m_kernel_precomp.getRadius(), dataSet.numParticles,
		&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
		dataSet.p_id, dataSet.p_id_sorted);

	//since it the init iter I'll sort both even if it's the boundaries
	cuda_sortData(data, dataSet, is_boundaries);


	//and now I cna compute the start and end of each cell :)
	cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
		&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);

	fprintf(stderr, "finished init neighbors \n");

	/*
	if (is_boundaries) {
		std::ostringstream oss;

		Vector3d pos_cell = (pos[dataSet.p_id_sorted[0]] / data.m_kernel_precomp.getRadius()) + 50; //on that line the radius is not yet squared
		int x = (int)pos_cell.x;
		int y = (int)pos_cell.y;
		int z = (int)pos_cell.z;
		unsigned int cur_cell_id = (x)+(y)*CELL_ROW_LENGTH + (z)*CELL_ROW_LENGTH*CELL_ROW_LENGTH;

		unsigned int start = dataSet.cell_start_end[cur_cell_id];
		unsigned int end = dataSet.cell_start_end[cur_cell_id + 1];
		oss <<"size" <<end - start<<std::endl;


		for (unsigned int cur_particle = start; cur_particle < end; ++cur_particle) {
			unsigned int j = dataSet.p_id_sorted[cur_particle];
			oss << j<<": (" << pos[j].x << " " << pos[j].y << " " << pos[j].z << ")      ";
		}


		fprintf(stderr, "cell size2:  %s \n", oss.str().c_str());
	}
	//*/
}


void cuda_renderFluid(SPH::DFSPHCData& data) {
	cuda_opengl_renderFluid(data);
}



void cuda_renderBoundaries(SPH::DFSPHCData& data) {
	cuda_opengl_renderBoundaries(data);
}


#include <GL/glew.h>
#include <cuda_gl_interop.h>

void cuda_opengl_initFluidRendering(SPH::DFSPHCData& data) {
	glGenVertexArrays(1, &data.vao); // Cr�er le VAO
	glBindVertexArray(data.vao); // Lier le VAO pour l'utiliser


	glGenBuffers(1, &data.pos_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, data.pos_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	data.numFluidParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FORMAT, GL_FALSE, 0, 0);

	glGenBuffers(1, &data.vel_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, data.vel_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	data.numFluidParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 3, GL_FORMAT, GL_FALSE, 0, 0);

	// nettoyage
	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// Registration with CUDA.
	gpuErrchk(hipGraphicsGLRegisterBuffer(&vboRes_pos, data.pos_buffer, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&vboRes_vel, data.vel_buffer, hipGraphicsRegisterFlagsNone));

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &vboRes_pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &vboRes_vel, 0));

	//set the openglbuffer for direct use in cuda
	Vector3d* vboPtr = NULL;
	size_t size = 0;

	// pos
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, vboRes_pos));//get cuda ptr
	data.posFluid = vboPtr;

	// vel
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, vboRes_vel));//get cuda ptr
	data.velFluid = vboPtr;

}

void cuda_opengl_renderFluid(SPH::DFSPHCData& data) {

	//unlink the pos and vel buffer from cuda
	gpuErrchk(hipGraphicsUnmapResources(1, &vboRes_pos, 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &vboRes_vel, 0));

	//Actual opengl rendering
	// link the vao
	glBindVertexArray(data.vao);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//show it
	glDrawArrays(GL_POINTS, 0, data.numFluidParticles);

	// unlink the vao
	glBindVertexArray(0);

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &vboRes_pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &vboRes_vel, 0));

}


void cuda_opengl_renderBoundaries(SPH::DFSPHCData& data) {

	static Vector3d* buff = NULL;
	static bool first_time = true;

	if (first_time) {
		first_time = false;
		buff = new Vector3d[data.numBoundaryParticles];
	}


	gpuErrchk(hipMemcpy(buff, data.posBoundary, data.numBoundaryParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));

	glEnableVertexAttribArray(0);

	glVertexAttribPointer(0, 3, GL_FORMAT, GL_FALSE, 0, data.posBoundary);
	glDrawArrays(GL_POINTS, 0, data.numBoundaryParticles);

	glDisableVertexAttribArray(0);

}










/*
	THE NEXT FUNCTIONS ARE FOR THE MEMORY ALLOCATION
*/

void allocate_rigid_body_container_cuda(SPH::RigidBodyContainer& container) {
	fprintf(stderr, "start of reset values gpu: \n");
	
	hipMallocManaged(&(container.pos), container.numParticles * sizeof(Vector3d));
	hipMalloc(&(container.vel), container.numParticles * sizeof(Vector3d));
	hipMalloc(&(container.psi), container.numParticles * sizeof(RealCuda));
	hipMalloc(&(container.F), container.numParticles * sizeof(Vector3d));

	/*
	int identifier = 356 + (15 << 0x10);
	int neighborIndex = identifier & 0xFFFF;
	int bodyIndex = (identifier & ~0xFFFF) >> 0x10;
	fprintf(stderr, "test computations: %d // %d // %d \n", identifier, neighborIndex, bodyIndex);
	//*/
}


void load_rigid_body_container_cuda(SPH::RigidBodyContainer& container, Vector3d* pos, Vector3d* vel, RealCuda* psi) {

	gpuErrchk(hipMemcpy(container.pos, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.vel, vel, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.psi, psi, container.numParticles * sizeof(RealCuda), hipMemcpyHostToDevice));

	int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);

}

void read_rigid_body_force_cuda(SPH::RigidBodyContainer& container) {
	gpuErrchk(hipMemcpy(container.F_cpu, container.F, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
}


void allocate_dynamic_bodies_vector_cuda(SPH::DFSPHCData& data) {
	
	gpuErrchk(hipMalloc(&(data.vector_dynamic_bodies_data_cuda), data.numDynamicBodies * sizeof(SPH::RigidBodyContainer)));

	gpuErrchk(hipMemcpy(data.vector_dynamic_bodies_data_cuda, data.vector_dynamic_bodies_data, 
		data.numDynamicBodies * sizeof(SPH::RigidBodyContainer), hipMemcpyHostToDevice));
}


void allocate_c_array_struct_cuda_managed(SPH::DFSPHCData& data, bool minimize_managed) {

	hipMallocManaged(&(data.posBoundary), data.numBoundaryParticles * sizeof(Vector3d));
	hipMalloc(&(data.velBoundary), data.numBoundaryParticles * sizeof(Vector3d));
	hipMalloc(&(data.boundaryPsi), data.numBoundaryParticles * sizeof(RealCuda));


	//handle the fluid
	hipMalloc(&(data.mass), data.numFluidParticles * sizeof(RealCuda));
	//hipMalloc(&(data.posFluid), data.numFluidParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
	//hipMalloc(&(data.velFluid), data.numFluidParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
	hipMalloc(&(data.accFluid), data.numFluidParticles * sizeof(Vector3d));
	hipMallocManaged(&(data.numberOfNeighbourgs), data.numFluidParticles * 3 * sizeof(int));
	hipMalloc(&(data.neighbourgs), data.numFluidParticles * MAX_NEIGHBOURS * sizeof(int));

	hipMalloc(&(data.density), data.numFluidParticles * sizeof(RealCuda));
	hipMalloc(&(data.factor), data.numFluidParticles * sizeof(RealCuda));
	hipMalloc(&(data.kappa), data.numFluidParticles * sizeof(RealCuda));
	hipMalloc(&(data.kappaV), data.numFluidParticles * sizeof(RealCuda));
	hipMalloc(&(data.densityAdv), data.numFluidParticles * sizeof(RealCuda));

}

void reset_c_array_struct_cuda_from_values(SPH::DFSPHCData& data, Vector3d* posBoundary, Vector3d* velBoundary,
	RealCuda* boundaryPsi, Vector3d* posFluid, Vector3d* velFluid, RealCuda* mass) {

	fprintf(stderr, "start of reset values gpu: \n");

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	//boundaries
	gpuErrchk(hipMemcpy(data.posBoundary, posBoundary, data.numBoundaryParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(data.velBoundary, velBoundary, data.numBoundaryParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(data.boundaryPsi, boundaryPsi, data.numBoundaryParticles * sizeof(RealCuda), hipMemcpyHostToDevice));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init of boundaries particles from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//fluid
	gpuErrchk(hipMemcpy(data.posFluid, posFluid, data.numFluidParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(data.velFluid, velFluid, data.numFluidParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(data.mass, mass, data.numFluidParticles * sizeof(RealCuda), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(data.accFluid, 0, data.numFluidParticles * sizeof(Vector3d)));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init of fluid particles from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//ohter values normaly only kappa and kappaV are necessary (ut this function is only called 
	//when reseting the fluid so it does not cost much to make sure everything is clean
	gpuErrchk(hipMemset(data.density, 0, data.numFluidParticles * sizeof(RealCuda)));
	gpuErrchk(hipMemset(data.factor, 0, data.numFluidParticles * sizeof(RealCuda)));
	gpuErrchk(hipMemset(data.kappa, 0, data.numFluidParticles * sizeof(RealCuda)));
	gpuErrchk(hipMemset(data.kappaV, 0, data.numFluidParticles * sizeof(RealCuda)));
	gpuErrchk(hipMemset(data.densityAdv, 0, data.numFluidParticles * sizeof(RealCuda)));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init of fluid other values from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}


	fprintf(stderr, "end of reset gpu\n");
}


void allocate_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel, bool minimize_managed) {

	if (minimize_managed) {
		hipMalloc(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		hipMalloc(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
	else {
		fprintf(stderr, "trying to use managed buffers for the kernels\n");
		exit(1256);
		//hipMallocManaged(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		//hipMallocManaged(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
}


void init_precomputed_kernel_from_values(SPH::PrecomputedCubicKernelPerso& kernel, RealCuda* w, RealCuda* grad_W) {
	hipError_t cudaStatus;
	//W
	cudaStatus = hipMemcpy(kernel.m_W,
		w,
		kernel.m_resolution * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//grad W
	cudaStatus = hipMemcpy(kernel.m_gradW,
		grad_W,
		(kernel.m_resolution + 1) * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of grad W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

}


void allocate_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet) {

	//allocatethe mme for fluid particles
	hipMallocManaged(&(dataSet.cell_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.cell_id_sorted), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.local_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.p_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.p_id_sorted), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.cell_start_end), (CELL_COUNT + 1) * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.hist), (CELL_COUNT + 1) * sizeof(unsigned int));

	//init variables for cub calls
	dataSet.d_temp_storage_pair_sort = NULL;
	dataSet.temp_storage_bytes_pair_sort = 0;
	dataSet.d_temp_storage_cumul_hist = NULL;
	dataSet.temp_storage_bytes_cumul_hist = 0;

	//reset the particle id
	int numBlocks = (dataSet.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id, dataSet.numParticles);
	DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id_sorted, dataSet.numParticles);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "p_id init idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	dataSet.internal_buffers_allocated = true;
}


void release_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet, bool keep_result_buffers) {
	//allocatethe mme for fluid particles
	hipFree(dataSet.cell_id); dataSet.cell_id = NULL;
	hipFree(dataSet.local_id); dataSet.local_id = NULL;
	hipFree(dataSet.p_id); dataSet.p_id = NULL;
	hipFree(dataSet.cell_id_sorted); dataSet.cell_id_sorted = NULL;
	hipFree(dataSet.hist); dataSet.hist = NULL;

	//init variables for cub calls
	hipFree(dataSet.d_temp_storage_pair_sort);
	dataSet.d_temp_storage_pair_sort = NULL;
	dataSet.temp_storage_bytes_pair_sort = 0;
	hipFree(dataSet.d_temp_storage_cumul_hist);
	dataSet.d_temp_storage_cumul_hist = NULL;
	dataSet.temp_storage_bytes_cumul_hist = 0;

	dataSet.internal_buffers_allocated = false;

	if (!keep_result_buffers) {
		hipFree(dataSet.p_id_sorted); dataSet.p_id_sorted = NULL;
		hipFree(dataSet.cell_start_end); dataSet.cell_start_end = NULL;
	}
}























/*
	AFTER THIS ARE ONLY THE TEST FUNCTION TO HAVE CUDA WORKING ...
*/


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//*
__global__ void addKernel(Vector3d* vect)
{
	int i = threadIdx.x;
	vect[i].z = vect[i].x + vect[i].y;
}

__global__ void setVectkernel(Vector3d& vect)
{
	vect.x = 5;
	vect.y = 6;
	vect.z = 7;
}
//*/
int test_cuda()
{
	//DFSPHCData* data;

	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	//*
	Vector3d* vect;
	hipMallocManaged(&vect, arraySize * sizeof(Vector3d));
	for (int i = 0; i < arraySize; ++i) {
		vect[i].x = a[i];
		vect[i].y = b[i];
	}
	//*/*

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	printf("macro val: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	for (int i = 0; i < arraySize; ++i) {
		c[i] = 0;
	}


	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, arraySize >> > (vect);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	printf("with vects {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		(int)(vect[0].z), (int)(vect[1].z), (int)(vect[2].z), (int)(vect[3].z), (int)(vect[4].z));

	hipFree(vect);



	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	/*
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	//*/

	printf("Finished test cuda\n");


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
