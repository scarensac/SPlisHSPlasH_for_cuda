#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_cuda_basic.h"
#include <stdio.h>
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"
#include <chrono>
#include "SPlisHSPlasH/Utilities/Timing.h"
#include <iostream>

#define BLOCKSIZE 256
#define m_eps 1.0e-5
#define CELL_ROW_LENGTH 256
#define CELL_COUNT CELL_ROW_LENGTH*CELL_ROW_LENGTH*CELL_ROW_LENGTH

#define USE_WARMSTART
#define USE_WARMSTART_V

#define BITSHIFT_INDEX_DYNAMIC_BODIES

#ifdef BITSHIFT_INDEX_DYNAMIC_BODIES
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(neighbors_ptr,body_index,particle_index)
#else
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(neighbors_ptr,body_index,particle_index)
#endif

//those defines are to create and read the dynamic bodies indexes
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)  particle_index + (body_index << 0x10)
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)  particle_index + (body_index * 1000000)

//WARNING his one declare the body/particle index by itself
//you just have to give it the variable name you want
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(neighbors_ptr, body_index,particle_index)  \
const unsigned int identifier = *neighbors_ptr++;\
const unsigned int particle_index = identifier & 0xFFFF;\
const unsigned int body_index = (identifier & ~0xFFFF) >> 0x10;

#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(neighbors_ptr, body_index,particle_index)   \
const unsigned int identifier = *neighbors_ptr++;\
const unsigned int particle_index = identifier % (1000000);\
const unsigned int body_index=identifier / 1000000;


//using norton bitshift for the cells is slower than using a normal index, not that much though
//#define BITSHIFT_INDEX_NEIGHBORS_CELL


#ifdef BITSHIFT_INDEX_NEIGHBORS_CELL

#define USE_COMPLETE

__device__ void interleave_2_bits_magic_numbers(unsigned int& x) {
	x = (x | (x << 16)) & 0x030000FF;
	x = (x | (x << 8)) & 0x0300F00F;
	x = (x | (x << 4)) & 0x030C30C3;
	x = (x | (x << 2)) & 0x09249249;
}
__device__ unsigned int compute_morton_magic_numbers(unsigned int x, unsigned int y, unsigned int z) {
	interleave_2_bits_magic_numbers(x);
	interleave_2_bits_magic_numbers(y);
	interleave_2_bits_magic_numbers(z);

	return x | (y << 1) | (z << 2);
}

#define COMPUTE_CELL_INDEX(x,y,z) compute_morton_magic_numbers(x,y,z)

#else
#define COMPUTE_CELL_INDEX(x,y,z) (x)+(y)*CELL_ROW_LENGTH+(z)*CELL_ROW_LENGTH*CELL_ROW_LENGTH
#endif



//those two variables are the identifiers that  link the ongle buffers to cuda
//hipGraphicsResource_t vboRes_pos;
//hipGraphicsResource_t vboRes_vel;

//easy function to check errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
//this is the bases for all kernels based function
__global__ void DFSPH__kernel(SPH::DFSPHCData m_data) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= m_data.numFluidParticles) { return; }

}
void cuda_(SPH::DFSPHCData& data) {
int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
DFSPH__kernel << <numBlocks, BLOCKSIZE >> > (data);

hipError_t cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
exit(1598);
}
}
//*/

FUNCTION inline int* getNeighboursPtr(int * neighbourgs, int particle_id) {
	//	return neighbourgs + body_id*numFluidParticles*MAX_NEIGHBOURS + particle_id*MAX_NEIGHBOURS;
	return neighbourgs + particle_id*MAX_NEIGHBOURS;
}

FUNCTION inline unsigned int getNumberOfNeighbourgs(int* numberOfNeighbourgs, int particle_id, int body_id = 0) {
	//return numberOfNeighbourgs[body_id*numFluidParticles + particle_id]; 
	return numberOfNeighbourgs[particle_id * 3 + body_id];
}

__device__ void computeDensityChange(SPH::DFSPHCData& m_data, const unsigned int index) {
	unsigned int numNeighbors = m_data.fluid_data_cuda->getNumberOfNeighbourgs(index);
	// in case of particle deficiency do not perform a divergence solve
	if (numNeighbors < 20) {
		for (unsigned int pid = 1; pid < 3; pid++)
		{
			numNeighbors += m_data.fluid_data_cuda->getNumberOfNeighbourgs(index, pid);
		}
	}
	if (numNeighbors < 20) {
		m_data.fluid_data_cuda->densityAdv[index] = 0;
	}
	else {
		RealCuda densityAdv = 0;
		const Vector3d &xi = m_data.fluid_data_cuda->pos[index];
		const Vector3d &vi = m_data.fluid_data_cuda->vel[index];
		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(index);
		int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(index);
		{
			SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
			while (neighbors_ptr != end_ptr)
			{
				const unsigned int neighborIndex = *neighbors_ptr++;
				densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
			}
		}
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
		{
			SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);
			end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(index, 1);
			while (neighbors_ptr != end_ptr)
			{
				const unsigned int neighborIndex = *neighbors_ptr++;
				densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
			}
		}

		//////////////////////////////////////////////////////////////////////////
		// Dynamic Bodies
		//////////////////////////////////////////////////////////////////////////
		end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(index, 2);
		while (neighbors_ptr != end_ptr)
		{
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
			SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
		}




		// only correct positive divergence
		m_data.fluid_data_cuda->densityAdv[index] = MAX_MACRO_CUDA(densityAdv, 0.0);
	}
}
template <bool warm_start> __device__ void divergenceSolveParticle(SPH::DFSPHCData& m_data, const unsigned int i) {
	Vector3d v_i = Vector3d(0, 0, 0);
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? m_data.fluid_data_cuda->kappaV[i] : (m_data.fluid_data_cuda->densityAdv[i])*m_data.fluid_data_cuda->factor[i];

#ifdef USE_WARMSTART_V
	if (!warm_start) { m_data.fluid_data_cuda->kappaV[i] += ki; }
#endif

	const Vector3d &xi = m_data.fluid_data_cuda->pos[i];


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(i);
	{
		SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
			if (fabs(kSum) > m_eps)
			{
				// ki, kj already contain inverse density
				v_i += kSum *  body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
			}
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 1);
		SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			const Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
			v_i += delta;// ki already contains inverse density
		}

	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 2);
		while (neighbors_ptr != end_ptr)
		{
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
			SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			
			Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
			v_i += delta;// ki already contains inverse density

			//we apply the force to the body particle (no invH since it has been fatorized at the end)
			delta *= -m_data.fluid_data_cuda->mass[i];
			atomicAdd(&(body.F[neighborIndex].x), delta.x);
			atomicAdd(&(body.F[neighborIndex].y), delta.y);
			atomicAdd(&(body.F[neighborIndex].z), delta.z);
		}
	}

	m_data.fluid_data_cuda->vel[i] += v_i*m_data.h;
}

__device__ void computeDensityAdv(SPH::DFSPHCData& m_data, const unsigned int index) {
	const Vector3d xi = m_data.fluid_data_cuda->pos[index];
	const Vector3d vi = m_data.fluid_data_cuda->vel[index];
	RealCuda delta = 0;


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(index);
	int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(index);
	{
		SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(index, 1);
	{
		SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(index, 2);
	while (neighbors_ptr != end_ptr)
	{
		READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
		SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
		delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
	}




	m_data.fluid_data_cuda->densityAdv[index] = MAX_MACRO_CUDA(m_data.fluid_data_cuda->density[index] + m_data.h_future*delta - m_data.density0, 0.0);
}

__device__ void computeDensityAdv(const unsigned int index, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
	RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
	SPH::UnifiedParticleSet* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
	const Vector3d xi = posFluid[index];
	const Vector3d vi = velFluid[index];
	RealCuda delta = 0;

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = getNeighboursPtr(neighbourgs, index);
	int* end_ptr = neighbors_ptr + getNumberOfNeighbourgs(numberOfNeighbourgs, index);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += mass[neighborIndex] * (vi - velFluid[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posFluid[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 1);
	while (neighbors_ptr != end_ptr)
	{
		const unsigned int neighborIndex = *neighbors_ptr++;
		delta += boundaryPsi[neighborIndex] * (vi - velBoundary[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posBoundary[neighborIndex]));
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 2);
	while (neighbors_ptr != end_ptr)
	{
		READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
		SPH::UnifiedParticleSet& body = vector_dynamic_bodies_data_cuda[bodyIndex];
		delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_kernel_precomp.gradW(xi - body.pos[neighborIndex]));
	}




	densityAdv[index] = MAX_MACRO_CUDA(density[index] + h_future*delta - density0, 0.0);
}

template <bool warm_start> __device__ void pressureSolveParticle(SPH::DFSPHCData& m_data, const unsigned int i) {
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? m_data.fluid_data_cuda->kappa[i] : (m_data.fluid_data_cuda->densityAdv[i])*m_data.fluid_data_cuda->factor[i];

#ifdef USE_WARMSTART
	if (!warm_start) { m_data.fluid_data_cuda->kappa[i] += ki; }
#endif


	Vector3d v_i = Vector3d(0, 0, 0);
	const Vector3d &xi = m_data.fluid_data_cuda->pos[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(i);
	{
		SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
			if (fabs(kSum) > m_eps)
			{
				// ki, kj already contain inverse density
				v_i += kSum * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
			}
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 1);
		SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;
			const Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);

			v_i += delta;// ki already contains inverse density
		}
	}

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	if (fabs(ki) > m_eps)
	{
		end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 2);
		while (neighbors_ptr != end_ptr)
		{
			READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
			SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
			Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);

			v_i += delta;// ki already contains inverse density

			//we apply the force to the body particle (no invH since it has been fatorized at the end)
			delta *= -m_data.fluid_data_cuda->mass[i];
			atomicAdd(&(body.F[neighborIndex].x), delta.x);
			atomicAdd(&(body.F[neighborIndex].y), delta.y);
			atomicAdd(&(body.F[neighborIndex].z), delta.z);
		}
	}



	// Directly update velocities instead of storing pressure accelerations
	m_data.fluid_data_cuda->vel[i] += v_i*m_data.h_future;
}

__global__ void DFSPH_divergence_warmstart_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.fluid_data_cuda->kappaV[i] = MAX_MACRO_CUDA(m_data.fluid_data_cuda->kappaV[i] * m_data.h_ratio_to_past / 2, -0.25);
	computeDensityChange(m_data, i);
}
void cuda_divergence_warmstart_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_warmstart_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_warmstart_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

template<bool warmstart> __global__ void DFSPH_divergence_compute_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	if (warmstart) {
		if (m_data.fluid_data_cuda->densityAdv[i] > 0.0) {
			divergenceSolveParticle<warmstart>(m_data, i);
		}
	}
	else {
		divergenceSolveParticle<warmstart>(m_data, i);
	}

}
template<bool warmstart> void cuda_divergence_compute(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_divergence_compute<true>(SPH::DFSPHCData& data);
template void cuda_divergence_compute<false>(SPH::DFSPHCData& data);

__global__ void DFSPH_divergence_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	{
		///TODO when doing this kernel I can actually fuse the code for all those computation to limit the number
		///of time I read the particles positions
		computeDensityChange(m_data, i);

		//I can actually make the factor and desity computation here
		{
			//////////////////////////////////////////////////////////////////////////
			// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
			//////////////////////////////////////////////////////////////////////////
			const Vector3d &xi = m_data.fluid_data_cuda->pos[i];
			RealCuda sum_grad_p_k = 0;
			Vector3d grad_p_i;
			grad_p_i.setZero();

			RealCuda density = m_data.fluid_data_cuda->mass[i] * m_data.W_zero;

			//////////////////////////////////////////////////////////////////////////
			// Fluid
			//////////////////////////////////////////////////////////////////////////
			int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(i);
			int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(i);
			{
				SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
				while (neighbors_ptr != end_ptr)
				{
					const unsigned int neighborIndex = *neighbors_ptr++;
					const Vector3d &xj = body.pos[neighborIndex];
					density += body.mass[neighborIndex] * m_data.W(xi - xj);
					const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
					sum_grad_p_k += grad_p_j.squaredNorm();
					grad_p_i += grad_p_j;
				}
			}

			//////////////////////////////////////////////////////////////////////////
			// Boundary
			//////////////////////////////////////////////////////////////////////////
			end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 1);
			{
				SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);
				while (neighbors_ptr != end_ptr)
				{
					const unsigned int neighborIndex = *neighbors_ptr++;
					const Vector3d &xj = body.pos[neighborIndex];
					density += body.mass[neighborIndex] * m_data.W(xi - xj);
					const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
					sum_grad_p_k += grad_p_j.squaredNorm();
					grad_p_i += grad_p_j;
				}
			}

			//////////////////////////////////////////////////////////////////////////
			// Dynamic bodies
			//////////////////////////////////////////////////////////////////////////
			//*
			end_ptr += m_data.fluid_data_cuda->getNumberOfNeighbourgs(i, 2);
			while (neighbors_ptr != end_ptr)
			{
				READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
				SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];
				const Vector3d &xj = body.pos[neighborIndex];
				density += body.mass[neighborIndex] * m_data.W(xi - xj);
				const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
				sum_grad_p_k += grad_p_j.squaredNorm();
				grad_p_i += grad_p_j;
			}
			//*/


			sum_grad_p_k += grad_p_i.squaredNorm();

			//////////////////////////////////////////////////////////////////////////
			// Compute pressure stiffness denominator
			//////////////////////////////////////////////////////////////////////////
			m_data.fluid_data_cuda->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
			m_data.fluid_data_cuda->density[i] = density;

		}

#ifdef USE_WARMSTART_V
		m_data.fluid_data_cuda->kappaV[i] = 0;
#endif
	}

}
void cuda_divergence_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_divergence_loop_end_kernel(SPH::DFSPHCData m_data, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityChange(m_data, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}
RealCuda cuda_divergence_loop_end(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	RealCuda* avg_density_err = NULL;
	if (avg_density_err == NULL) {
		hipMalloc(&(avg_density_err), sizeof(RealCuda));
	}

	DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, avg_density_err);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
	static void     *d_temp_storage = NULL;
	static size_t   temp_storage_bytes = 0;

	if (d_temp_storage == NULL) {
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.numFluidParticles);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
	}
	// Run sum-reduction
	hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.numFluidParticles);

	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));

	return result;
}

__global__ void DFSPH_viscosityXSPH_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	const Vector3d &xi = m_data.fluid_data_cuda->pos[i];
	const Vector3d &vi = m_data.fluid_data_cuda->vel[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	int* neighbors_ptr = m_data.fluid_data_cuda->getNeighboursPtr(i);
	int* end_ptr = neighbors_ptr + m_data.fluid_data_cuda->getNumberOfNeighbourgs(i);
	{
		SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);
		while (neighbors_ptr != end_ptr)
		{
			const unsigned int neighborIndex = *neighbors_ptr++;

			// Viscosity
			ai -= m_data.invH * m_data.viscosity * (body.mass[neighborIndex] / body.density[neighborIndex]) *
				(vi - body.vel[neighborIndex]) * m_data.W(xi - body.pos[neighborIndex]);

		}
	}

	m_data.fluid_data_cuda->acc[i] = m_data.gravitation + ai;
}
void cuda_viscosityXSPH(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_viscosityXSPH_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_viscosityXSPH failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_CFL_kernel(SPH::DFSPHCData m_data, RealCuda* maxVel) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	for (unsigned int i = 0; i < m_data.numFluidParticles; i++)
	{
		const RealCuda velMag = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
		if (velMag > *maxVel)
			*maxVel = velMag;
	}
}

__global__ void DFSPH_CFLVelSquaredNorm_kernel(SPH::DFSPHCData m_data, RealCuda* sqaredNorm) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	sqaredNorm[i] = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
}

__global__ void DFSPH_CFLAdvanced_kernel(SPH::DFSPHCData m_data, RealCuda *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ RealCuda cache[256];


	RealCuda temp = 0;
	while (index + offset < n) {
		int i = index + offset;
		const RealCuda velMag = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
		temp = fmaxf(temp, velMag);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = MAX_MACRO_CUDA(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) != 0);  //lock
		*max = MAX_MACRO_CUDA(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}
void cuda_CFL(SPH::DFSPHCData& m_data, const RealCuda minTimeStepSize, RealCuda m_cflFactor, RealCuda m_cflMaxTimeStepSize) {

	//we compute the square norm

	std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();

	RealCuda* out_buff;
	hipMalloc(&(out_buff), sizeof(RealCuda));

	if (true) {

		//cub version
		static RealCuda* temp_buff = NULL;
		if (temp_buff == NULL) {
			hipMallocManaged(&(temp_buff), m_data.numFluidParticles * sizeof(RealCuda));
		}
		int numBlocks = (m_data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_CFLVelSquaredNorm_kernel << <numBlocks, BLOCKSIZE >> > (m_data, temp_buff);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl squared norm failed: %d\n", (int)cudaStatus);
			exit(1598);
		}

		// Determine temporary device storage requirements
		static void     *d_temp_storage = NULL;
		static size_t   temp_storage_bytes = 0;
		if (d_temp_storage == NULL) {
			hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.numFluidParticles);
			// Allocate temporary storage
			hipMalloc(&d_temp_storage, temp_storage_bytes);
		}
		// Run max-reduction
		hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.numFluidParticles);

	}
	else {
		//manual
		int *d_mutex;
		hipMalloc((void**)&d_mutex, sizeof(int));
		hipMemset(d_mutex, 0, sizeof(float));

		int numBlocks = (m_data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_CFLAdvanced_kernel << < numBlocks, BLOCKSIZE >> > (m_data, out_buff, d_mutex, m_data.numFluidParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl failed: %d\n", (int)cudaStatus);
			exit(1598);
		}
		hipFree(d_mutex);
	}

	RealCuda maxVel;
	hipMemcpy(&maxVel, out_buff, sizeof(RealCuda), hipMemcpyDeviceToHost);
	hipFree(out_buff);

	std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

	RealCuda h = m_data.h;

	// Approximate max. time step size 		
	h = m_cflFactor * .4 * (2.0*m_data.particleRadius / (sqrt(maxVel)));

	h = min(h, m_cflMaxTimeStepSize);
	h = max(h, minTimeStepSize);

	m_data.updateTimeStep(h);//*/


	std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();



	float time_search = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
	float time_comp = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;

	printf("Time to do cfl (search,comp): %f    %f\n", time_search, time_comp);
}

__global__ void DFSPH_update_vel_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.fluid_data_cuda->vel[i] += m_data.h * m_data.fluid_data_cuda->acc[i];

#ifdef USE_WARMSTART	
	//done here to have one less kernel
	m_data.fluid_data_cuda->kappa[i] = MAX_MACRO_CUDA(m_data.fluid_data_cuda->kappa[i] * m_data.h_ratio_to_past2, -0.5);
#endif
}
void cuda_update_vel(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_update_vel_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_vel failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

template<bool warmstart> __global__ void DFSPH_pressure_compute_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	pressureSolveParticle<warmstart>(m_data, i);

}
template<bool warmstart> void cuda_pressure_compute(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_pressure_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_pressure_compute<true>(SPH::DFSPHCData& data);
template void cuda_pressure_compute<false>(SPH::DFSPHCData& data);

__global__ void DFSPH_pressure_init_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityAdv(m_data, i);

	m_data.fluid_data_cuda->factor[i] *= m_data.invH_future;
#ifdef USE_WARMSTART
	m_data.fluid_data_cuda->kappa[i] = 0;
#endif

}
void cuda_pressure_init(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_pressure_loop_end_kernel(SPH::DFSPHCData m_data, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	computeDensityAdv(m_data, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}
//*
__global__ void DFSPH_pressure_loop_end_kernel(int numFluidParticles, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
	RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
	SPH::UnifiedParticleSet* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }

	computeDensityAdv(i, posFluid, velFluid, neighbourgs, numberOfNeighbourgs,
		mass, m_kernel_precomp, boundaryPsi, posBoundary, velBoundary,
		vector_dynamic_bodies_data_cuda, densityAdv, density, h_future, density0);
}//*/

RealCuda cuda_pressure_loop_end(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;

	std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
	static RealCuda* avg_density_err = NULL;
	if (avg_density_err == NULL) {
		hipMalloc(&(avg_density_err), sizeof(RealCuda));
	}

	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, avg_density_err);

	/*
	///LOL the detailed implementation is slower so no need to even think about developping data
	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles, data.posFluid, data.velFluid,
	data.neighbourgs, data.numberOfNeighbourgs,
	data.mass, data.m_kernel_precomp, data.boundaryPsi, data.posBoundary, data.velBoundary,
	data.vector_dynamic_bodies_data_cuda, data.densityAdv, data.density, data.h_future, data.density0);
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
	static void     *d_temp_storage = NULL;
	static size_t   temp_storage_bytes = 0;

	if (d_temp_storage == NULL) {
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.numFluidParticles);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
	}
	// Run sum-reduction
	hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.numFluidParticles);


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));


	std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
	float time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
	float time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;

	//std::cout << "pressure loop end details: " << time1 << "  " << time2 << std::endl;

	return result;
}

__global__ void DFSPH_update_pos_kernel(SPH::DFSPHCData m_data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.numFluidParticles) { return; }

	m_data.fluid_data_cuda->pos[i] += m_data.h * m_data.fluid_data_cuda->vel[i];
}
void cuda_update_pos(SPH::DFSPHCData& data) {
	int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_update_pos_kernel << <numBlocks, BLOCKSIZE >> > (data);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_pos failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


int cuda_divergenceSolve(SPH::DFSPHCData& m_data, const unsigned int maxIter, const RealCuda maxError) {
	//////////////////////////////////////////////////////////////////////////
	// Init parameters
	//////////////////////////////////////////////////////////////////////////

	const RealCuda h = m_data.h;
	const int numParticles = m_data.numFluidParticles;
	const RealCuda density0 = m_data.density0;

#ifdef USE_WARMSTART_V
	cuda_divergence_warmstart_init(m_data);
	cuda_divergence_compute<true>(m_data);
#endif


	//////////////////////////////////////////////////////////////////////////
	// Compute velocity of density change
	//////////////////////////////////////////////////////////////////////////
	cuda_divergence_init(m_data);


	unsigned int m_iterationsV = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	// use maximal density error divided by time step size
	const RealCuda eta = maxError * 0.01 * density0 / h;  // maxError is given in percent

	RealCuda avg_density_err = 0.0;
	while (((avg_density_err > eta) || (m_iterationsV < 1)) && (m_iterationsV < maxIter))
	{

		//////////////////////////////////////////////////////////////////////////
		// Perform Jacobi iteration over all blocks
		//////////////////////////////////////////////////////////////////////////	
		cuda_divergence_compute<false>(m_data);

		avg_density_err = cuda_divergence_loop_end(m_data);

		avg_density_err /= numParticles;
		m_iterationsV++;
	}

	return m_iterationsV;
}
int cuda_pressureSolve(SPH::DFSPHCData& m_data, const unsigned int m_maxIterations, const RealCuda m_maxError) {
	const RealCuda density0 = m_data.density0;
	const int numParticles = (int)m_data.numFluidParticles;
	RealCuda avg_density_err = 0.0;


	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


#ifdef USE_WARMSTART		
	cuda_pressure_compute<true>(m_data);
#endif


	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();

	//////////////////////////////////////////////////////////////////////////
	// Compute rho_adv
	//////////////////////////////////////////////////////////////////////////
	cuda_pressure_init(m_data);


	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();


	unsigned int m_iterations = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	const RealCuda eta = m_maxError * 0.01 * density0;  // maxError is given in percent

	float time_3_1 = 0;
	float time_3_2 = 0;
	while (((avg_density_err > eta) || (m_iterations < 2)) && (m_iterations < m_maxIterations))
	{
		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
		cuda_pressure_compute<false>(m_data);
		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
		avg_density_err = cuda_pressure_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
		avg_density_err /= numParticles;

		m_iterations++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
	}
	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - start).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

	std::cout << "detail pressure solve (iter total (warm init actual_comp (t1 t2))): " <<m_iterations <<"  "<< time_1 + time_2 +time_3 <<
	"  (" << time_1 << "  " << time_2<< "  "<< time_3 <<"("<< time_3_1<<" "<< time_3_2<<") )" << std::endl;

	//*/

	return m_iterations;

}


template<unsigned int grid_size, bool z_curve>
__global__ void DFSPH_computeGridIdx_kernel(Vector3d* in, unsigned int* out, RealCuda kernel_radius, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	if (z_curve) {

	}
	else {
		//the +50 is an offset so that I don't use the border of the grid
		//it allosw me to be sure that I won't have particles outside of the grid
		//the main thing is that their domain has negative position values
		//that +10 prevent having any negative index by positioning the bounding area of the particles 
		//incide the area  described by our cells
		Vector3d pos = (in[i] / kernel_radius) + 50;
		out[i] = COMPUTE_CELL_INDEX((int)pos.x, (int)pos.y, (int)pos.z);
		//	(int)pos.x + ((int)pos.y)*CELL_ROW_LENGTH + ((int)pos.z)*grid_size*grid_size;
	}
}

__global__ void DFSPH_Histogram_kernel(unsigned int* in, unsigned int* out, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	atomicAdd(&(out[in[i]]), 1);

}

__global__ void DFSPH_setBufferValueToItself_kernel(unsigned int* buff, unsigned int buff_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= buff_size) { return; }

	buff[i] = i;
}

__global__ void DFSPH_setVector3dBufferToZero_kernel(Vector3d* buff, unsigned int buff_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= buff_size) { return; }

	buff[i] = Vector3d(0, 0, 0);
}

__global__ void DFSPH_neighborsSearch_kernel(unsigned int numFluidParticles, RealCuda radius,
	SPH::UnifiedParticleSet* fluid_data,
	SPH::UnifiedParticleSet* boundaries_data,
	SPH::UnifiedParticleSet* vect_dynamic_bodies, int nb_dynamic_bodies) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }

	RealCuda radius_sq = radius;
	Vector3d pos = fluid_data->pos[i];
	Vector3d pos_cell = (pos / radius_sq) + 50; //on that line the radius is not yet squared
	int x = (int)pos_cell.x;
	int y = (int)pos_cell.y;
	int z = (int)pos_cell.z;
	radius_sq *= radius_sq;

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr = fluid_data->neighbourgs + i*MAX_NEIGHBOURS;
	//int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
	//int neighbors_boundary[MAX_NEIGHBOURS];

#ifdef USE_COMPLETE
	///this version uses the morton indexes
	//this needsto be recoded since the data structure changed



#else
	///this version uses  standart indexes

	//since this version use the std index to be able to iterate on 3 successive cells
	//I can do the -1 at the start on x.
	//one thing: it x=0 then we can only iterate 2 cells at a time
	unsigned int successive_cells_count = (x > 0) ? 3 : 2;
	x = (x > 0) ? x - 1 : x;

#define ITER_CELLS_FOR_BODY(input_body,code){\
		const SPH::UnifiedParticleSet& body = input_body;\
		for (int k = -1; k < 2; ++k) {\
			for (int m = -1; m < 2; ++m) {\
				unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + m, z + k);\
				unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];\
				for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {\
					unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];\
					if ((pos - body.pos[j]).squaredNorm() < radius_sq) {\
						code\
					}\
				}\
			}\
		}\
	}
	
	//fluid
	ITER_CELLS_FOR_BODY(*fluid_data, if (i != j) {*cur_neighbor_ptr++ = j;	nb_neighbors_fluid++;})
	 
	//boundaries
	ITER_CELLS_FOR_BODY(*boundaries_data, *cur_neighbor_ptr++ = j; nb_neighbors_boundary++; )

	
	if (vect_dynamic_bodies != NULL) {
		for (int id_body = 0; id_body < nb_dynamic_bodies; ++id_body) {
			ITER_CELLS_FOR_BODY(vect_dynamic_bodies[id_body], 
				*cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j); nb_neighbors_dynamic_objects++; )
		}
	}
#endif


	fluid_data->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
	fluid_data->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	fluid_data->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;

	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS*2), neighbors_fluid, sizeof(int)*nb_neighbors_fluid);
	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS), neighbors_boundary, sizeof(int)*nb_neighbors_boundary);


}

void cuda_neighborsSearchInternal_sortParticlesId(Vector3d* pos, RealCuda kernel_radius, int numParticles, void **d_temp_storage_pair_sort,
	size_t   &temp_storage_bytes_pair_sort, unsigned int* cell_id, unsigned int* cell_id_sorted,
	unsigned int* p_id, unsigned int* p_id_sorted) {
	hipError_t cudaStatus;

	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel<CELL_ROW_LENGTH, false> << <numBlocks, BLOCKSIZE >> > (pos, cell_id,
		kernel_radius, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//do the actual sort
	//first Determine temporary device storage requirements
	if ((*d_temp_storage_pair_sort) == NULL) {
		temp_storage_bytes_pair_sort = 0;
		hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
			cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
		// Allocate temporary storage
		hipMalloc(d_temp_storage_pair_sort, temp_storage_bytes_pair_sort);

	}

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
		cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
	//*/


	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sort failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

}

void cuda_neighborsSearchInternal_computeCellStartEnd(int numParticles, unsigned int* cell_id_sorted,
	unsigned int* hist, void **d_temp_storage_cumul_hist, size_t   &temp_storage_bytes_cumul_hist, unsigned int* cell_start_end) {
	hipError_t cudaStatus;
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "histogram value reset failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//compute the actual histogram (done here with atomic adds)
	DFSPH_Histogram_kernel << <numBlocks, BLOCKSIZE >> > (cell_id_sorted, hist, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "histogram failed: " << (int)cudaStatus << std::endl;
		exit(1598);
	}

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value

	if ((*d_temp_storage_cumul_hist) == NULL) {
		temp_storage_bytes_cumul_hist = 0;
		//get the necessary size
		hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));
		// Allocate temporary storage
		hipMalloc(d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist);
	}
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cumulative histogram failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}



//this is the bases for all kernels based function
template<typename T>
__global__ void DFSPH_sortFromIndex_kernel(T* in, T* out, unsigned int* index, unsigned int nbElements) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nbElements) { return; }

	out[i] = in[index[i]];
}


#include <sstream>
void cuda_sortData(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& neighborsDataSet) {
	//*
	unsigned int numParticles = neighborsDataSet.numParticles;
	int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	unsigned int *p_id_sorted = neighborsDataSet.p_id_sorted;

	Vector3d* intermediate_buffer_v3d = NULL;
	hipMallocManaged(&(intermediate_buffer_v3d), numParticles * sizeof(Vector3d));
	RealCuda* intermediate_buffer_real = NULL;
	hipMalloc(&(intermediate_buffer_real), numParticles * sizeof(RealCuda));

	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	
	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//mass
	DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

	if (particleSet.velocity_impacted_by_fluid_solver) {
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	}


	hipFree(intermediate_buffer_v3d); intermediate_buffer_v3d = NULL;
	hipFree(intermediate_buffer_real); intermediate_buffer_real = NULL;


	//now that everything is sorted we can set each particle index to itself
	gpuErrchk(hipMemcpy(p_id_sorted, neighborsDataSet.p_id, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice));

	std::cout << "particle set sorting done" << std::endl;
}




void cuda_neighborsSearch(SPH::DFSPHCData& data) {

	std::chrono::steady_clock::time_point begin_global = std::chrono::steady_clock::now();
	static unsigned int time_count = 0;
	float time_global;
	static float time_avg_global = 0;
	time_count++;

	hipError_t cudaStatus;
	{

		float time;
		static float time_avg = 0;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();


		//first let's generate the cell start end for the dynamic bodies
		for (int i = 0; i < data.numDynamicBodies; ++i) {
			SPH::UnifiedParticleSet& body = data.vector_dynamic_bodies_data[i];
			body.initNeighborsSearchData(data.m_kernel_precomp.getRadius(), false);
		}
		

		//now update the cell start end of the fluid particles
		{

			//since it the init iter I'll sort both even if it's the boundaries
			static int step_count = 0;
			step_count++;

			data.fluid_data->initNeighborsSearchData(data.m_kernel_precomp.getRadius(), (step_count%25)==0);


			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "before neighbors search: %d\n", (int)cudaStatus);
				exit(1598);
			}


		}

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		//printf("Time to generate cell start end: %f ms   avg: %f ms \n", time, time_avg / time_count);
	}
	//and we can now do the actual search of the neaighbor for eahc fluid particle
	if (true)
	{
		float time;
		static float time_avg = 0;

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		//cuda way
		int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;


		DFSPH_neighborsSearch_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles,
			data.m_kernel_precomp.getRadius(), 
			data.fluid_data_cuda, 
			data.boundaries_data_cuda,
			data.vector_dynamic_bodies_data_cuda, data.numDynamicBodies);



		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "cuda neighbors search failed: " << (int)cudaStatus << std::endl;
			exit(1598);
		}

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		//printf("Time to generate neighbors buffers: %f ms   avg: %f ms \n", time, time_avg / time_count);

		/*
		//a simple check to know the max nbr of neighbors
		static int absolute_max = 0;
		int max = 0;

		static int absolute_max_d[3] = { 0 };
		int max_d[3] = { 0 };



		for (int j = 0; j < data.numFluidParticles; j++)
		{
		//check the global value
		int count_neighbors = 0;
		for (int k = 0; k < 2; ++k) {
		count_neighbors += data.getNumberOfNeighbourgs(j, k);
		}
		if (count_neighbors > max)max = count_neighbors;

		//chekc the max for each category
		for (unsigned int k = 0; k < 3; ++k) {
		if ((int)data.getNumberOfNeighbourgs(j,k) > max_d[k])max_d[k] = data.getNumberOfNeighbourgs(j,k);
		}

		}
		if (max>absolute_max)absolute_max = max;
		for (unsigned int k = 0; k < 3; ++k) {
		if (max_d[k]>absolute_max_d[k])absolute_max_d[k] = max_d[k];
		}
		printf("max nbr of neighbors %d  (%d) \n", absolute_max, max);
		printf("max nbr of neighbors %d  (%d)      absolute max  fluid // boundaries // bodies   %d // %d // %d\n",
		absolute_max, max, absolute_max_d[0], absolute_max_d[1], absolute_max_d[2]);
		//*/
	}

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	time_global = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin_global).count() / 1000000.0f;

	time_avg_global += time_global;
	//printf("time taken by the neighbor function: %f ms   avg: %f ms \n", time_global, time_avg_global / time_count);
}



void cuda_initNeighborsSearchDataSet(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet,
	RealCuda kernel_radius, bool sortBuffers){

	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(particleSet.pos, kernel_radius, dataSet.numParticles,
		&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
		dataSet.p_id, dataSet.p_id_sorted);

	//since it the init iter I'll sort both even if it's the boundaries
	if (sortBuffers) {
		cuda_sortData(particleSet, dataSet);
	}


	//and now I cna compute the start and end of each cell :)
	cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
		&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);

}


void cuda_renderFluid(SPH::DFSPHCData& data) {
	cuda_opengl_renderParticleSet(*data.fluid_data->renderingData,data.numFluidParticles);
}



void cuda_renderBoundaries(SPH::DFSPHCData& data, bool renderWalls) {
	if (renderWalls) {
		cuda_opengl_renderParticleSet(*(data.boundaries_data->renderingData), data.boundaries_data->numParticles);
	}

	for (int i = 0; i < data.numDynamicBodies; ++i) {
		std::cout<<"test"<<i<<std::endl;
		SPH::UnifiedParticleSet& body= data.vector_dynamic_bodies_data[i];
		cuda_opengl_renderParticleSet(*body.renderingData, body.numParticles);
	}
}

/*
THE NEXT FUNCTIONS ARE FOR THE RENDERING
*/


void cuda_opengl_initParticleRendering(ParticleSetRenderingData& renderingData, unsigned int numParticles,
	Vector3d** pos, Vector3d** vel) {
	glGenVertexArrays(1, &renderingData.vaoFluid); // Cr�er le VAO
	glBindVertexArray(renderingData.vaoFluid); // Lier le VAO pour l'utiliser


	glGenBuffers(1, &renderingData.pos_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, renderingData.pos_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	numParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FORMAT, GL_FALSE, 0, 0);

	glGenBuffers(1, &renderingData.vel_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, renderingData.vel_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	numParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 3, GL_FORMAT, GL_FALSE, 0, 0);

	// nettoyage
	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// Registration with CUDA.
	gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.pos, renderingData.pos_buffer, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.vel, renderingData.vel_buffer, hipGraphicsRegisterFlagsNone));

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));

	//set the openglbuffer for direct use in cuda
	Vector3d* vboPtr = NULL;
	size_t size = 0;

	// pos
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.pos));//get cuda ptr
	*pos = vboPtr;

	// vel
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.vel));//get cuda ptr
	*vel = vboPtr;

}

void cuda_opengl_renderParticleSet(ParticleSetRenderingData& renderingData, unsigned int numParticles) {

	//unlink the pos and vel buffer from cuda
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.pos), 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.vel), 0));

	//Actual opengl rendering
	// link the vao
	glBindVertexArray(renderingData.vaoFluid);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//show it
	glDrawArrays(GL_POINTS, 0, numParticles);

	// unlink the vao
	glBindVertexArray(0);

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));

}






/*
THE NEXT FUNCTIONS ARE FOR THE MEMORY ALLOCATION
*/


void allocate_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container) {

	//hipMalloc(&(container.pos), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
	//hipMalloc(&(container.vel), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
	hipMalloc(&(container.mass), container.numParticles * sizeof(RealCuda));


	if (container.has_factor_computation) {
		//*
		hipMallocManaged(&(container.numberOfNeighbourgs), container.numParticles * 3 * sizeof(int));
		hipMalloc(&(container.neighbourgs), container.numParticles * MAX_NEIGHBOURS * sizeof(int));

		hipMalloc(&(container.density), container.numParticles * sizeof(RealCuda));
		hipMalloc(&(container.factor), container.numParticles * sizeof(RealCuda));
		hipMalloc(&(container.densityAdv), container.numParticles * sizeof(RealCuda));
		
		if (container.velocity_impacted_by_fluid_solver) {
			hipMalloc(&(container.acc), container.numParticles * sizeof(Vector3d));
			hipMalloc(&(container.kappa), container.numParticles * sizeof(RealCuda));
			hipMalloc(&(container.kappaV), container.numParticles * sizeof(RealCuda));
		}
		//*/
	}

	if (container.is_dynamic_object) {
		hipMalloc(&(container.F), container.numParticles * sizeof(Vector3d));
	}

}

void load_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container, Vector3d* pos, Vector3d* vel, RealCuda* mass) {
	gpuErrchk(hipMemcpy(container.pos, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.vel, vel, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.mass, mass, container.numParticles * sizeof(RealCuda), hipMemcpyHostToDevice));

	if (container.is_dynamic_object) {
		int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
		DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);
	}

	if (container.has_factor_computation) {
		
		if (container.velocity_impacted_by_fluid_solver) {
			gpuErrchk(hipMemset(container.kappa, 0, container.numParticles * sizeof(RealCuda)));
			gpuErrchk(hipMemset(container.kappaV, 0, container.numParticles * sizeof(RealCuda)));
		}
	}

}

void read_rigid_body_force_cuda(SPH::UnifiedParticleSet& container) {
	if (container.is_dynamic_object) {
		gpuErrchk(hipMemcpy(container.F_cpu, container.F, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
	}
}

void allocate_and_copy_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** out_vector, SPH::UnifiedParticleSet* in_vector, int numSets) {
	//before to do anythng we need to make a copy of the data structure since
	//we will have to change the neighborsdataset from the cpu to the gpu
	//*
	SPH::UnifiedParticleSet* temp;
	temp = new SPH::UnifiedParticleSet[numSets];
	std::copy(in_vector, in_vector + numSets, temp);
	
	for (int i = 0; i < numSets; ++i) {
		SPH::UnifiedParticleSet& body = temp[i];
		gpuErrchk(hipMalloc(&(body.neighborsDataSet), sizeof(SPH::NeighborsSearchDataSet)));

		gpuErrchk(hipMemcpy(body.neighborsDataSet, in_vector[i].neighborsDataSet,
			sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));

	}
	//*/

	gpuErrchk(hipMalloc(out_vector, numSets * sizeof(SPH::UnifiedParticleSet)));

	gpuErrchk(hipMemcpy(*out_vector, temp,	numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));
}



void allocate_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel, bool minimize_managed) {

	if (minimize_managed) {
		hipMalloc(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		hipMalloc(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
	else {
		fprintf(stderr, "trying to use managed buffers for the kernels\n");
		exit(1256);
		//hipMallocManaged(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		//hipMallocManaged(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
}


void init_precomputed_kernel_from_values(SPH::PrecomputedCubicKernelPerso& kernel, RealCuda* w, RealCuda* grad_W) {
	hipError_t cudaStatus;
	//W
	cudaStatus = hipMemcpy(kernel.m_W,
		w,
		kernel.m_resolution * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//grad W
	cudaStatus = hipMemcpy(kernel.m_gradW,
		grad_W,
		(kernel.m_resolution + 1) * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of grad W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

}


void allocate_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet) {

	//allocatethe mme for fluid particles
	hipMallocManaged(&(dataSet.cell_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.cell_id_sorted), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.local_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.p_id), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.p_id_sorted), dataSet.numParticles * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.cell_start_end), (CELL_COUNT + 1) * sizeof(unsigned int));
	hipMallocManaged(&(dataSet.hist), (CELL_COUNT + 1) * sizeof(unsigned int));

	//init variables for cub calls
	dataSet.d_temp_storage_pair_sort = NULL;
	dataSet.temp_storage_bytes_pair_sort = 0;
	dataSet.d_temp_storage_cumul_hist = NULL;
	dataSet.temp_storage_bytes_cumul_hist = 0;

	//reset the particle id
	int numBlocks = (dataSet.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
	DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id, dataSet.numParticles);
	DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id_sorted, dataSet.numParticles);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "p_id init idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	dataSet.internal_buffers_allocated = true;
}


void release_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet, bool keep_result_buffers) {
	//allocatethe mme for fluid particles
	hipFree(dataSet.cell_id); dataSet.cell_id = NULL;
	hipFree(dataSet.local_id); dataSet.local_id = NULL;
	hipFree(dataSet.p_id); dataSet.p_id = NULL;
	hipFree(dataSet.cell_id_sorted); dataSet.cell_id_sorted = NULL;
	hipFree(dataSet.hist); dataSet.hist = NULL;

	//init variables for cub calls
	hipFree(dataSet.d_temp_storage_pair_sort);
	dataSet.d_temp_storage_pair_sort = NULL;
	dataSet.temp_storage_bytes_pair_sort = 0;
	hipFree(dataSet.d_temp_storage_cumul_hist);
	dataSet.d_temp_storage_cumul_hist = NULL;
	dataSet.temp_storage_bytes_cumul_hist = 0;

	dataSet.internal_buffers_allocated = false;

	if (!keep_result_buffers) {
		hipFree(dataSet.p_id_sorted); dataSet.p_id_sorted = NULL;
		hipFree(dataSet.cell_start_end); dataSet.cell_start_end = NULL;
	}
}























/*
AFTER THIS ARE ONLY THE TEST FUNCTION TO HAVE CUDA WORKING ...
*/


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//*
__global__ void addKernel(Vector3d* vect)
{
	int i = threadIdx.x;
	vect[i].z = vect[i].x + vect[i].y;
}

__global__ void setVectkernel(Vector3d& vect)
{
	vect.x = 5;
	vect.y = 6;
	vect.z = 7;
}
//*/
int test_cuda()
{
	//DFSPHCData* data;

	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	//*
	Vector3d* vect;
	hipMallocManaged(&vect, arraySize * sizeof(Vector3d));
	for (int i = 0; i < arraySize; ++i) {
		vect[i].x = a[i];
		vect[i].y = b[i];
	}
	//*/*

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	printf("macro val: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	for (int i = 0; i < arraySize; ++i) {
		c[i] = 0;
	}


	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, arraySize >> > (vect);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	printf("with vects {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		(int)(vect[0].z), (int)(vect[1].z), (int)(vect[2].z), (int)(vect[3].z), (int)(vect[4].z));

	hipFree(vect);



	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	/*
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipDeviceReset failed!");
	return 1;
	}
	//*/

	printf("Finished test cuda\n");


	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
