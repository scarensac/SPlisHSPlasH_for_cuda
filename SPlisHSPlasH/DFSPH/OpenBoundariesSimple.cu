#include "hip/hip_runtime.h"
#include "OpenBoundariesSimple.h"
#include "DFSPH_core_cuda.h"

#include <stdio.h>
#include <chrono>
#include <iostream>
#include <thread>
#include <sstream>
#include <fstream>

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"
#include "DFSPH_static_variables_structure_cuda.h"


#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"

#include "SPlisHSPlasH/Utilities/SegmentedTiming.h"
#include "SPH_other_systems_cuda.h"


#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "basic_kernels_cuda.cuh"


namespace SPH {
	class OpenBoundariesSimple {
	public:
		bool _isinitialized;

		//reprent the actual simulation boundary
		BufferFluidSurface S_boundary;

		//is slighly smaller than the actual boundary
		//is is used to quickly extract the layer of particles that are right next to the boundary
		BufferFluidSurface S_fluidInterior;

		//should represent the fluid surface/height
		BufferFluidSurface S_fluidSurface;

		//technicaly I could use a simple vector3d* but since I have to use
		//an unified particle set to load it I might as well keep it it may be usefull oneday
		UnifiedParticleSet* inflowPositionsSet;

		OpenBoundariesSimple() {
			_isinitialized = false;
			inflowPositionsSet = NULL;
		};

		~OpenBoundariesSimple() {

		};

		static OpenBoundariesSimple& getStructure() {
			static OpenBoundariesSimple obs;
			return obs;
		}

		bool isInitialized() { return _isinitialized; }

		void init(DFSPHCData& data, OpenBoundariesSimpleInterface::InitParameters& params);

		void applyOpenBoundary(DFSPHCData& data, OpenBoundariesSimpleInterface::ApplyParameters& params);
	};
}


void OpenBoundariesSimpleInterface::init(DFSPHCData& data, OpenBoundariesSimpleInterface::InitParameters& params) {
	OpenBoundariesSimple::getStructure().init(data, params);
}

void OpenBoundariesSimpleInterface::applyOpenBoundary(DFSPHCData& data, OpenBoundariesSimpleInterface::ApplyParameters& params) {
	OpenBoundariesSimple::getStructure().applyOpenBoundary(data, params);
}


__global__ void inflow_compute_and_store_constant_density_contribution_kernel(DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	Vector3d p_i = particleSet->pos[i];

	RealCuda density = particleSet->getMass(i) * data.W_zero;

		ITER_NEIGHBORS_INIT(data, particleSet, i);
	SPH::UnifiedParticleSet* otherSet;

	//I need to skip the fluid particles from the buffer since I only whan the constant contribution
	ADVANCE_END_PTR(end_ptr, particleSet->getNumberOfNeighbourgs(i, 0));
	neighbors_ptr = end_ptr;


	//for boundaries and solids since they do not move it is only needed to compute it once at the start
	//then boundaires
	otherSet = data.boundaries_data_cuda;
	ITER_NEIGHBORS_FROM_STORAGE(data, particleSet, i, 1,
		{
			RealCuda density_delta = otherSet->getMass(neighborIndex) * KERNEL_W(data, p_i - otherSet->pos[neighborIndex]);
			density += density_delta;
		}
	);

	particleSet->densityAdv[i] = density;
}

void OpenBoundariesSimple::init(DFSPHCData& data, OpenBoundariesSimpleInterface::InitParameters& params) {
	if (isInitialized()) {
		std::cout << "OpenBoundariesSimple::init was already initialized" << std::endl;
		return;
	}



	//init the surfaces
	if (params.simulation_config == 0) {
		S_boundary.setCylinder(Vector3d(0, 0, 0), 10, 1.5);
		S_fluidInterior.setCylinder(Vector3d(0, 0, 0), 10, S_boundary.getRadius() - data.particleRadius * 3);
		S_fluidSurface.setPlane(Vector3d(0, 1, 0), Vector3d(0, -1, 0));
	}
	else if (params.simulation_config == 1) {
		//S_boundary.setCylinder(Vector3d(0, 0, 0), 10, 1.5);
		//S_fluidInterior.setCylinder(Vector3d(0, 0, 0), 10, S_boundary.getradius() - data.particleRadius * 3);

		S_boundary.setCuboid(Vector3d(0, 0, 0), Vector3d(1, 10, 5));
		S_fluidInterior.setCuboid(Vector3d(0, 0, -1), Vector3d(1, 10, 1+S_boundary.getHalfLength().z - data.particleRadius * 5));
		
		S_fluidSurface.setPlane(Vector3d(0, 1, 0), Vector3d(0, -1, 0));
	}
	else {
		std::cout << "OpenBoundariesSimple::init no existing config detected" << std::endl;
		exit(5986);
	}

	//load the positions for the inflow
	Vector3d min_fluid_buffer;
	Vector3d max_fluid_buffer;
	SPH::UnifiedParticleSet* dummy = NULL;
	inflowPositionsSet = new SPH::UnifiedParticleSet();
	inflowPositionsSet->load_from_file(data.fluid_files_folder + "inflowPositionsSet_file.txt", false, &min_fluid_buffer, &max_fluid_buffer, false);
	allocate_and_copy_UnifiedParticleSet_vector_cuda(&dummy, inflowPositionsSet, 1);



	//we need to remove any particle that will not be part of the infow
	//the inflow is a single layer of particle near the boundary
	//so I can use S_fluidInterior for that
	if(true){
		int* outInt = SVS_CU::get()->count_invalid_position;
		*outInt = 0;

		//we have to reverse that surface here 
		S_fluidInterior.setReversedSurface(true);
		
		//clear the buffer used for tagging
		set_buffer_to_value<unsigned int>(inflowPositionsSet->neighborsDataSet->cell_id, TAG_UNTAGGED, inflowPositionsSet->numParticles);

		//find the particles to rmv
		//by limiting to the area near boundary
		{
			int numBlocks = calculateNumBlocks(inflowPositionsSet->numParticles);
			tag_outside_of_surface_kernel<false> << <numBlocks, BLOCKSIZE >> > (inflowPositionsSet->gpu_ptr, S_fluidInterior, outInt, TAG_REMOVAL);
			gpuErrchk(hipDeviceSynchronize());
		}

		//and restricting it to the height decided by the inflow
		///TODO move this so that th inflow height can by dynamic through the simulation if desired
		{
			int numBlocks = calculateNumBlocks(inflowPositionsSet->numParticles);
			tag_outside_of_surface_kernel<false> << <numBlocks, BLOCKSIZE >> > (inflowPositionsSet->gpu_ptr, S_fluidSurface, outInt, TAG_REMOVAL);
			gpuErrchk(hipDeviceSynchronize());
		}

		gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::init before callign removal function: ", params.show_debug));

		//and remove the particle if needed
		int count_to_rmv = *outInt;
		if (count_to_rmv > 0) {
			//and remove the particles	
			//*
			remove_tagged_particles(inflowPositionsSet, inflowPositionsSet->neighborsDataSet->cell_id,
				inflowPositionsSet->neighborsDataSet->cell_id_sorted, count_to_rmv,false, false);
			//*/
		}

		//return the surface to normal
		S_fluidInterior.setReversedSurface(false);
	}




	//a test that replaces the fluid data with the inflow buffer data to see what is hapening
	if(false){
		data.fluid_data->updateActiveParticleNumber(inflowPositionsSet->numParticles);

		gpuErrchk(hipMemcpy(data.fluid_data->mass, inflowPositionsSet->mass, 
			inflowPositionsSet->numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
		gpuErrchk(hipMemcpy(data.fluid_data->pos, inflowPositionsSet->pos,
			inflowPositionsSet->numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
		gpuErrchk(hipMemcpy(data.fluid_data->vel, inflowPositionsSet->vel,
			inflowPositionsSet->numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
		gpuErrchk(hipMemcpy(data.fluid_data->color, inflowPositionsSet->color,
			inflowPositionsSet->numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	}


	gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::init before computing constant contrib: ", params.show_debug));

	//compute the contribution of the boundaries and store it
	//no need or that a density based condition for adding the particle does not work...
	if (false){
		int numBlocks = calculateNumBlocks(inflowPositionsSet->numParticles);
		inflow_compute_and_store_constant_density_contribution_kernel << <numBlocks, BLOCKSIZE >> > (data,
			inflowPositionsSet->gpu_ptr);
		gpuErrchk(hipDeviceSynchronize());
		
	}


	_isinitialized = true;

	gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::init end: ", params.show_debug));

}

__global__ void inflow_with_predefined_positions_kernel(DFSPHCData data, SPH::UnifiedParticleSet* particleSet, 
	int* countAdded, RealCuda allowedNewDistance, BufferFluidSurface S_boundary){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }
	
	Vector3d p_i = particleSet->pos[i];

	RealCuda min_dist=100;
	
	//compute the contribution that change every time step
	{
		//i don't have the neighbors stored since the structure will only be used once every timestep
		ITER_NEIGHBORS_INIT_FROM_STRUCTURE(data, particleSet, i);

		//search the fluid neighbors
		UnifiedParticleSet* otherSet = data.fluid_data_cuda;
		ITER_NEIGHBORS_FROM_STRUCTURE(otherSet->neighborsDataSet, otherSet->pos,
			{
				RealCuda dist= (p_i - otherSet->pos[j]).norm();
				min_dist = MIN_MACRO_CUDA(min_dist, dist);
			}
		);


		//the dynamic bodies
		if (data.numDynamicBodies > 0) {
			for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
				otherSet = &data.vector_dynamic_bodies_data_cuda[id_body];
				ITER_NEIGHBORS_FROM_STRUCTURE(otherSet->neighborsDataSet, otherSet->pos,
					{
						RealCuda dist = (p_i - otherSet->pos[j]).norm();
						min_dist = MIN_MACRO_CUDA(min_dist, dist);
					}
				);
			}
		}
	}

	//writte the particle to the memory and initialize whatever is needed
	if (min_dist > allowedNewDistance) {
		//first compute the velocity of the new particle
		//do it with a pondered avg
		Vector3d v(0, 0, 0);
		bool add_particle = true;
		if(false){
			RealCuda sum_weights = 0;
			ITER_NEIGHBORS_INIT_FROM_STRUCTURE(data, particleSet, i);

			//iter over existing fluid particles
			UnifiedParticleSet* otherSet = data.fluid_data_cuda;
			ITER_NEIGHBORS_FROM_STRUCTURE(otherSet->neighborsDataSet, otherSet->pos,
				{
					RealCuda weight = KERNEL_W(data, p_i - otherSet->pos[j]);
					v += otherSet->vel[j] * weight;
					sum_weights += weight;
				}
			);
			if (sum_weights > 0) {
				v /= sum_weights;
			}

			//check that the velocity is not toward the boundary
			Vector3d surface_normal=S_boundary.getNormal(p_i);
			
			if (v.dot(surface_normal) < 0){
				add_particle = false;
			}
		}


		if(add_particle){
			//if all fine add the new particle

			//first get a unique index
			int id= atomicAdd(countAdded, 1);
			id += data.fluid_data_cuda->numParticles;

			//and writte the information 
			data.fluid_data_cuda->pos[id] = p_i;
			data.fluid_data_cuda->vel[id] = v;
			data.fluid_data_cuda->mass[id] = data.fluid_data_cuda->mass[0];
			data.fluid_data_cuda->kappa[id] = 0;
			data.fluid_data_cuda->kappaV[id] = 0;
			if (data.fluid_data_cuda->has_color_buffer) {
				data.fluid_data_cuda->color[id] = Vector3d(-1,-1,-1);
			}
		}


	}

	
}



__global__ void outflow_basic_kernel(DFSPHCData data, SPH::UnifiedParticleSet* particleSet,
	int* countToRmv, BufferFluidSurface S_fluidInterior, BufferFluidSurface S_fluidSurface) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	Vector3d p_i = particleSet->pos[i];

	if (!S_fluidInterior.isinside(p_i)) 
	{
		if (!S_fluidSurface.isinside(p_i)) 
		{
			particleSet->neighborsDataSet->cell_id[i] = TAG_REMOVAL;
			atomicAdd(countToRmv, 1);
		}
	}

}



/*

*/


void OpenBoundariesSimple::applyOpenBoundary(DFSPHCData& data, OpenBoundariesSimpleInterface::ApplyParameters& params) {
	if (!isInitialized()) {
		std::cout << "OpenBoundariesSimple::applyOpenBoundary the structure need to be initialized before" << std::endl;
		return;
	}

	//first let's apply the inflow
	//for this version of the inflow, the strategy is to check at every positions of the inflow buffer
	//then check if there is enougth space for a new particle, and if there is compute the velocity of the new particle
	
	int* outInt = SVS_CU::get()->count_invalid_position;
	if(params.useInflow){
		if (params.allowedNewDistance <= 0) {
			std::cout << "OpenBoundariesSimple::applyOpenBoundary: an invalid min distance was spacified for inflow: " <<
				params.allowedNewDistance << std::endl;
			exit(1256);
		}


		//add more particle in case there might be a max near the curretn number
		if (data.fluid_data->numParticles > (data.fluid_data->numParticlesMax*0.75)) {
			data.fluid_data->changeMaxParticleNumber(data.fluid_data->numParticlesMax * 2);
		}

		data.fluid_data->initNeighborsSearchData(data, false, false);


		gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::applyOpenBoundary before applying inflow: ", params.show_debug))


		*outInt = 0;
		{
			int numBlocks = calculateNumBlocks(inflowPositionsSet->numParticles);
			inflow_with_predefined_positions_kernel << <numBlocks, BLOCKSIZE >> > (data, inflowPositionsSet->gpu_ptr, 
				outInt, params.allowedNewDistance, S_boundary);
			gpuErrchk(hipDeviceSynchronize());
		}
		int count_to_add = *outInt;


		gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::applyOpenBoundary after applying inflow: ", params.show_debug))

		//if some particles have been added change the count
		if (count_to_add > 0) {
			data.fluid_data->updateActiveParticleNumber(data.fluid_data->numParticles + count_to_add);
		}
	}


	//next the outflow
	//for this version of the outflow I'll simply remove any particle too close from the boundary and above the desired fluid surface
	//clear the buffer used for tagging
	if (params.useOutflow) {
		set_buffer_to_value<unsigned int>(data.fluid_data->neighborsDataSet->cell_id, TAG_UNTAGGED, inflowPositionsSet->numParticles);
		*outInt = 0;
		{
			int numBlocks = calculateNumBlocks(data.fluid_data->numParticles);
			outflow_basic_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data->gpu_ptr,
				outInt, S_fluidInterior, S_fluidSurface);
			gpuErrchk(hipDeviceSynchronize());
		}
		int count_to_remove = *outInt;

		if (count_to_remove > 0) {
			remove_tagged_particles(data.fluid_data, data.fluid_data->neighborsDataSet->cell_id,
				data.fluid_data->neighborsDataSet->cell_id_sorted, count_to_remove);
		}
	}

	gpuErrchk(read_last_error_cuda("OpenBoundariesSimple::applyOpenBoundary end: ", params.show_debug));
}