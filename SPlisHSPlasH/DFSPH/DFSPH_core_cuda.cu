#include "hip/hip_runtime.h"
#include "DFSPH_core_cuda.h"


#include <stdio.h>
#include <chrono>
#include <iostream>
#include <thread>
#include <sstream>
#include <fstream>

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"
#include "DFSPH_static_variables_structure_cuda.h"


#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"



#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "basic_kernels_cuda.cuh"

//#include "SPH_memory_storage_precomp_kernels.cuh"



////////////////////////////////////////////////////
/////////       constant memory kernel /////////////
////////////////////////////////////////////////////

#include "SPlisHSPlasH\BasicTypes.h"
#include <string>
#include <vector>

#include "SPlisHSPlasH\Vector.h"
#include "SPlisHSPlasH\Quaternion.h"

#include "DFSPH_define_c.h"
#include "hip/hip_runtime.h"

namespace CoreCuda
{
	__global__ void init_buffer_kernel(Vector3d* buff, unsigned int size, Vector3d val) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= size) { return; }

		buff[i] = val;
	}
}

//#include "SPH_memory_storage_precomp_kernels.cuh"
//*
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 600)
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif//*/

#ifdef PRECOMPUTED_KERNELS_USE_CONSTANT_MEMORY

__constant__ RealCuda m_W[PRECOMPUTED_KERNELS_SAMPLE_COUNT];
__constant__ RealCuda m_gradW[PRECOMPUTED_KERNELS_SAMPLE_COUNT];
__constant__ RealCuda m_radius;
__constant__ RealCuda m_radius2;
__constant__ RealCuda m_invStepSize;





__device__  RealCuda get_constant_W_cuda(const SPH::Vector3d &r)
{
	RealCuda res = 0.0;
	const RealCuda r2 = r.squaredNorm();
	if (r2 <= m_radius2)
	{
		const RealCuda r = sqrt(r2);
		const unsigned int pos = (unsigned int)(r * m_invStepSize);
		res = m_W[pos];
	}
	return res;
}

//*
__device__  RealCuda get_constant_W_cuda(const RealCuda r)
{
	RealCuda res = 0.0;
	if (r <= m_radius)
	{
		const unsigned int pos = (unsigned int)(r * m_invStepSize);
		res = m_W[pos];
	}
	return res;
}
__device__  SPH::Vector3d get_constant_grad_W_cuda(const SPH::Vector3d &r)
{
	SPH::Vector3d res;
	const RealCuda r2 = r.squaredNorm();
	if (r2 <= m_radius2)
	{
		const RealCuda rl = sqrt(r2);
		const unsigned int pos = (unsigned int)(rl * m_invStepSize);
		res = m_gradW[pos] * r;
	}
	else
		res.setZero();

	return res;
}
//*/


#include "SPH_memory_storage_precomp_kernels.h"
#include "SPH_other_systems_cuda.h"
#include <iostream>

void writte_to_precomp_kernel(RealCuda* W_i, RealCuda* gradW_i, RealCuda radius, RealCuda radius2, RealCuda invStepSize) {
	hipMemcpyToSymbol(HIP_SYMBOL(m_W), W_i, sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_gradW), gradW_i, sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_radius), &radius, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_radius2), &radius2, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_invStepSize), &invStepSize, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipDeviceSynchronize();
	/*
	for (int i = 0; i < PRECOMPUTED_KERNELS_SAMPLE_COUNT; ++i) {
	std::cout << "kernel values: " << W_i[i] << "  " << gradW_i[i] << std::endl;
	}

	test_constant_mem_precomp_kernel_cuda();
	//*/
}
#include "DFSPH_macro_cuda.h"

__global__ void test_constant_mem_precomp_kernel_kernel(RealCuda* W, SPH::Vector3d* gradW, RealCuda* r, RealCuda* r2, RealCuda* invd) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= PRECOMPUTED_KERNELS_SAMPLE_COUNT) { return; }

	const RealCuda posX = 1.0 / m_invStepSize * (RealCuda)i;
	SPH::Vector3d distance = Vector3d(posX, 0.0, 0.0);

	W[i] = KERNEL_W("data", distance);
	gradW[i] = KERNEL_GRAD_W("data", distance);

	if (i == 0) {
		*r = m_radius;
		*r2 = m_radius2;
		*invd = m_invStepSize;
	}
}

void test_constant_mem_precomp_kernel_cuda() {
	RealCuda* W;
	SPH::Vector3d* gradW;
	RealCuda* r;
	RealCuda* r2;
	RealCuda* invd;

	hipMallocManaged(&(W), sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	hipMallocManaged(&(gradW), sizeof(SPH::Vector3d) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	hipMallocManaged(&(r), sizeof(RealCuda));
	hipMallocManaged(&(r2), sizeof(RealCuda));
	hipMallocManaged(&(invd), sizeof(RealCuda));

	{//fluid
		int numBlocks = (PRECOMPUTED_KERNELS_SAMPLE_COUNT + BLOCKSIZE - 1) / BLOCKSIZE;
		test_constant_mem_precomp_kernel_kernel << <numBlocks, BLOCKSIZE >> > (W, gradW, r, r2, invd);
	}
	hipDeviceSynchronize();

	for (int i = 0; i < PRECOMPUTED_KERNELS_SAMPLE_COUNT; ++i) {
		std::cout << "kernel values: " << W[i] << "  " << gradW[i].x << std::endl;
	}

	std::cout << "end_values: " << *r << "  " << *r2 << "  " << *invd << std::endl;

	CUDA_FREE_PTR(W);
	CUDA_FREE_PTR(gradW);
	CUDA_FREE_PTR(r);
	CUDA_FREE_PTR(r2);
	CUDA_FREE_PTR(invd);
}
#endif // !BLOCKER//see the macro_cuda_file_for an explanaitions







////////////////////////////////////////////////////
/////////       DIVERGENCE SOLVER      /////////////
////////////////////////////////////////////////////



template <bool ignore_when_no_fluid_near>
__global__ void DFSPH_divergence_warmstart_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE) &&
			(particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE_NEIGHBORS)) {
			return;
		}
	}else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active + m_data.count_active_neighbors)) { return; }
	}

	if (ignore_when_no_fluid_near) {
		if (particleSet->getNumberOfNeighbourgs(i) == 0) {
			return;
		}
	}

	particleSet->kappaV[i] = MAX_MACRO_CUDA(particleSet->kappaV[i] * m_data.h_ratio_to_past / 2, -0.5);
	//computeDensityChange(m_data, i);


	//I can actually make the factor and density computation here
	{
#ifndef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = 0;
	#define computeDensityChange_additional numNeighbors++;
#else
	#define computeDensityChange_additional  
#endif // !STORE_PARTICLE_NEIGHBORS


		//////////////////////////////////////////////////////////////////////////
		// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
		//////////////////////////////////////////////////////////////////////////
		const Vector3d &xi = particleSet->pos[i];
		const Vector3d &vi = particleSet->vel[i];
		RealCuda sum_grad_p_k = 0;
		Vector3d grad_p_i;
		grad_p_i.setZero();

		RealCuda density =  particleSet->getMass(i) * m_data.W_zero;
		RealCuda densityAdv = 0;

		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data,particleSet, i);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);


		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES

		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
		density += mass * KERNEL_W(m_data, xi - xj);
		const Vector3d grad_p_j = mass * KERNEL_GRAD_W(m_data, xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		//No Vj for statics boundaries
		densityAdv += (vi).dot(grad_p_j);


#else

		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);

#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////
		//*
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);
		//*/


		sum_grad_p_k += grad_p_i.squaredNorm();

		//////////////////////////////////////////////////////////////////////////
		// Compute pressure stiffness denominator
		//////////////////////////////////////////////////////////////////////////
		particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
		particleSet->density[i] = density;
		/*
		if ((particleSet->neighborsDataSet->cell_id[i] == TAG_ACTIVE_NEIGHBORS)) {
			printf("mass/height/density: %f %f %f\n", particleSet->mass[i], xi.y, density);
		}
		//*/

		//end the density adv computation
#ifdef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(i);
#endif //STORE_PARTICLE_NEIGHBORS
		// in case of particle deficiency do not perform a divergence solve
		if (numNeighbors < 20) {
			for (unsigned int pid = 1; pid < 3; pid++)
			{
				numNeighbors += particleSet->getNumberOfNeighbourgs(i, pid);
			}
		}
		if (numNeighbors < 20) {
			particleSet->densityAdv[i] = 0;
		}
		else {
			particleSet->densityAdv[i] = MAX_MACRO_CUDA(densityAdv, 0.0);

		}

		/*
		if (density < 998) {
			particleSet->color[i] = Vector3d(1, 0, 0);
			if (density < 995) {
				particleSet->color[i] = Vector3d(0, 1, 0);
				if (density < 990) {
					particleSet->color[i] = Vector3d(0, 0, 1);
				}
			}
		}
		//*/
	}

}

void cuda_divergence_warmstart_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_warmstart_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_divergence_warmstart_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}
	//*/


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_warmstart_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


template <bool warm_start> __device__ void divergenceSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
	Vector3d v_i = Vector3d(0, 0, 0);
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? particleSet->kappaV[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART_V
	if (!warm_start) { particleSet->kappaV[i] += ki; }
#endif

	const Vector3d &xi = particleSet->pos[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum *  body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);

#ifdef USE_BOUNDARIES_DYNAMIC_PROPERTiES
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum *  body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);
#endif


	if (fabs(ki) > m_eps)
	{
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
#ifndef USE_BOUNDARIES_DYNAMIC_PROPERTiES

#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
		const Vector3d delta = ki * mass * KERNEL_GRAD_W(m_data, xi - xj);
		v_i += delta;// ki already contains inverse density
#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			const Vector3d delta = ki * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density
		);
#endif

#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////

		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			Vector3d delta = ki * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density

					 //we apply the force to the body particle (no invH since it has been fatorized at the end)
		delta *= -particleSet->getMass(i);
		atomicAdd(&(body.F[neighborIndex].x), delta.x);
		atomicAdd(&(body.F[neighborIndex].y), delta.y);
		atomicAdd(&(body.F[neighborIndex].z), delta.z);
		);
	}

	particleSet->vel[i] += v_i*m_data.h;
}


//WARNING !!! this is not suposed to be called for the fluid this function is used for boundaries and object for witch 
//doing the velocity variation computation makes no sense but still need the accumulation of kappa for the warm start
__global__ void DFSPH_divergence_accumulate_kappaV_kernel(SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	const RealCuda ki = (particleSet->densityAdv[i])*particleSet->factor[i];
	particleSet->kappaV[i] += ki;
}


template<bool warmstart> __global__ void DFSPH_divergence_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE)) {
			return;
		}
	}else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active)) { return; }
	}

	if (warmstart) {
		if (particleSet->densityAdv[i] > 0.0) {
			divergenceSolveParticle<warmstart>(m_data, particleSet, i);
		}
	}
	else {
		divergenceSolveParticle<warmstart>(m_data, particleSet, i);
	}

}

template<bool warmstart> void cuda_divergence_compute(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		if (!warmstart) {
			int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
			DFSPH_divergence_accumulate_kappaV_kernel << <numBlocks, BLOCKSIZE >> > (data.boundaries_data[0].gpu_ptr);
		}
	}
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_divergence_compute<true>(SPH::DFSPHCData& data);
template void cuda_divergence_compute<false>(SPH::DFSPHCData& data);



__device__ void computeDensityChange(const SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
#ifdef STORE_PARTICLE_NEIGHBORS
#define computeDensityChange_additional  
	unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(index);
	// in case of particle deficiency do not perform a divergence solve
	if (numNeighbors < 20) {
		for (unsigned int pid = 1; pid < 3; pid++)
		{
			numNeighbors += particleSet->getNumberOfNeighbourgs(index, pid);
		}
	}
	if (numNeighbors < 20) {
		particleSet->densityAdv[index] = 0;
	}
	else 
#endif //STORE_PARTICLE_NEIGHBORS
	{
#ifndef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = 0;
#define computeDensityChange_additional numNeighbors++;
#endif //STORE_PARTICLE_NEIGHBORS

		RealCuda densityAdv = 0;
		const Vector3d &xi = particleSet->pos[index];
		const Vector3d &vi = particleSet->vel[index];
		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data, particleSet, index);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			index,
			densityAdv += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[index];
		const RealCuda mass = particleSet->V_rigids[index] * particleSet->density0;
		densityAdv += mass* (vi).dot(KERNEL_GRAD_W(m_data, xi - xj));
#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			index,
			densityAdv += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);
#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic Bodies
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			index,
			densityAdv += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);

#ifndef STORE_PARTICLE_NEIGHBORS
		if (numNeighbors < 20) {
			densityAdv = 0;
		}
#endif //STORE_PARTICLE_NEIGHBORS

		// only correct positive divergence
		particleSet->densityAdv[index] = MAX_MACRO_CUDA(densityAdv, 0.0);
	}
}



__global__ void DFSPH_divergence_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE) &&
			(particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE_NEIGHBORS)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active + m_data.count_active_neighbors)) { return; }
	}

	{
#ifdef USE_WARMSTART_V
		particleSet->kappaV[i] = 0;
#endif

		///TODO when doing this kernel I can actually fuse the code for all those computation to limit the number
		///of time I read the particles positions
		computeDensityChange(m_data, particleSet, i);

#ifndef USE_WARMSTART_V
		//I can actually make the factor and desity computation here
		{
			//////////////////////////////////////////////////////////////////////////
			// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
			//////////////////////////////////////////////////////////////////////////
			const Vector3d &xi = particleSet->pos[i];
			RealCuda sum_grad_p_k = 0;
			Vector3d grad_p_i;
			grad_p_i.setZero();

			RealCuda density = particleSet->getMass(i) * m_data.W_zero;

			//////////////////////////////////////////////////////////////////////////
			// Fluid
			//////////////////////////////////////////////////////////////////////////
			ITER_NEIGHBORS_INIT(m_data, particleSet, i);

			ITER_NEIGHBORS_FLUID(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);

			//////////////////////////////////////////////////////////////////////////
			// Boundary
			//////////////////////////////////////////////////////////////////////////
			ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);

			//////////////////////////////////////////////////////////////////////////
			// Dynamic bodies
			//////////////////////////////////////////////////////////////////////////
			//*
			ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.getMass(neighborIndex) * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);
			//*/


			sum_grad_p_k += grad_p_i.squaredNorm();

			//////////////////////////////////////////////////////////////////////////
			// Compute pressure stiffness denominator
			//////////////////////////////////////////////////////////////////////////
			particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
			particleSet->density[i] = density;

		}
#endif


	}

}

void cuda_divergence_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


__global__ void DFSPH_divergence_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE) &&
			(particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE_NEIGHBORS)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active + m_data.count_active_neighbors)) { return; }
	}

	computeDensityChange(m_data, particleSet, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}

RealCuda cuda_divergence_loop_end(SPH::DFSPHCData& data) {
	RealCuda* avg_density_err = SVS_CU::get()->avg_density_err;

	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles );
		DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles );
		DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr, avg_density_err);
	}
	//*/


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}


	// Run sum-reduction
	hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);
	gpuErrchk(hipDeviceSynchronize());


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));

	return result;
}


int cuda_divergenceSolve(SPH::DFSPHCData& m_data, const unsigned int maxIter, const RealCuda maxError) {
	//////////////////////////////////////////////////////////////////////////
	// Init parameters
	//////////////////////////////////////////////////////////////////////////

	const RealCuda h = m_data.h;
	int numParticles = m_data.fluid_data[0].numParticles;
	const RealCuda density0 = m_data.density0;
	if (m_data.true_particle_count >= 0) {
		numParticles = m_data.true_particle_count;
	}

	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

#ifdef USE_WARMSTART_V
	cuda_divergence_warmstart_init(m_data);

	std::chrono::steady_clock::time_point m0 = std::chrono::steady_clock::now();
	cuda_divergence_compute<true>(m_data);
#endif

	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();
	//////////////////////////////////////////////////////////////////////////
	// Compute velocity of density change
	//////////////////////////////////////////////////////////////////////////
	cuda_divergence_init(m_data);

	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();

	unsigned int m_iterationsV = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	// use maximal density error divided by time step size
	const RealCuda eta = maxError * 0.01 * density0 / h;  // maxError is given in percent
	//std::cout << "divergence eta: " << eta << std::endl;

	float time_3_1 = 0;
	float time_3_2 = 0;
	RealCuda avg_density_err = 0.0;
	while (((avg_density_err > eta) || (m_iterationsV < 3)) && (m_iterationsV < maxIter))
	{

		//////////////////////////////////////////////////////////////////////////
		// Perform Jacobi iteration over all blocks
		//////////////////////////////////////////////////////////////////////////
		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
		cuda_divergence_compute<false>(m_data);
		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();

		avg_density_err = cuda_divergence_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();

		avg_density_err /= numParticles;
		m_iterationsV++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
	}

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_0 = std::chrono::duration_cast<std::chrono::nanoseconds> (m0 - start).count() / 1000000.0f;
	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - m0).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

	std::cout << "detail pressure solve (iter total (varible_comp warm_comp init actual_comp (t1 t2))): " << m_iterationsV << "  " << time_0+ time_1 + time_2 + time_3 <<
	"  (" << time_0 << "  " << time_1 << "  " << time_2 << "  " << time_3 << "(" << time_3_1 << " " << time_3_2 << ") )" << std::endl;

	//*/
	return m_iterationsV;
}

////////////////////////////////////////////////////
/////////          DENSITY SOLVER      /////////////
////////////////////////////////////////////////////

template <bool warm_start> __device__ void pressureSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? particleSet->kappa[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART
	//if (!warm_start) { particleSet->kappa[i] += ki; } //moved to the evaluation
#endif


	Vector3d v_i = Vector3d(0, 0, 0);
	const Vector3d &xi = particleSet->pos[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);

#ifdef USE_BOUNDARIES_DYNAMIC_PROPERTiES
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);
#endif

	if (fabs(ki) > m_eps)
	{
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////

#ifndef USE_BOUNDARIES_DYNAMIC_PROPERTiES

#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;

		v_i += ki * mass * KERNEL_GRAD_W(m_data, xi - xj);

#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			v_i += ki * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		);
#endif


#endif


		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			Vector3d delta = ki * body.getMass(neighborIndex) * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density

					 //we apply the force to the body particle (no invH since it has been fatorized at the end)
		delta *= -particleSet->getMass(i);
		atomicAdd(&(body.F[neighborIndex].x), delta.x);
		atomicAdd(&(body.F[neighborIndex].y), delta.y);
		atomicAdd(&(body.F[neighborIndex].z), delta.z);
		);
	}

	// Directly update velocities instead of storing pressure accelerations
	particleSet->vel[i] += v_i*m_data.h_future;
}



//WARNING !!! this is not suposed to be called for the fluid this function is used for boundaries and object for witch 
//doing the velocity variation computation makes no sense but still need the accumulation of kappa for the warm start
__global__ void DFSPH_density_accumulate_kappa_kernel(SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	const RealCuda ki = (particleSet->densityAdv[i])*particleSet->factor[i];
	particleSet->kappa[i] += ki;
}

template<bool warmstart> __global__ void DFSPH_pressure_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }
	
	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active)) { return; }
	}

	pressureSolveParticle<warmstart>(m_data, particleSet, i);

}

template<bool warmstart> void cuda_pressure_compute(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		if (!warmstart) {
			int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
			DFSPH_density_accumulate_kappa_kernel << <numBlocks, BLOCKSIZE >> > (data.boundaries_data[0].gpu_ptr);
		}
	}

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_pressure_compute<true>(SPH::DFSPHCData& data);
template void cuda_pressure_compute<false>(SPH::DFSPHCData& data);


__global__ void DFSPH_pressure_compute_p1_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	{
		//////////////////////////////////////////////////////////////////////////
		// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
		//////////////////////////////////////////////////////////////////////////
		const Vector3d& xi = particleSet->pos[i];
		const Vector3d& vi = particleSet->vel[i];

		RealCuda density = particleSet->getMass(i) * m_data.W_zero;

		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data, particleSet, i);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			i,
			const Vector3d & xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data, xi - xj);
			);


		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			const Vector3d & xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data, xi - xj);
			);


		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////
		//*
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			const Vector3d & xj = body.pos[neighborIndex];
		density += body.getMass(neighborIndex) * KERNEL_W(m_data, xi - xj);
			);
		//*/


		//////////////////////////////////////////////////////////////////////////
		// Compute pressure stiffness denominator
		//////////////////////////////////////////////////////////////////////////
		particleSet->density[i] = density;
		RealCuda C = density / m_data.density0 - 1;
		RealCuda lambda = 0;
		if (C > 0) {
			lambda = m_data.density0 * m_data.density0 * particleSet->factor[i] * C * m_data.h_future* m_data.h_future;
		}

		//let's store it in the density adv for now
		particleSet->densityAdv[i] = lambda;
		
	}

}

__global__ void DFSPH_pressure_compute_p2_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	Vector3d dx(0, 0, 0);
	const Vector3d& xi = particleSet->pos[i];
	RealCuda lambda_i = particleSet->densityAdv[i];
	{
		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data, particleSet, i);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			i,
			const Vector3d & xj = body.pos[neighborIndex];
			RealCuda lambda_sum = lambda_i + body.densityAdv[neighborIndex];
			if (fabs(lambda_sum) > m_eps) 
			{
				dx += body.getMass(neighborIndex) * lambda_sum * KERNEL_GRAD_W(m_data, xi - xj);
			}
		);

		if (fabs(lambda_i) > m_eps) 
		{
			//////////////////////////////////////////////////////////////////////////
			// Boundary
			//////////////////////////////////////////////////////////////////////////
			ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
				i,
				const Vector3d & xj = body.pos[neighborIndex];
				dx += body.getMass(neighborIndex) * lambda_i * KERNEL_GRAD_W(m_data, xi - xj);
			);


			//////////////////////////////////////////////////////////////////////////
			// Dynamic bodies
			//////////////////////////////////////////////////////////////////////////
			//*
			ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
				i,
				const Vector3d & xj = body.pos[neighborIndex];
				dx += body.getMass(neighborIndex) * lambda_i * KERNEL_GRAD_W(m_data, xi - xj);
			);
			//*/
		}

	}
	dx /= m_data.density0;
	m_data.posBufferGroupedDynamicBodies[i]=dx;

}

__global__ void DFSPH_pressure_compute_p3_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	particleSet->pos[i] += data.posBufferGroupedDynamicBodies[i];

}

//this is the position based condition
//paper: Physics - Based Simulation of Ocean Scenes in Marine Simulator Visual System
void cuda_pressure_compute_v2(SPH::DFSPHCData& data) {
	//comp the necessary values
	{
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_compute_p1_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
		gpuErrchk(hipDeviceSynchronize());
	}

	//comp the particle displacement
	{
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_compute_p2_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
		gpuErrchk(hipDeviceSynchronize());
	}

	//affect the displacmeent
	{
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_compute_p3_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
		gpuErrchk(hipDeviceSynchronize());
	}

	/*
	RealCuda displacement = 0;
	for (int i = 0; i < data.fluid_data->numParticles; ++i) {
		displacement += data.posBufferGroupedDynamicBodies[i].norm();
	}
	std::cout << "total displacmeent: " << displacement << std::endl;
	//*/

}

__device__ void computeDensityAdv(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
	const Vector3d xi = particleSet->pos[index];
	const Vector3d vi = particleSet->vel[index];
	RealCuda delta = 0;


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, index);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		index,
		delta += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
	);

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES
	const Vector3d& xj = particleSet->X_rigids[index];
	const RealCuda mass = particleSet->V_rigids[index] * particleSet->density0;

	delta += mass * (vi - xj).dot(KERNEL_GRAD_W(m_data, xi - xj));

#else
	
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		index,
		delta += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
	);
#endif

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
		index,
		delta += body.getMass(neighborIndex) * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
	)

		particleSet->densityAdv[index] = MAX_MACRO_CUDA(particleSet->density[index] + m_data.h_future*delta - m_data.density0, 0.0);


#ifdef USE_WARMSTART
	particleSet->kappa[index] += (particleSet->densityAdv[index]) * particleSet->factor[index];
	//particleSet->kappa[index] = 0;
#endif
}


__global__ void DFSPH_pressure_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE) &&
			(particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE_NEIGHBORS)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active + m_data.count_active_neighbors)) { return; }
	}

#ifdef USE_WARMSTART
	particleSet->kappa[i] = 0;
#endif

	particleSet->factor[i] *= m_data.invH_future;

	computeDensityAdv(m_data, particleSet, i);


}




void cuda_pressure_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_pressure_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE) &&
			(particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE_NEIGHBORS)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active + m_data.count_active_neighbors)) { return; }
	}

#ifdef USE_POSITION_BASED_DENSITY_CONTRAINT
	particleSet->densityAdv[i] = MAX_MACRO_CUDA(particleSet->density[i] - m_data.density0, 0.0);
#else
	computeDensityAdv(m_data, particleSet, i);
#endif 
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}

RealCuda cuda_pressure_loop_end(SPH::DFSPHCData& data) {

	std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();

	RealCuda* avg_density_err = SVS_CU::get()->avg_density_err;

	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr, avg_density_err);
	}



	/*
	///LOL the detailed implementation is slower so no need to even think about developping data
	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles, data.posFluid, data.velFluid,
	data.neighbourgs, data.numberOfNeighbourgs,
	data.mass, data.m_kernel_precomp, data.boundaryPsi, data.posBoundary, data.velBoundary,
	data.vector_dynamic_bodies_data_cuda, data.densityAdv, data.density, data.h_future, data.density0);
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();

	// Run sum-reduction
	hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));


	std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
	float time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
	float time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;

	//std::cout << "pressure loop end details: " << time1 << "  " << time2 << std::endl;

	return result;
}


int cuda_pressureSolve(SPH::DFSPHCData& m_data, const unsigned int m_maxIterations, const RealCuda m_maxError) {
	const RealCuda density0 = m_data.density0;
	int numParticles = (int)m_data.fluid_data[0].numParticles;
	if (m_data.true_particle_count >= 0) {
		numParticles = m_data.true_particle_count;
	}

	RealCuda avg_density_err = 0.0;

	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


#ifdef USE_WARMSTART		
	cuda_pressure_compute<true>(m_data);
#endif

	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();

	//////////////////////////////////////////////////////////////////////////
	// Compute rho_adv
	//////////////////////////////////////////////////////////////////////////
	cuda_pressure_init(m_data);


	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();


	unsigned int m_iterations = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	const RealCuda eta = m_maxError * 0.01 * density0;  // maxError is given in percent
	//std::cout << "density eta: " << eta << std::endl;
	static int count_step = 0; count_step++;

	float time_3_1 = 0;
	float time_3_2 = 0;
	while (((avg_density_err > eta) || (m_iterations < 2)) && (m_iterations < m_maxIterations))
	{

		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
#ifdef USE_POSITION_BASED_DENSITY_CONTRAINT
		cuda_pressure_compute_v2(m_data);
#else
		cuda_pressure_compute<false>(m_data);
#endif // DEBUG

		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
		avg_density_err = cuda_pressure_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
		//std::cout << "total density err: " << avg_density_err << std::endl;
		avg_density_err /= numParticles;

		m_iterations++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;



		//std::cout << "see density convergence: " << count_step << "  " << m_iterations << " " << avg_density_err << std::endl;
		
		//std::cout << "presure solve out: " << m_iterations << "   " << avg_density_err << std::endl;
	}
	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - start).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

	std::cout << "detail pressure solve (iter total (warm init actual_comp (t1 t2))): " <<m_iterations <<"  "<< time_1 + time_2 +time_3 <<
	"  (" << time_1 << "  " << time_2<< "  "<< time_3 <<"("<< time_3_1<<" "<< time_3_2<<") )" << std::endl;

	//*/

	return m_iterations;

}


// also prepare the normals for the adhesion force
__global__ void DFSPH_viscosityXSPH_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active)) { return; }
	}

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	Vector3d ni = Vector3d(0, 0, 0);
	const Vector3d &xi = particleSet->pos[i];
	const Vector3d &vi = particleSet->vel[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
//*	

	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	//*
	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		Vector3d xixj = xi - body.pos[neighborIndex];
	RealCuda mass_div_density = body.getMass(neighborIndex) / body.density[neighborIndex];
	ai -= m_data.invH * m_data.viscosity * (mass_div_density) * (vi - body.vel[neighborIndex]) * KERNEL_W(m_data,xixj);
	ni += mass_div_density * KERNEL_GRAD_W(m_data,xixj);
	)
		//*/
		/*
		//viscosity only
		ITER_NEIGHBORS_FLUID(
		i,
		ai -= m_data.invH * m_data.viscosity * (body.getMass(neighborIndex) / body.density[neighborIndex]) *
		(vi - body.vel[neighborIndex]) * KERNEL_W(m_data,xi - body.pos[neighborIndex]);

		)//*/

		particleSet->acc[i] = m_data.gravitation + ai;

	/*
	//I'm gona use the vector3D used for the agglomerated neigbor search to store the normals
	ni *= m_data.getKernelRadius();
	m_data.posBufferGroupedDynamicBodies[i] = ni;
	//*/
}


__global__ void DFSPH_applySurfaceAkinci2013SurfaceTension_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	//for more lisability of the code
	Vector3d* normals = m_data.posBufferGroupedDynamicBodies;
	RealCuda supportRadius = m_data.getKernelRadius();
	RealCuda k = m_data.getSurfaceTension();
	RealCuda density0 = m_data.density0;

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	Vector3d ni = normals[i];
	RealCuda rhoi = particleSet->density[i];
	const Vector3d &xi = particleSet->pos[i];

	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		RealCuda K_ij = 2.0*density0 / (rhoi + body.density[neighborIndex]);

	Vector3d accel = Vector3d(0, 0, 0);


	// Cohesion force
	Vector3d xixj = xi - body.pos[neighborIndex];
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		accel -= k * body.getMass(neighborIndex) * xixj * m_data.WCohesion(xixj);
	}

	// Curvature
	accel -= k * supportRadius* (ni - normals[neighborIndex]);

	ai += K_ij * accel;
	//*/
	);
	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES

	const Vector3d& xj = particleSet->X_rigids[i];
	const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
	
	Vector3d xixj = (xi - xj);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real)1.0 / sqrt(length2)) * xixj;
		ai -= k * mass * xixj * m_data.WAdhesion(xixj);
	}

#else

	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		// adhesion force
		Vector3d xixj = (xi - body.pos[neighborIndex]);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		ai -= k * body.getMass(neighborIndex) * xixj * m_data.WAdhesion(xixj);
	}
	);


#endif


	//////////////////////////////////////////////////////////////////////////
	// Dynamic Bodies
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
		i,
		// adhesion force
		Vector3d xixj = (xi - body.pos[neighborIndex]);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		ai -= k * body.getMass(neighborIndex) * xixj * m_data.WAdhesion(xixj);
	}
	);

	particleSet->acc[i] += ai;


}


void cuda_externalForces(SPH::DFSPHCData& data) {
	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
	DFSPH_viscosityXSPH_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_viscosityXSPH failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//end the computations for the surface tension

	//DFSPH_applySurfaceAkinci2013SurfaceTension_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	gpuErrchk(hipDeviceSynchronize());
}




////////////////////////////////////////////////////
/////////         NEIGHBORS SEARCH     /////////////
////////////////////////////////////////////////////


__global__ void DFSPH_fill_aggregated_pos_buffer_kernel(SPH::DFSPHCData data, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	if (data.is_fluid_aggregated) {
		if (i<data.fluid_data_cuda->numParticles) {

			//writte de pos
			data.posBufferGroupedDynamicBodies[i] = data.fluid_data_cuda->pos[i];

			return;
		}
	}

	//find the current dynamic body
	int count_particles_previous_bodies = (data.is_fluid_aggregated) ? data.fluid_data_cuda->numParticles : 0;
	int body_id = 0;
	while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<=i) {
		count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
		body_id++;
	}

	//writte de pos
	data.posBufferGroupedDynamicBodies[i] = data.vector_dynamic_bodies_data_cuda[body_id].pos[i - count_particles_previous_bodies];
}




template<unsigned int grid_size, bool z_curve>
__global__ void DFSPH_computeGridIdx_kernel(Vector3d* in, unsigned int* out, RealCuda kernel_radius, unsigned int num_particles,
	Vector3i gridOffset) {

#ifndef INDEX_NEIGHBORS_CELL_FROM_STORAGE


	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//i *= 4;
	if (i >= num_particles) { return; }

	if (z_curve) {

	}
	else {
		//the offset is used to be able to use a small grid bu placing the simulation correctly inside it
		Vector3d pos = (in[i] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		/*
		pos = (in[i + 1] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 1] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);

		pos = (in[i + 2] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 2] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);

		pos = (in[i + 3] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 3] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		//*/
	}


#endif // !INDEX_NEIGHBORS_CELL_FROM_STORAGE
}




void cuda_neighborsSearchInternal_sortParticlesId(Vector3d* pos, RealCuda kernel_radius, Vector3i gridOffset, int numParticles,
	void **d_temp_storage_pair_sort, size_t   &temp_storage_bytes_pair_sort,
	unsigned int* cell_id, unsigned int* cell_id_sorted,
	unsigned int* p_id, unsigned int* p_id_sorted) {
	hipError_t cudaStatus;

#ifdef INDEX_NEIGHBORS_CELL_FROM_STORAGE
	exit(2563);
#endif // INDEX_NEIGHBORS_CELL_FROM_STORAGE


	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	int numBlocks = calculateNumBlocks(numParticles);


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel<CELL_ROW_LENGTH, false> << <numBlocks, BLOCKSIZE >> > (pos, cell_id,
		kernel_radius, numParticles, gridOffset);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//std::chrono::steady_clock::time_point middle = std::chrono::steady_clock::now();

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
		cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
	//*/

	hipGetLastError();
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sort failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle).count() / 1000000.0f;

	time_avg += time0 + time1;
	printf("cuda_neighborsSearchInternal_sortParticlesId: %f ms (%f,%f)   avg: %f ms \n", time0 + time1, time0, time1, time_avg / time_count);
	//*/

}


void cuda_neighborsSearchInternal_computeCellStartEnd(int numParticles, unsigned int* cell_id_sorted,
	unsigned int* hist, void **d_temp_storage_cumul_hist, size_t   &temp_storage_bytes_cumul_hist, unsigned int* cell_start_end) {
	hipError_t cudaStatus;
	int numBlocks = calculateNumBlocks(numParticles);


	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));
	gpuErrchk(hipDeviceSynchronize());

	//compute the actual histogram (done here with atomic adds)
	//*
	DFSPH_Histogram_kernel << <numBlocks, BLOCKSIZE >> > (cell_id_sorted, hist, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "histogram failed: " << (int)cudaStatus << std::endl;
		exit(1598);
	}//*/

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cumulative histogram failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}



__global__ void DFSPH_computeGridIdx_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	//the offset is used to be able to use a small grid bu placing the simulation correctly inside it
	Vector3d pos = (particleSet->pos[i] / data.getKernelRadius()) + data.gridOffset;
	pos.toFloor();
	particleSet->neighborsDataSet->cell_id[i] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		
	//we can accumulate directly here
	particleSet->neighborsDataSet->cell_id_sorted[i] =
		atomicAdd(&(particleSet->neighborsDataSet->hist[particleSet->neighborsDataSet->cell_id[i]]), 1);

}


__global__ void DFSPH_CountingSortIds_kernel(SPH::UnifiedParticleSet* particleSet) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	int new_pos=particleSet->neighborsDataSet->cell_start_end[particleSet->neighborsDataSet->cell_id[i]] + particleSet->neighborsDataSet->cell_id_sorted[i];
	particleSet->neighborsDataSet->p_id_sorted[new_pos] = i;
}

void cuda_neighborsSearchInternal_sortParticlesId(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet, SPH::DFSPHCData& data) {
	hipError_t cudaStatus;

	//for (int i = 0; i < particleSet.numParticles; ++i)
	

	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	
	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(particleSet.neighborsDataSet->hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));
	gpuErrchk(hipDeviceSynchronize());

	std::chrono::steady_clock::time_point middle1 = std::chrono::steady_clock::now();

	int numBlocks = calculateNumBlocks(particleSet.numParticles);


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel << <numBlocks, BLOCKSIZE >> > (data, particleSet.gpu_ptr);
	gpuErrchk(hipDeviceSynchronize());

	std::chrono::steady_clock::time_point middle2 = std::chrono::steady_clock::now();

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(particleSet.neighborsDataSet->d_temp_storage_cumul_hist, particleSet.neighborsDataSet->temp_storage_bytes_cumul_hist, 
		particleSet.neighborsDataSet->hist, particleSet.neighborsDataSet->cell_start_end, (CELL_COUNT + 1));
	gpuErrchk(hipDeviceSynchronize());

	

	DFSPH_CountingSortIds_kernel << <numBlocks, BLOCKSIZE >> > (particleSet.gpu_ptr);
	gpuErrchk(hipDeviceSynchronize());

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	float time2;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle1 - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle2 - middle1).count() / 1000000.0f;
	time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle2).count() / 1000000.0f;

	time_avg += time0 + time1 + time2;
	printf("cuda_neighborsSearchInternal_sortParticlesId: %f ms (%f,%f,%f)   avg: %f ms \n", time0 + time1 + time2, time0, time1, time2, time_avg / time_count);
	//*/

}




void cuda_initNeighborsSearchDataSet(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet,
	SPH::DFSPHCData& data, bool sortBuffers) {



	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(particleSet, dataSet, data);

	//cuda_neighborsSearchInternal_sortParticlesId(particleSet.pos, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
	//	&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
	//	dataSet.p_id, dataSet.p_id_sorted);
	std::chrono::steady_clock::time_point middle1 = std::chrono::steady_clock::now();

	//since it the init iter I'll sort both even if it's the boundaries
	if (sortBuffers) {
		cuda_sortData(particleSet, dataSet.p_id_sorted);
	}

	std::chrono::steady_clock::time_point middle2 = std::chrono::steady_clock::now();


	//and now I cna compute the start and end of each cell :)
	//cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
	//	&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);





	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	float time2;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle1 - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle2 - middle1).count() / 1000000.0f;
	time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle2).count() / 1000000.0f;

	time_avg += time0 + time1 + time2;
	printf("Time to generate cell start end internal: %f ms (%f,%f,%f)   avg: %f ms \n", time0 + time1 + time2, time0, time1, time2, time_avg / time_count);
	//*/

}

void cuda_initNeighborsSearchDataSetGroupedDynamicBodies(SPH::DFSPHCData& data) {
	if (data.numDynamicBodies<1) {
		return;
	}

	SPH::NeighborsSearchDataSet& dataSet = *(data.neighborsDataSetGroupedDynamicBodies);


	//before anything I need to update the number of active particles
	int numParticles = (data.is_fluid_aggregated) ? data.fluid_data[0].numParticles : 0;
	for (int i = 0; i<data.numDynamicBodies; ++i) {
		numParticles += data.vector_dynamic_bodies_data[i].numParticles;
	}

	if (dataSet.numParticles != numParticles) {
		if (numParticles <= (int)dataSet.numParticlesMax) {
			dataSet.updateActiveParticleNumber(numParticles);
		}
		else {
			std::ostringstream oss;
			oss << "TODO::I need to add particles to the grouped data struct when the number of particle goes above the max" <<
				" current max: " << dataSet.numParticlesMax << "  number of particles: " << numParticles << std::endl;
			throw(oss.str());
		}
	}

	// now fill itr
	int numBlocks = calculateNumBlocks(dataSet.numParticles);
	DFSPH_fill_aggregated_pos_buffer_kernel << <numBlocks, BLOCKSIZE >> > (data, dataSet.numParticles);
	gpuErrchk(hipDeviceSynchronize());

	//and now we can do the neighbor search
	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(data.posBufferGroupedDynamicBodies, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
		&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
		dataSet.p_id, dataSet.p_id_sorted);



	//and now I cna compute the start and end of each cell :)
	cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
		&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);



}

template<typename T>
__global__ void DFSPH_sortFromIndex_kernel(T* in, T* out, unsigned int* index, unsigned int nbElements) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nbElements) { return; }

	out[i] = in[index[i]];
}



void cuda_sortData(SPH::UnifiedParticleSet& particleSet, unsigned int * sort_id) {
	//*
	/*
	static std::chrono::steady_clock::time_point start;
	static std::chrono::steady_clock::time_point end;
	start = std::chrono::steady_clock::now();
	//*/
	unsigned int numParticles = particleSet.neighborsDataSet->numParticles;
	int numBlocks = calculateNumBlocks(numParticles);
	unsigned int *p_id_sorted = sort_id;

	Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
	RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//color
	if (particleSet.has_color_buffer) {
		DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.color, intermediate_buffer_v3d, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.color, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	}
	
	//mass
	DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));


	if (particleSet.has_factor_computation) {
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	}



	//now that everything is sorted we can set each particle index to itself
	gpuErrchk(hipMemcpy(p_id_sorted, particleSet.neighborsDataSet->p_id, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice));

	/*
	end = std::chrono::steady_clock::now();

	float time_iter = std::chrono::duration_cast<std::chrono::nanoseconds> (end-start).count() / 1000000.0f;
	static std::vector<float> timmings;

	timmings.push_back(time_iter);

	static int count = 0;
	count++;

	if (count==1300) {
		for (int i = 0; i < 1300; ++i) {
			std::cout << i << " " << timmings[i] << std::endl;
		}
	}//*/
}



__global__ void generateShuffleIndex_kernel(unsigned int *shuffle_index, unsigned int nbElements, hiprandState *state) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= 1) { return; }

	for (int j = 0; j < nbElements; ++j) {
		shuffle_index[j] = j;
	}



	hiprandState localState = *state;
	for (int j = 0; j < nbElements; ++j) {
		float x = hiprand_uniform(&localState);
		x *= nbElements;
		unsigned int idx = x;
		if (x < nbElements) {
			unsigned int temp = shuffle_index[idx];
			shuffle_index[idx] = shuffle_index[i];
			shuffle_index[i] = temp;
		}
	}
	*state = localState;
}


void cuda_shuffleData(SPH::UnifiedParticleSet& particleSet) {
	unsigned int numParticles = particleSet.numParticles;
	int numBlocks = calculateNumBlocks(numParticles);

	//create a random sorting index
	unsigned int* shuffle_index = SVS_CU::get()->shuffle_index;
	hiprandState *state = SVS_CU::get()->curand_state;
	if (shuffle_index == NULL) {
		hipMallocManaged(&(SVS_CU::get()->shuffle_index), particleSet.numParticlesMax * sizeof(unsigned int));
		shuffle_index = SVS_CU::get()->shuffle_index;
		gpuErrchk(hipDeviceSynchronize());
	}


	generateShuffleIndex_kernel << <1, 1 >> > (shuffle_index, numParticles, state);
	gpuErrchk(hipDeviceSynchronize());


	unsigned int *p_id_sorted = shuffle_index;

	Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
	RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//mass
	DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

	if (particleSet.has_factor_computation) {
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	}



}






template <bool is_fluid_container>
__global__ void DFSPH_neighborsSearch_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	ITER_NEIGHBORS_INIT_FROM_STRUCTURE(data, particleSet, i);

	//this variable is need for the interleave but I'm not modifying all the macro for only a single case
#ifdef INTERLEAVE_NEIGHBORS
	int numParticles = particleSet->numParticles;
#endif

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects =0;
	int* cur_neighbor_ptr= particleSet->getNeighboursPtr(i);
	//int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
	//int neighbors_boundary[MAX_NEIGHBOURS];


	
	if (data.is_fluid_aggregated) {
		int neighbors_solids[MAX_NEIGHBOURS];

		//dynamic bodies
		if (data.numDynamicBodies >0) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
			ITER_NEIGHBORS_FROM_STRUCTURE(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
				if (j<data.fluid_data_cuda->numParticles) {
					if (i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; }
				}
				else {
					int body_id = 0; int count_particles_previous_bodies = data.fluid_data_cuda->numParticles;
					while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<=j) {
						count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
						body_id++;
					}
					//*cur_neighbor_ptr++ = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j-count_particles_previous_bodies);
					neighbors_solids[nb_neighbors_dynamic_objects] = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j - count_particles_previous_bodies);
					nb_neighbors_dynamic_objects++;
				})
#else
			for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
				ITER_NEIGHBORS_FROM_STRUCTURE(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
					*cur_neighbor_ptr++ = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j); nb_neighbors_dynamic_objects++; )
			}
#endif

		}
		else {
			//fluid
			ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
				if (i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });
		}

		//boundaries
#ifndef BENDER2019_BOUNDARIES
		ITER_NEIGHBORS_FROM_STRUCTURE(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; );
#endif

		//copy the dynamic bodies at the end
		for (int j = 0; j<nb_neighbors_dynamic_objects; ++j) {
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbors_solids[j]);
		}

	}
	else {

		//uses the standart version
		//fluid
		if (is_fluid_container) {
#ifndef NEIGHBORS_RANGE_EXPLORATION
			ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
				if (!is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });
#else
			//*
			if (true) {
				//here is a code that can explore chains of cells for hilber index
				bool found_none = true;
				int8_t count_analysed = 0;
				int last = -1;
				int min = 0;
				do {
					found_none = true;
					//get the minimum of the remainings
					int8_t k_min, m_min, n_min;
					for (int8_t k = -1; k < 2; ++k) {
						for (int8_t m = -1; m < 2; ++m) {
							for (int8_t n = -1; n < 2; ++n) {
								int id = COMPUTE_CELL_INDEX(x + n, y + k, z + m);
								if (id > last) {
									if (found_none) {
										min = id;
										found_none = false;
									}
									else if (id < min) {
										min = id;
									}
								}
							}
						}
					}

					if (found_none) {
						break;
					}
					last = min;

					unsigned int end = data.fluid_data_cuda->neighborsDataSet->cell_start_end[min + 1];
					for (unsigned int cur_particle = data.fluid_data_cuda->neighborsDataSet->cell_start_end[min]; cur_particle < end; ++cur_particle) {
						unsigned int j = data.fluid_data_cuda->neighborsDataSet->p_id_sorted[cur_particle];
						if ((pos - data.fluid_data_cuda->pos[j]).squaredNorm() < radius_sq) {
							if (!is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; };
						}
					}

					count_analysed++;;

					//printf("count analysed // count contiguous: %i // %i\n", count_analysed, count_contiguous + 1);
				} while (count_analysed != 27);


					/*
					int count_found = 0;
					ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
						if (!is_fluid_container || i != j) { count_found++; });

					if (nb_neighbors_fluid != count_found) {
						//printf("FLUID pb when storing neighbors old/new: %i // %i \n", count_found, nb_neighbors_fluid);

						int count_duplicate = 0;
						for (int k = 0; k < nb_neighbors_fluid; ++k) {
							int* neighbors_ptr = particleSet->getNeighboursPtr(i);
							if (neighbors_ptr[k*numParticles] == i) {
								printf("FLUID particle %i is inside it's own neighbors", i);
							}

							for (int l = k+1; l < nb_neighbors_fluid; ++l) {
								if (neighbors_ptr[k*numParticles] == neighbors_ptr[l*numParticles]) {
									count_duplicate++;
								}
							}
						}
						if (count_duplicate > 0) {
							printf("FLUID particle %i has duplicates inside it's neighboors, count: %i\n", i, count_duplicate);
						}
					}
					//*/
			}else {
				ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
					if (!is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });

			}
#endif
		}

		//boundaries
#ifndef BENDER2019_BOUNDARIES

#ifndef NEIGHBORS_RANGE_EXPLORATION
		ITER_NEIGHBORS_FROM_STRUCTURE(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
			if (is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; });
#else
		{
			//here is a code that can explore chains of cells for hilber index
			bool found_none = true;
			int8_t count_analysed = 0;
			int last = -1;
			int min = 0;
			do {
				found_none = true;
				//get the minimum of the remainings
				int8_t k_min, m_min, n_min;
				for (int8_t k = -1; k < 2; ++k) {
					for (int8_t m = -1; m < 2; ++m) {
						for (int8_t n = -1; n < 2; ++n) {
							int id = COMPUTE_CELL_INDEX(x + n, y + k, z + m);


							if (id > last) {
								if (found_none) {
									min = id;
									found_none = false;
								}
								else if (id < min) {
									min = id;
								}
							}
						}
					}
				}

				if (found_none) {
					break;
				}
				last = min;

				unsigned int end = data.boundaries_data_cuda->neighborsDataSet->cell_start_end[min + 1];
				for (unsigned int cur_particle = data.boundaries_data_cuda->neighborsDataSet->cell_start_end[min]; cur_particle < end; ++cur_particle) {
					unsigned int j = data.boundaries_data_cuda->neighborsDataSet->p_id_sorted[cur_particle];
					if ((pos - data.boundaries_data_cuda->pos[j]).squaredNorm() < radius_sq) {
						if (is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; };
					}
				}

				count_analysed ++;

				//printf("count analysed // count contiguous: %i // %i\n", count_analysed, count_contiguous + 1);
			} while (count_analysed != 27);

			/*
			//do a check
			int count_found = 0;
			ITER_NEIGHBORS_FROM_STRUCTURE(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
				if (is_fluid_container || i != j) { count_found++; });

			if (nb_neighbors_boundary != count_found) {
				printf("BOUNDARIES pb when storing neighbors old/new: %i // %i \n", count_found, nb_neighbors_boundary);
			}
			//*/

		}

#endif

#endif

		if (data.numDynamicBodies > 0) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
			ITER_NEIGHBORS_FROM_STRUCTURE(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
			{ int body_id = 0; int count_particles_previous_bodies = 0;
			while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<=j) {
				count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
				body_id++;
			}
			int neighbor_idx= WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j - count_particles_previous_bodies);
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbor_idx);
			nb_neighbors_dynamic_objects++; })
#else
			for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
				ITER_NEIGHBORS_FROM_STRUCTURE(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
					{int neighbor_idx = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j);
				    WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbor_idx);
					nb_neighbors_dynamic_objects++;

					//printf("wrote a neighbor %i\n", nb_neighbors_boundary+nb_neighbors_fluid+nb_neighbors_dynamic_objects);
					});
			}
#endif

		}

	}

#ifdef SORT_NEIGHBORS
	//let's use an insertion sort
	//and for now i'll only sort the fluid and boundarries neighbors

	//the fluid
	int* neighbors_ptr = particleSet->getNeighboursPtr(i);
	for (int k = 1; k < nb_neighbors_fluid; ++k) {
		if (neighbors_ptr[k*numParticles] < neighbors_ptr[(k - 1)*numParticles]) {
			int stored_idx = neighbors_ptr[k*numParticles];
			int l = k;
			do {
				neighbors_ptr[l*numParticles] = neighbors_ptr[(l - 1)*numParticles];
				l--;
			} while ((l > 0) && (stored_idx < neighbors_ptr[(l - 1)*numParticles]));
			neighbors_ptr[l*numParticles] = stored_idx;
		}
	}

	//and the boundaries
	//*
	neighbors_ptr += nb_neighbors_fluid* numParticles;
	for (int k = 1; k < nb_neighbors_boundary; ++k) {
		if (neighbors_ptr[k*numParticles] < neighbors_ptr[(k - 1)*numParticles]) {
			int stored_idx = neighbors_ptr[k*numParticles];
			int l = k;
			do {
				neighbors_ptr[l*numParticles] = neighbors_ptr[(l - 1)*numParticles];
				l--;
			} while ((l > 0) && (stored_idx < neighbors_ptr[(l - 1)*numParticles]));
			neighbors_ptr[l*numParticles] = stored_idx;
		}
	}//*/

#endif // SORT_NEIGHBORS


	particleSet->numberOfNeighbourgs[3 * i] =  nb_neighbors_fluid;
	particleSet->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	particleSet->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;
	
	
	
	/*
	//simple splashless surface detection
	if (((nb_neighbors_fluid+nb_neighbors_boundary) < 35)&& (nb_neighbors_fluid + nb_neighbors_boundary) >15) {
		particleSet->color[i] = Vector3d(0, 1, 0);
	}
	//*/

	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS*2), neighbors_fluid, sizeof(int)*nb_neighbors_fluid);
	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS), neighbors_boundary, sizeof(int)*nb_neighbors_boundary);


}


///WARNING this version of the neighbor storage is special 
//it will fit what was given as parameter and as such may not correspond to the usual fluid/boudaries/solids
//the last parameter is so that more that 2 set can be given as parameters
//also when stored the stored index will obviously use the same compression method as the one used for the dynamic bodies in the noraml execution
__global__ void DFSPH_neighborsSearchSelf_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet_self,
	SPH::UnifiedParticleSet* particleSet_other,
	SPH::UnifiedParticleSet* particleSet_vect_additionals, int nb_additional_bodies, int iter_debug=-1) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet_self->numParticles) { return; }

	if (data.restriction_mode == 1) {
		int tag = particleSet_self->neighborsDataSet->cell_id[i];
		if (tag != TAG_ACTIVE && tag != TAG_ACTIVE_NEIGHBORS) {
			return;
		}
	}else if (data.restriction_mode == 2) {
		if (i >= (data.count_active + data.count_active_neighbors)) { return; }
	}

	ITER_NEIGHBORS_INIT_FROM_STRUCTURE(data, particleSet_self, i);

	//this variable is need for the interleave but I'm not modifying all the macro for only a single case
#ifdef INTERLEAVE_NEIGHBORS
	int numParticles = particleSet_self->numParticles;
#endif

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr = particleSet_self->getNeighboursPtr(i);
	//int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
	//int neighbors_boundary[MAX_NEIGHBOURS];

	//uses the standart version
	//self

	ITER_NEIGHBORS_FROM_STRUCTURE(particleSet_self->neighborsDataSet, particleSet_self->pos,
		if (i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });

		
	//other
	ITER_NEIGHBORS_FROM_STRUCTURE(particleSet_other->neighborsDataSet, particleSet_other->pos,
		{ WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; });


	//additional
	if (nb_additional_bodies > 0) {
		for (int id_body = 0; id_body < nb_additional_bodies; ++id_body) {
			ITER_NEIGHBORS_FROM_STRUCTURE(particleSet_vect_additionals[id_body].neighborsDataSet, particleSet_vect_additionals[id_body].pos,
				{ int neighbor_idx = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j);
				WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbor_idx);
				nb_neighbors_dynamic_objects++;	});
		}
	}	

	particleSet_self->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
	particleSet_self->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	particleSet_self->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;
}



__global__ void DFSPH_neighborsSearchBasic_kernel(unsigned int numFluidParticles, RealCuda radius,
	SPH::UnifiedParticleSet* fluid_data,
	SPH::UnifiedParticleSet* boundaries_data,
	SPH::UnifiedParticleSet* vect_dynamic_bodies, int nb_dynamic_bodies) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }


	RealCuda radius_sq = radius;
	Vector3d pos = fluid_data->pos[i];
	radius_sq *= radius_sq;

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr = fluid_data->neighbourgs + i*MAX_NEIGHBOURS;

	for (int k = 0; k < fluid_data->numParticles; ++k) {
		if (i != k) {
			if ((fluid_data->pos[k] - pos).squaredNorm() < radius_sq) {
				*cur_neighbor_ptr++ = k;	nb_neighbors_fluid++;
			}
		}
	}

	/*
	for (int k = 0; k < boundaries_data->numParticles; ++k) {
	if ((boundaries_data->pos[k] - pos).squaredNorm() < radius_sq) {
	*cur_neighbor_ptr++ = k; nb_neighbors_boundary++;
	}
	}
	//*/

	/*
	for (int id_body = 0; id_body < nb_dynamic_bodies; ++id_body) {
	for (int k = 0; k < vect_dynamic_bodies[id_body].numParticles; ++k) {
	if ((vect_dynamic_bodies[id_body].pos[k] - pos).squaredNorm() < radius_sq) {
	*cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, k); nb_neighbors_dynamic_objects++;
	}
	}
	}
	//*/


	fluid_data->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
	fluid_data->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	fluid_data->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;

}




void cuda_neighborsSearch(SPH::DFSPHCData& data, bool need_sort) {

	static unsigned int* precomputedIndexPtr = NULL;

	if (precomputedIndexPtr == NULL) {
		precomputedIndexPtr = data.precomputedCellIndex;
	}
	data.precomputedCellIndex = precomputedIndexPtr;

	//std::chrono::steady_clock::time_point begin_global = std::chrono::steady_clock::now();
	static unsigned int time_count = 0;
	float time_global;
	static float time_avg_global = 0;
	time_count++;

	/*
	if (time_count<5) {
	cuda_shuffleData(data.fluid_data[0]);
	std::cout << "randomizing particle order" << std::endl;
	}
	//*/


	if (need_sort) {
		//std::cout<<"doing full neighbor search"<<std::endl;
	}

	bool old_fluid_aggregated = data.is_fluid_aggregated;
	hipError_t cudaStatus;
	if (true) {
		if (need_sort&&data.is_fluid_aggregated) {
			data.is_fluid_aggregated = false;
		}


		//*
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		//*/

		//first let's generate the cell start end for the dynamic bodies
#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
		cuda_initNeighborsSearchDataSetGroupedDynamicBodies(data);
#else
		for (int i = 0; i < data.numDynamicBodies; ++i) {
			SPH::UnifiedParticleSet& body = data.vector_dynamic_bodies_data[i];
			body.initNeighborsSearchData(data, false);
		}
#endif
		std::chrono::steady_clock::time_point middle = std::chrono::steady_clock::now();

		//no need to ever do it forthe boundaries since they don't ever move

		//now update the cell start end of the fluid particles
		if ((!data.is_fluid_aggregated) || data.numDynamicBodies<1) {

			//since it the init iter I'll sort both even if it's the boundaries
			static int step_count = 0;
			step_count++;

			data.fluid_data->initNeighborsSearchData(data, need_sort);


			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "before neighbors search: %d\n", (int)cudaStatus);
				exit(1598);
			}


		}

		/*

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		float time0;
		float time1;
		static float time_avg = 0;
		time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle - begin).count() / 1000000.0f;
		time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle).count() / 1000000.0f;

		time_avg += time0+time1;
		printf("Time to generate cell start end: %f ms (%f,%f)   avg: %f ms \n", time0+time1,time0,time1, time_avg / time_count);

		if (time_count > 150) {
		time_avg = 0;
		}
		//*/


	}
	//and we can now do the actual search of the neaighbor for eahc fluid particle
#ifdef STORE_PARTICLE_NEIGHBORS

	if (true)
	{
		//*
		float time;
		static float time_avg = 0;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		//*/

		//cuda way
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);

		//*
		DFSPH_neighborsSearch_kernel<true> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data->gpu_ptr);

		//*
		if (data.boundaries_data->has_factor_computation) {
			DFSPH_neighborsSearch_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data_cuda);
		}
		gpuErrchk(hipDeviceSynchronize());
		//*/

		//*/
		/*
		//this test show that even just computing the neighbors for the fluid particle
		//with a basic method take more time than building the whole structure
		DFSPH_neighborsSearchBasic_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles,
		data.getKernelRadius(),
		data.fluid_data_cuda,
		data.boundaries_data_cuda,
		data.vector_dynamic_bodies_data_cuda, data.numDynamicBodies);
		//*/


		/*
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		printf("Time to generate neighbors buffers: %f ms   avg: %f ms \n", time, time_avg / time_count);

		if (time_count > 150) {
		time_avg = 0;
		time_count = 0;
		}
		//*/



		/*
		{
			{
			std::cout << "test: " << data.fluid_data->neighborsDataSet->cell_id_sorted[0] << "   " <<
				data.fluid_data->neighborsDataSet->cell_id_sorted[10] << "   " <<
				data.fluid_data->neighborsDataSet->cell_id_sorted[50] << "   " << std::endl;

			int count_valid = 0;
			for (int i = 0; i < data.fluid_data->numParticles; ++i) {
				if (data.fluid_data->neighborsDataSet->cell_id_sorted[i] != 0) {
					count_valid++;
				}
			}
			std::cout << "test2: " << count_valid << std::endl;
			}
		}
			//*/
		/*
		{
			//a simple check to know the max nbr of neighbors
			static int absolute_max = 0;
			int max = 0;

			static int absolute_max_d[3] = { 0 };
			int max_d[3] = { 0 };



			for (int j = 0; j < data.fluid_data->numParticles; j++)
			{

				//check the global value
				int count_neighbors = 0;
				for (int k = 0; k < 3; ++k) {
					count_neighbors += data.fluid_data->getNumberOfNeighbourgs(j, k);
				}
				if (count_neighbors > max) { 
					max = count_neighbors;
				}

				//chekc the max for each category
				for (unsigned int k = 0; k < 3; ++k) {
					if ((int)data.fluid_data->getNumberOfNeighbourgs(j,k) > max_d[k])max_d[k] = data.fluid_data->getNumberOfNeighbourgs(j,k);
				}

			}

			if (max>absolute_max)absolute_max = max;

			for (unsigned int k = 0; k < 3; ++k) {
				if (max_d[k]>absolute_max_d[k])absolute_max_d[k] = max_d[k];
			}

			printf("max nbr of neighbors %d  (%d) \n", absolute_max, max);
			printf("absolute??max  fluid // boundaries // bodies   %d // %d // %d  ?? %d // %d // %d\n",
			absolute_max_d[0], absolute_max_d[1], absolute_max_d[2], max_d[0], max_d[1], max_d[2]);
		}


		//*/
		/*
		{
		//another test ot be sure the contruction of the boundries neighbors works orrectly
		if (data.boundaries_data->has_factor_computation) {
		//a simple check to know the  nbr of neighbors of the first boundries particle

		int nb_neighbors[3] = { 0 };

		for (int k = 0; k < 3; ++k) {
		nb_neighbors[k] = data.boundaries_data->getNumberOfNeighbourgs(0, k);
		}

		printf(" nbr of neighbors %d     fluid // boundaries // bodies   %d // %d // %d\n",
		nb_neighbors[0] + nb_neighbors[1] + nb_neighbors[2], nb_neighbors[0], nb_neighbors[1], nb_neighbors[2]);
		}
		}

		//*/

		
	}
#endif //STORE_PARTICLE_NEIGHBORS

	//reactive the aggragation if we desactivated it because a sort was required
	if (need_sort&&old_fluid_aggregated) {
		data.is_fluid_aggregated = true;
	}

	data.precomputedCellIndex = NULL;

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	time_global = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin_global).count() / 1000000.0f;

	time_avg_global += time_global;
	printf("time taken by the neighbor function: %f ms   avg: %f ms \n", time_global, time_avg_global / time_count);
	//*/
}

void cuda_updateNeighborsStorage(SPH::DFSPHCData& data, SPH::UnifiedParticleSet& particleSet, int iter_debug) {
	int count_particles = particleSet.numParticles;
	if (data.restriction_mode == 2) {
		count_particles = (data.count_active + data.count_active_neighbors);
	}

	int numBlocks = calculateNumBlocks(count_particles);

	//*
	
	DFSPH_neighborsSearchSelf_kernel<< <numBlocks, BLOCKSIZE >> > (data, particleSet.gpu_ptr,
			data.boundaries_data->gpu_ptr, data.vector_dynamic_bodies_data_cuda, data.numDynamicBodies, iter_debug);

	
	gpuErrchk(hipDeviceSynchronize());
}

////////////////////////////////////////////////////
/////////             OTHERS           /////////////
////////////////////////////////////////////////////

__global__ void DFSPH_update_vel_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	if (m_data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE)) {
			return;
		}
	}
	else if (m_data.restriction_mode == 2) {
		if (i >= (m_data.count_active)) { return; }
	}

	particleSet->vel[i] += m_data.h * particleSet->acc[i];

#ifdef USE_WARMSTART	
	//done here to have one less kernel
	particleSet->kappa[i] = MAX_MACRO_CUDA(particleSet->kappa[i] * m_data.h_ratio_to_past2, -0.5);
	//this line is from the most recent impelmentation but it does not change anything
	//particleSet->kappa[i] = MAX_MACRO_CUDA(particleSet->kappa[i] * m_data.h_ratio_to_past2, -0.5 * m_data.density0*m_data.density0);
#endif

}




void cuda_update_vel(SPH::DFSPHCData& data) {
	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);


#ifdef USE_WARMSTART
	/*
	Real count_capped = 0;

	for (int i = 0; i < (data.fluid_data[0].numParticles); ++i) {
		if ( data.fluid_data->kappa[i] * data.h_ratio_to_past2 < -0.5) {
			count_capped++;
		}
	}
	//*/
#endif


	DFSPH_update_vel_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_vel failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
	

#ifdef USE_WARMSTART
	/*
	RealCuda avg_warmstart = 0;
	for (int i = 0; i < (data.fluid_data[0].numParticles); ++i) {
		avg_warmstart += data.fluid_data->kappa[i];
		//std::cout << "pressure wamstart post clamping: " << i << " " << data.fluid_data->kappa[i] << std::endl;
	}
	avg_warmstart /= data.fluid_data[0].numParticles;
	//std::cout << "pressure wamstart avg: " << avg_warmstart << std::endl;

	static std::vector<Real> avgs_warmstart;
	static std::vector<int> counts_capped;
	avgs_warmstart.push_back(avg_warmstart);
	counts_capped.push_back(count_capped);
	int count = 1000;
	if (avgs_warmstart.size() == count) {
		for (int i = 0; i < avgs_warmstart.size(); i++) {
			std::cout << "pressure_wamstart_avg: " << i << " " << avgs_warmstart[i] << " " << counts_capped[i] << std::endl;
		}
		exit(0);
	}
	//*/
#endif
}


__global__ void DFSPH_update_pos_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (data.restriction_mode == 1) {
		if ((particleSet->neighborsDataSet->cell_id[i] != TAG_ACTIVE)) {
			return;
		}
	}
	else if (data.restriction_mode == 2) {
		if (i >= (data.count_active)) { return; }
	}

	if (data.damp_borders) {
		/*
		RealCuda max_vel_sq = (data.particleRadius / 2.0f) / data.h;
		max_vel_sq *= max_vel_sq;
		RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
		if (cur_vel_sq> max_vel_sq)
		{
		particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
		}//*/

		RealCuda affected_distance_sq = data.particleRadius * 6;
		affected_distance_sq *= affected_distance_sq;

		for (int k = 0; k < data.damp_planes_count; ++k) {
			Vector3d plane = data.damp_planes[k];
			if ((particleSet->pos[i] * plane.abs() / plane.norm() - plane).squaredNorm() < affected_distance_sq) {
				if (data.damp_borders_steps_count>1) {
					RealCuda max_vel_sq = (data.particleRadius / 25.0f) / data.h;
					max_vel_sq *= max_vel_sq;
					RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
					if (cur_vel_sq> max_vel_sq)
					{
						particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
					}
					//if we triggered once no need to check for the other planes
					break;
				}
				else {
					particleSet->vel[i] *= 0.1;
				}
			}
		}
	}

	if (data.cancel_wave) {
		RealCuda affected_distance_sq = data.getKernelRadius();
		affected_distance_sq *= affected_distance_sq;
		for (int k = 0; k < 2; ++k) {
			Vector3i plane = data.cancel_wave_planes[k];
			if ((particleSet->pos[i] * plane.abs() / plane.norm() - plane).squaredNorm() < affected_distance_sq) {
				//particleSet->vel[i]=Vector3d(0,1,0);
			}
		}
		Vector3d axis = data.cancel_wave_planes[0].abs() / data.cancel_wave_planes[0].norm();
		if (particleSet->pos[i].y>data.cancel_wave_lowest_point) {
			if ((particleSet->pos[i].dot(axis))<(data.cancel_wave_planes[0].dot(axis))) {
				if ((particleSet->vel[i].dot(axis))<0) {
					particleSet->vel[i] -= particleSet->vel[i] * axis;
				}
			}

			if ((particleSet->pos[i].dot(axis))>(data.cancel_wave_planes[1].dot(axis))) {
				if ((particleSet->vel[i].dot(axis))>0) {
					particleSet->vel[i] -= particleSet->vel[i] * axis;
				}
			}
		}

	}

	if (data.restriction_mode == 1) {
		if (particleSet->pos[i].y > 1.95) {
			if (particleSet->vel[i].y > 0) {
				//particleSet->vel[i].y *= 0.25;
			}
		}
	}

	/*
	if (particleSet->density[i] < 900) {
		particleSet->color[i] = Vector3d(0, 1, 0);
	}
	//*/
	particleSet->pos[i] += data.h * particleSet->vel[i];
}



void cuda_update_pos(SPH::DFSPHCData& data) {


	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
	DFSPH_update_pos_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_pos failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	if (data.damp_borders) {
		for (int k = 0; k < data.damp_planes_count; ++k) {
			Vector3d plane = data.damp_planes[k];
			//std::cout << "damping plane: " << plane.x << "  " << plane.y << "  " << plane.z << std::endl;
		}
		data.damp_borders_steps_count--;
		if (data.damp_borders_steps_count == 0) {
			data.damp_borders = false;
			data.damp_planes_count = 0;
		}
	}
	if (data.cancel_wave) {
		//*

		for (int k = 0; k < 2; ++k) {
			Vector3d plane = data.cancel_wave_planes[k];
			std::cout << "cancel wave plane: " << plane.x << "  " << plane.y << "  " << plane.z << std::endl;
		}
		data.cancel_wave_steps_count--;
		if (data.cancel_wave_steps_count == 0) {
			data.cancel_wave = false;
		}
		//*/
	}

}




__global__ void DFSPH_CFL_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet particleSet, RealCuda* maxVel) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.fluid_data[0].numParticles) { return; }

	for (unsigned int i = 0; i < m_data.fluid_data[0].numParticles; i++)
	{
		const RealCuda velMag = (particleSet.vel[i] + particleSet.acc[i] * m_data.h).squaredNorm();
		if (velMag > *maxVel)
			*maxVel = velMag;
	}
}

__global__ void DFSPH_CFLVelSquaredNorm_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* sqaredNorm) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	sqaredNorm[i] = (particleSet->vel[i] + particleSet->acc[i] * m_data.h).squaredNorm();
}

__global__ void DFSPH_CFLAdvanced_kernel(SPH::DFSPHCData m_data, RealCuda *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ RealCuda cache[256];


	RealCuda temp = 0;
	while (index + offset < n) {
		int i = index + offset;
		const RealCuda velMag = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
		temp = fmaxf(temp, velMag);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = MAX_MACRO_CUDA(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) != 0);  //lock
		*max = MAX_MACRO_CUDA(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}

void cuda_CFL(SPH::DFSPHCData& m_data, const RealCuda minTimeStepSize, RealCuda m_cflFactor, RealCuda m_cflMaxTimeStepSize) {

	//we compute the square norm

	std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();

	RealCuda* out_buff;
	hipMalloc(&(out_buff), sizeof(RealCuda));

	if (true) {

		//cub version
		static RealCuda* temp_buff = NULL;
		if (temp_buff == NULL) {
			hipMallocManaged(&(temp_buff), m_data.fluid_data[0].numParticles * sizeof(RealCuda));
		}
		int numBlocks = calculateNumBlocks(m_data.fluid_data[0].numParticles);
		DFSPH_CFLVelSquaredNorm_kernel << <numBlocks, BLOCKSIZE >> > (m_data, m_data.fluid_data[0].gpu_ptr, temp_buff);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl squared norm failed: %d\n", (int)cudaStatus);
			exit(1598);
		}

		// Determine temporary device storage requirements
		static void     *d_temp_storage = NULL;
		static size_t   temp_storage_bytes = 0;
		if (d_temp_storage == NULL) {
			hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);
			// Allocate temporary storage
			hipMalloc(&d_temp_storage, temp_storage_bytes);
		}
		// Run max-reduction
		hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);

	}
	else {
		//manual
		int *d_mutex;
		hipMalloc((void**)&d_mutex, sizeof(int));
		hipMemset(d_mutex, 0, sizeof(float));

		int numBlocks = calculateNumBlocks(m_data.fluid_data[0].numParticles);
		DFSPH_CFLAdvanced_kernel << < numBlocks, BLOCKSIZE >> > (m_data, out_buff, d_mutex, m_data.fluid_data[0].numParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl failed: %d\n", (int)cudaStatus);
			exit(1598);
		}
		hipFree(d_mutex);
	}
	RealCuda maxVel;
	hipMemcpy(&maxVel, out_buff, sizeof(RealCuda), hipMemcpyDeviceToHost);
	hipFree(out_buff);

	std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

	RealCuda h = m_data.h;

	// Approximate max. time step size
	h = m_cflFactor * .4 * (2.0*m_data.particleRadius / (sqrt(maxVel)));

	h = min(h, m_cflMaxTimeStepSize);
	h = max(h, minTimeStepSize);

	m_data.updateTimeStep(h);//*/


	std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();



	float time_search = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
	float time_comp = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;

	printf("Time to do cfl (search,comp): %f    %f\n", time_search, time_comp);
}




//this is the bases for all kernels based function
//I also use that kernel to reset the force

__global__ void DFSPH_updateDynamicObjectParticles_kernel(int numParticles, Vector3d* pos, Vector3d* vel, Vector3d* pos0,
	Vector3d position, Vector3d velocity, Quaternion q, Vector3d angular_vel, Vector3d* F) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numParticles) { return; }

	//reset the force
	F[i] = Vector3d(0, 0, 0);

	//update location and velocity
	pos[i] = q.rotate(pos0[i]) + position;
	vel[i] = angular_vel.cross(pos[i] - position) + velocity;

}

void update_dynamicObject_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& particle_set) {


	if (particle_set.is_dynamic_object) {
		int numBlocks = calculateNumBlocks(particle_set.numParticles);


		//update the particle location and velocity
		DFSPH_updateDynamicObjectParticles_kernel << <numBlocks, BLOCKSIZE >> > (particle_set.numParticles,
			particle_set.pos, particle_set.vel, particle_set.pos0,
			particle_set.rigidBody_cpu->position, particle_set.rigidBody_cpu->velocity,
			particle_set.rigidBody_cpu->q, particle_set.rigidBody_cpu->angular_vel,
			particle_set.F);

		//also we can use that time to reset the force buffer
		//directly done in the other kernel
		//DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "update_dynamicObject_UnifiedParticleSet_cuda failed: %d\n", (int)cudaStatus);
			exit(1369);
		}
	}


}



__global__ void compute_dynamic_body_particle_mass_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	Real delta = data.W_zero;

	RealCuda radius_sq = data.getKernelRadius();
	Vector3d pos = particleSet->pos[i];
	Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
	pos_cell.toFloor();
	int x = pos_cell.x;
	int y = pos_cell.y;
	int z = pos_cell.z;
	radius_sq *= radius_sq;


	//since this version use the std index to be able to iterate on 3 successive cells
	//I can do the -1 at the start on x.
	//one thing: it x=0 then we can only iterate 2 cells at a time
	unsigned int successive_cells_count = (x > 0) ? 3 : 2;
	x = (x > 0) ? x - 1 : x;


	const SPH::UnifiedParticleSet& body = *particleSet;
	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);
			unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];
			for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];
				if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
					if (i != j) { delta += KERNEL_W(data,pos - body.pos[j]); }
				}
			}
		}
	}


	const Real volume = 1.0 / delta;
	particleSet->mass[i] = particleSet->density0 * volume;
	particleSet->mass[i] = data.fluid_data_cuda->getMass(0);
}



__global__ void refine_dynamic_body_particle_mass_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	//the factor is due to the fact that we compensate only a part of the error (proportional to the importance of the mass in the density
	//particleSet->getMass(i) += (particleSet->getMass(i) * data.W_zero / particleSet->density[i])*(data.density0 - particleSet->density[i]) / (data.W_zero);
	particleSet->mass[i] += (0.3)*(data.density0 - particleSet->density[i]) / (data.W_zero);
}


__global__ void compute_boundaries_density_error_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet, RealCuda* err, int* err_max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	RealCuda density = particleSet->getMass(i) * data.W_zero;

	//*
	ITER_NEIGHBORS_INIT(data, particleSet, i);

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_BOUNDARIES(data, particleSet,
		i,
		density += body.getMass(neighborIndex) * KERNEL_W(data,particleSet->pos[i] - body.pos[neighborIndex]);
	);
	//*/
	/*
	RealCuda radius_sq = data.getKernelRadius();
	Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
	pos_cell.toFloor();
	int x = pos_cell.x;
	int y = pos_cell.y;
	int z = pos_cell.z;
	radius_sq *= radius_sq;


	//since this version use the std index to be able to iterate on 3 successive cells
	//I can do the -1 at the start on x.
	//one thing: it x=0 then we can only iterate 2 cells at a time
	unsigned int successive_cells_count = (x > 0) ? 3 : 2;
	x = (x > 0) ? x - 1 : x;


	const SPH::UnifiedParticleSet& body = *particleSet;
	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);
			unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];
			for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];
				if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
					if (i != j) { density += particleSet->getMass(j) * KERNEL_W(data,pos - body.pos[j]); }
				}
			}
		}
	}
	//*/

	particleSet->density[i] = density;
	const RealCuda error =  abs(data.density0-density);
	atomicAdd(err, error);
	atomicMax(err_max, (int)(error*10000));
}

void compute_UnifiedParticleSet_particles_mass_cuda(SPH::DFSPHCData& data, SPH::UnifiedParticleSet& container) {
	int numBlocks = calculateNumBlocks(container.numParticles);
	
	bool old_destructor_status = data.destructor_activated;
	data.destructor_activated = false;

	container.initNeighborsSearchData(data, false, false);
	//init the neighbors
	bool fluid_agg = data.is_fluid_aggregated;
	data.is_fluid_aggregated=false;
	//DFSPH_neighborsSearch_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data_cuda);
	data.is_fluid_aggregated = fluid_agg;

	//compute_dynamic_body_particle_mass_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr);
	//gpuErrchk(hipDeviceSynchronize());



	bool refine_masses = false;//This is a test using relaxed jacobi to calculate the true mass of the particle
	if (refine_masses) {


		RealCuda* err;
		int* err_max;
		hipMallocManaged(&(err), sizeof(RealCuda));
		hipMallocManaged(&(err_max), sizeof(int));
		*err = 0.0;
		*err_max = 0;

		//calc the error on the density
		compute_boundaries_density_error_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr, err, err_max);
		gpuErrchk(hipDeviceSynchronize());


		RealCuda target_error = data.density0 / 100.0*0.1;
		RealCuda target_error_max = data.density0 / 100.0*0.1;
		RealCuda avg_err = (*err) / container.numParticles;
		RealCuda err_max_float = (*err_max)/ 10000.0;
		*err = 0;
		*err_max = 0;
		std::cout << "current density error: " << avg_err << " // " << err_max_float << "  target error: " << target_error << std::endl;

		std::chrono::steady_clock::time_point begin= std::chrono::steady_clock::now();

		//and refine it
		//while (avg_err>(target_error)|| target_error_max>15)
		for (int i =0;i<100;++i )
		{
			//refine the values
			refine_dynamic_body_particle_mass_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr);
			gpuErrchk(hipDeviceSynchronize());

			//compute the new error
			compute_boundaries_density_error_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr, err, err_max);
			gpuErrchk(hipDeviceSynchronize());

			avg_err = (*err) / container.numParticles;
			err_max_float = (*err_max)/ 10000.0;
			*err = 0;
			*err_max = 0;
			//std::cout << "current density error: " << avg_err << " // " << err_max_float << "  target error: " << target_error << std::endl;
		}

		std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
		float time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;
		std::cout << "current density error: " << avg_err << " // " << err_max_float << "  computation_time: " << time << std::endl;


		if (true ) {
			std::string filename = "boundaries density adv.csv";
			std::remove(filename.c_str());
			std::ofstream myfile;
			myfile.open(filename, std::ios_base::app);
			if (myfile.is_open()) {
				for (int i = 0; i < data.boundaries_data->numParticles; ++i) {
					myfile << i << ", " << container.getNumberOfNeighbourgs(i, 0)
						<< ", " << container.getNumberOfNeighbourgs(i, 1)
						<< ", " << container.getNumberOfNeighbourgs(i, 2)
						<< ", " << container.density[i] << std::endl;
				}
				//myfile << total_time / (count_steps + 1) << ", " << m_iterations << ", " << m_iterationsV << std::endl;;
				myfile.close();
			}
			else {
				std::cout << "failed to open file: " << filename << "   reason: " << std::strerror(errno) << std::endl;
			}
		}

		CUDA_FREE_PTR(err);
		CUDA_FREE_PTR(err_max);
	}
	data.destructor_activated = old_destructor_status;
}


